#include "hip/hip_runtime.h"
#include "trimmer.h" 

namespace cuckoogpu {
#include "siphash.cuh"

__constant__ uint2 recoveredges[PROOFSIZE];

__global__ void Recovery(const siphash_keys &sipkeys, ulonglong4 *buffer, int *indexes) {
  const int gid = blockDim.x * blockIdx.x + threadIdx.x;
  const int lid = threadIdx.x;
  const int nthreads = blockDim.x * gridDim.x;
  const int loops = NEDGES / nthreads;
  __shared__ u32 nonces[PROOFSIZE];

  if (lid < PROOFSIZE) nonces[lid] = 0;
  __syncthreads();
  for (int i = 0; i < loops; i++) {
	u64 nonce = gid * loops + i;
	u64 u = dipnode(sipkeys, nonce, 0);
	u64 v = dipnode(sipkeys, nonce, 1);
	for (int i = 0; i < PROOFSIZE; i++) {
	  if (recoveredges[i].x == u && recoveredges[i].y == v)
		nonces[i] = nonce;
	}
  }
  __syncthreads();
  if (lid < PROOFSIZE) {
	if (nonces[lid] > 0)
	  indexes[lid] = nonces[lid];
  }
}


__device__ __forceinline__  void Increase2bCounter(u32 *ecounters, const int bucket) {
  int word = bucket >> 5;
  unsigned char bit = bucket & 0x1F;
  u32 mask = 1 << bit;

  u32 old = atomicOr(ecounters + word, mask) & mask;
  if (old)
    atomicOr(ecounters + word + NZ/32, mask);
}

__device__ __forceinline__  bool Read2bCounter(u32 *ecounters, const int bucket) {
  int word = bucket >> 5;
  unsigned char bit = bucket & 0x1F;
  u32 mask = 1 << bit;

  return (ecounters[word + NZ/32] & mask) != 0;
}

    __constant__ uint2 e0 = {0,0};

    __device__ __forceinline__ ulonglong4 Pack8(const u32 e0, const u32 e1, const u32 e2, const u32 e3, const u32 e4, const u32 e5, const u32 e6, const u32 e7) {
        return make_ulonglong4((u64)e0<<32|e1, (u64)e2<<32|e3, (u64)e4<<32|e5, (u64)e6<<32|e7);
    }

    __device__ bool null(u32 nonce) {
        return nonce == 0;
    }

    __device__ bool null(uint2 nodes) {
        return nodes.x == 0 && nodes.y == 0;
    }
    template<int maxOut, typename EdgeOut>
        __global__ void SeedA(const siphash_keys &sipkeys, ulonglong4 * __restrict__ buffer, int * __restrict__ indexes) {
            const int group = blockIdx.x;
            const int dim = blockDim.x;
            const int lid = threadIdx.x;
            const int gid = group * dim + lid;
            const int nthreads = gridDim.x * dim;
            const int FLUSHA2 = 2*FLUSHA;

            __shared__ EdgeOut tmp[NX][FLUSHA2]; // needs to be ulonglong4 aligned
            const int TMPPERLL4 = sizeof(ulonglong4) / sizeof(EdgeOut);
            __shared__ int counters[NX];

            for (int row = lid; row < NX; row += dim)
                counters[row] = 0;
            __syncthreads();

            const int col = group % NX;
            const int loops = NEDGES / nthreads;
            for (int i = 0; i < loops; i++) {
                u32 nonce = gid * loops + i;
                u32 node1, node0 = dipnode(sipkeys, (u64)nonce, 0);
                if (sizeof(EdgeOut) == sizeof(uint2))
                    node1 = dipnode(sipkeys, (u64)nonce, 1);
                int row = node0 & XMASK;
                int counter = min((int)atomicAdd(counters + row, 1), (int)(FLUSHA2-1));
                tmp[row][counter] = make_Edge(nonce, tmp[0][0], node0, node1);
                __syncthreads();
                if (counter == FLUSHA-1) {
                    int localIdx = min(FLUSHA2, counters[row]);
                    int newCount = localIdx % FLUSHA;
                    int nflush = localIdx - newCount;
                    int cnt = min((int)atomicAdd(indexes + row * NX + col, nflush), (int)(maxOut - nflush));
                    for (int i = 0; i < nflush; i += TMPPERLL4)
                        buffer[((u64)(row * NX + col) * maxOut + cnt + i) / TMPPERLL4] = *(ulonglong4 *)(&tmp[row][i]);
                    for (int t = 0; t < newCount; t++) {
                        tmp[row][t] = tmp[row][t + nflush];
                    }
                    counters[row] = newCount;
                }
                __syncthreads();
            }
            EdgeOut zero = make_Edge(0, tmp[0][0], 0, 0);
            for (int row = lid; row < NX; row += dim) {
                int localIdx = min(FLUSHA2, counters[row]);
                for (int j = localIdx; j % TMPPERLL4; j++)
                    tmp[row][j] = zero;
                for (int i = 0; i < localIdx; i += TMPPERLL4) {
                    int cnt = min((int)atomicAdd(indexes + row * NX + col, TMPPERLL4), (int)(maxOut - TMPPERLL4));
                    buffer[((u64)(row * NX + col) * maxOut + cnt) / TMPPERLL4] = *(ulonglong4 *)(&tmp[row][i]);
                }
            }
        }
    template<int maxIn, typename EdgeIn, int maxOut, typename EdgeOut>
        __global__ void Round(const int round, const siphash_keys &sipkeys, const EdgeIn * __restrict__ source, EdgeOut * __restrict__ destination, const int * __restrict__ sourceIndexes, int * __restrict__ destinationIndexes) {
            const int group = blockIdx.x;
            const int dim = blockDim.x;
            const int lid = threadIdx.x;
            const static int COUNTERWORDS = NZ / 16; // 16 2-bit counters per 32-bit word

            __shared__ u32 ecounters[COUNTERWORDS];

            for (int i = lid; i < COUNTERWORDS; i += dim)
                ecounters[i] = 0;
            __syncthreads();
            const int edgesInBucket = min(sourceIndexes[group], maxIn);
            const int loops = (edgesInBucket + dim-1) / dim;

            for (int loop = 0; loop < loops; loop++) {
                const int lindex = loop * dim + lid;
                if (lindex < edgesInBucket) {
                    const int index = maxIn * group + lindex;
                    EdgeIn edge = __ldg(&source[index]);
                    if (null(edge)) continue;
                    u32 node = endpoint(sipkeys, edge, round&1);
                    Increase2bCounter(ecounters, node >> (2*XBITS));
                }
            }
            __syncthreads();
            for (int loop = 0; loop < loops; loop++) {
                const int lindex = loop * dim + lid;
                if (lindex < edgesInBucket) {
                    const int index = maxIn * group + lindex;
                    EdgeIn edge = __ldg(&source[index]);
                    if (null(edge)) continue;
                    u32 node0 = endpoint(sipkeys, edge, round&1);
                    if (Read2bCounter(ecounters, node0 >> (2*XBITS))) {
                        u32 node1 = endpoint(sipkeys, edge, (round&1)^1);
                        const int bucket = node1 & X2MASK;
                        const int bktIdx = min(atomicAdd(destinationIndexes + bucket, 1), maxOut - 1);
                        destination[bucket * maxOut + bktIdx] = (round&1) ? make_Edge(edge, *destination, node1, node0)
                            : make_Edge(edge, *destination, node0, node1);
                    }
                }
            }
            // if (group==0&&lid==0) printf("round %d cnt(0,0) %d\n", round, sourceIndexes[0]);
        }
    template<int maxOut, typename EdgeOut>
        __global__ void SeedB(const siphash_keys &sipkeys, const EdgeOut * __restrict__ source, ulonglong4 * __restrict__ destination, const int * __restrict__ sourceIndexes, int * __restrict__ destinationIndexes) {
            const int group = blockIdx.x;
            const int dim = blockDim.x;
            const int lid = threadIdx.x;
            const int FLUSHB2 = 2 * FLUSHB;

            __shared__ EdgeOut tmp[NX][FLUSHB2];
            const int TMPPERLL4 = sizeof(ulonglong4) / sizeof(EdgeOut);
            __shared__ int counters[NX];

            // if (group>=0&&lid==0) printf("group  %d  -\n", group);
            for (int col = lid; col < NX; col += dim)
                counters[col] = 0;
            __syncthreads();
            const int row = group / NX;
            const int bucketEdges = min((int)sourceIndexes[group], (int)maxOut);
            const int loops = (bucketEdges + dim-1) / dim;
            for (int loop = 0; loop < loops; loop++) {
                int col; int counter = 0;
                const int edgeIndex = loop * dim + lid;
                if (edgeIndex < bucketEdges) {
                    const int index = group * maxOut + edgeIndex;
                    EdgeOut edge = __ldg(&source[index]);
                    if (null(edge)) continue;
                    u32 node1 = endpoint(sipkeys, edge, 0);
                    col = (node1 >> XBITS) & XMASK;
                    counter = min((int)atomicAdd(counters + col, 1), (int)(FLUSHB2-1));
                    tmp[col][counter] = edge;
                }
                __syncthreads();
                if (counter == FLUSHB-1) {
                    int localIdx = min(FLUSHB2, counters[col]);
                    int newCount = localIdx % FLUSHB;
                    int nflush = localIdx - newCount;
                    int cnt = min((int)atomicAdd(destinationIndexes + row * NX + col, nflush), (int)(maxOut - nflush));
                    for (int i = 0; i < nflush; i += TMPPERLL4)
                        destination[((u64)(row * NX + col) * maxOut + cnt + i) / TMPPERLL4] = *(ulonglong4 *)(&tmp[col][i]);
                    for (int t = 0; t < newCount; t++) {
                        tmp[col][t] = tmp[col][t + nflush];
                    }
                    counters[col] = newCount;
                }
                __syncthreads();
            }
            EdgeOut zero = make_Edge(0, tmp[0][0], 0, 0);
            for (int col = lid; col < NX; col += dim) {
                int localIdx = min(FLUSHB2, counters[col]);
                for (int j = localIdx; j % TMPPERLL4; j++)
                    tmp[col][j] = zero;
                for (int i = 0; i < localIdx; i += TMPPERLL4) {
                    int cnt = min((int)atomicAdd(destinationIndexes + row * NX + col, TMPPERLL4), (int)(maxOut - TMPPERLL4));
                    destination[((u64)(row * NX + col) * maxOut + cnt) / TMPPERLL4] = *(ulonglong4 *)(&tmp[col][i]);
                }
            }
        }

	template<int maxIn>
		__global__ void Tail(const uint2 *source, uint2 *destination, const int *sourceIndexes, int *destinationIndexes) {
	  const int lid = threadIdx.x;
	  const int group = blockIdx.x;
	  const int dim = blockDim.x;
	  int myEdges = sourceIndexes[group];
	  __shared__ int destIdx;

	  if (lid == 0)
		destIdx = atomicAdd(destinationIndexes, myEdges);
	  __syncthreads();
	  for (int i = lid; i < myEdges; i += dim)
		destination[destIdx + lid] = source[group * maxIn + lid];
	}

    __device__ u32 endpoint(const siphash_keys &sipkeys, u32 nonce, int uorv) {
        return dipnode(sipkeys, nonce, uorv);
    }

    __device__ u32 endpoint(const siphash_keys &sipkeys, uint2 nodes, int uorv) {
        return uorv ? nodes.y : nodes.x;
    }

    __device__ uint2 make_Edge(const u32 nonce, const uint2 dummy, const u32 node0, const u32 node1) {
        return make_uint2(node0, node1);
    }

    __device__ uint2 make_Edge(const uint2 edge, const uint2 dummy, const u32 node0, const u32 node1) {
        return edge;
    }

    __device__ u32 make_Edge(const u32 nonce, const u32 dummy, const u32 node0, const u32 node1) {
        return nonce;
    }

    edgetrimmer::edgetrimmer(const trimparams _tp) {
        indexesSize = NX * NY * sizeof(u32);
        tp = _tp;
        checkCudaErrors(hipMalloc((void**)&dt, sizeof(edgetrimmer)));
        checkCudaErrors(hipMalloc((void**)&uvnodes, PROOFSIZE * 2 * sizeof(u32)));
        checkCudaErrors(hipMalloc((void**)&dipkeys, sizeof(siphash_keys)));
        checkCudaErrors(hipMalloc((void**)&indexesE, indexesSize));
        checkCudaErrors(hipMalloc((void**)&indexesE2, indexesSize));
        sizeA = ROW_EDGES_A * NX * (tp.expand > 0 ? sizeof(u32) : sizeof(uint2));
        sizeB = ROW_EDGES_B * NX * (tp.expand > 1 ? sizeof(u32) : sizeof(uint2));
        const size_t bufferSize = sizeA + sizeB;
        printf("bufferSize: %lu\n", bufferSize);
        checkCudaErrors(hipMalloc((void**)&bufferA, bufferSize));
        bufferB  = bufferA + sizeA / sizeof(ulonglong4);
        bufferAB = bufferA + sizeB / sizeof(ulonglong4);
    }
    u64 edgetrimmer::globalbytes() const {
        return (sizeA+sizeB) + 2 * indexesSize + sizeof(siphash_keys) + PROOFSIZE * 2 * sizeof(u32) + sizeof(edgetrimmer);
    }
    edgetrimmer::~edgetrimmer() {
        hipFree(bufferA);
        hipFree(indexesE2);
        hipFree(indexesE);
        hipFree(dipkeys);
        hipFree(uvnodes);
        hipFree(dt);
        hipDeviceReset();
    }
    u32 edgetrimmer::trim(uint32_t device) {
        hipSetDevice(device);
        hipMemcpy(dt, this, sizeof(edgetrimmer), hipMemcpyHostToDevice);
        hipEvent_t start, stop;
        // hipEvent_t startall, stopall;
        // checkCudaErrors(hipEventCreate(&startall)); checkCudaErrors(hipEventCreate(&stopall));
        checkCudaErrors(hipEventCreate(&start)); checkCudaErrors(hipEventCreate(&stop));

        hipMemset(indexesE, 0, indexesSize);
        hipMemset(indexesE2, 0, indexesSize);
        hipMemcpy(dipkeys, &sipkeys, sizeof(sipkeys), hipMemcpyHostToDevice);

        hipDeviceSynchronize();
        float durationA, durationB;
        hipEventRecord(start, NULL);

		fprintf(stderr, "Tp block %zu and tpb %zu", tp.genA.blocks, tp.genA.tpb);
        if (tp.expand == 0)
            SeedA<EDGES_A, uint2><<<tp.genA.blocks, tp.genA.tpb>>>(*dipkeys, bufferAB, (int *)indexesE);
        else
            SeedA<EDGES_A,   u32><<<tp.genA.blocks, tp.genA.tpb>>>(*dipkeys, bufferAB, (int *)indexesE);

        checkCudaErrors(hipDeviceSynchronize());
        hipEventRecord(stop, NULL);
        hipEventSynchronize(stop); hipEventElapsedTime(&durationA, start, stop); hipEventRecord(start, NULL);

        const u32 halfA = sizeA/2 / sizeof(ulonglong4);
        const u32 halfE = NX2 / 2;
        if (tp.expand == 0) {
            SeedB<EDGES_A, uint2><<<tp.genB.blocks/2, tp.genB.tpb>>>(*dipkeys, (const uint2 *)bufferAB, bufferA, (const int *)indexesE, indexesE2);
            SeedB<EDGES_A, uint2><<<tp.genB.blocks/2, tp.genB.tpb>>>(*dipkeys, (const uint2 *)(bufferAB+halfA), bufferA+halfA, (const int *)(indexesE+halfE), indexesE2+halfE);
        } else {
            SeedB<EDGES_A,   u32><<<tp.genB.blocks/2, tp.genB.tpb>>>(*dipkeys, (const   u32 *)bufferAB, bufferA, (const int *)indexesE, indexesE2);
            SeedB<EDGES_A,   u32><<<tp.genB.blocks/2, tp.genB.tpb>>>(*dipkeys, (const   u32 *)(bufferAB+halfA), bufferA+halfA, (const int *)(indexesE+halfE), indexesE2+halfE);
        }

        checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
        hipEventSynchronize(stop); hipEventElapsedTime(&durationB, start, stop);
        // printf("Seeding completed in %.0f + %.0f ms\n", durationA, durationB);

        hipMemset(indexesE, 0, indexesSize);

        if (tp.expand == 0)
            Round<EDGES_A, uint2, EDGES_B, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(0, *dipkeys, (const uint2 *)bufferA, (uint2 *)bufferB, (const int *)indexesE2, (int *)indexesE); // to .632
        else if (tp.expand == 1)
            Round<EDGES_A,   u32, EDGES_B, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(0, *dipkeys, (const   u32 *)bufferA, (uint2 *)bufferB, (const int *)indexesE2, (int *)indexesE); // to .632
        else // tp.expand == 2
            Round<EDGES_A,   u32, EDGES_B,   u32><<<tp.trim.blocks, tp.trim.tpb>>>(0, *dipkeys, (const   u32 *)bufferA, (  u32 *)bufferB, (const int *)indexesE2, (int *)indexesE); // to .632

        hipMemset(indexesE2, 0, indexesSize);

        if (tp.expand < 2)
            Round<EDGES_B, uint2, EDGES_B/2, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(1, *dipkeys, (const uint2 *)bufferB, (uint2 *)bufferA, (const int *)indexesE, (int *)indexesE2); // to .296
        else
            Round<EDGES_B,   u32, EDGES_B/2, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(1, *dipkeys, (const   u32 *)bufferB, (uint2 *)bufferA, (const int *)indexesE, (int *)indexesE2); // to .296

        hipMemset(indexesE, 0, indexesSize);
        Round<EDGES_B/2, uint2, EDGES_A/4, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(2, *dipkeys, (const uint2 *)bufferA, (uint2 *)bufferB, (const int *)indexesE2, (int *)indexesE); // to .176
        hipMemset(indexesE2, 0, indexesSize);
        Round<EDGES_A/4, uint2, EDGES_B/4, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(3, *dipkeys, (const uint2 *)bufferB, (uint2 *)bufferA, (const int *)indexesE, (int *)indexesE2); // to .117

        hipDeviceSynchronize();

        for (int round = 4; round < tp.ntrims; round += 2) {
            hipMemset(indexesE, 0, indexesSize);
            Round<EDGES_B/4, uint2, EDGES_B/4, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(round, *dipkeys,  (const uint2 *)bufferA, (uint2 *)bufferB, (const int *)indexesE2, (int *)indexesE);
            hipMemset(indexesE2, 0, indexesSize);
            Round<EDGES_B/4, uint2, EDGES_B/4, uint2><<<tp.trim.blocks, tp.trim.tpb>>>(round+1, *dipkeys,  (const uint2 *)bufferB, (uint2 *)bufferA, (const int *)indexesE, (int *)indexesE2);
        }

        hipMemset(indexesE, 0, indexesSize);
        hipDeviceSynchronize();

        Tail<EDGES_B/4><<<tp.tail.blocks, tp.tail.tpb>>>((const uint2 *)bufferA, (uint2 *)bufferB, (const int *)indexesE2, (int *)indexesE);
        hipMemcpy(hostA, indexesE, NX * NY * sizeof(u32), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
		fprintf(stderr, "Host A [0]: %zu\n", hostA[0]);
        return hostA[0];
    }


};
