#include "hip/hip_runtime.h"
// Cuckoo Cycle, a memory-hard proof-of-work by John Tromp
// Copyright (c) 2018 Jiri Vadura (photon) and John Tromp
// This software is covered by the FAIR MINING license
#include <stdio.h>
#include <string.h>
#include <vector>
#include <assert.h>
#include <vector>
// #include <algorithm>
#include <stdint.h>
#include <sys/time.h> // gettimeofday
#include <unistd.h>
#include <sys/types.h>
#include "trimmer.h"
#include "../../miner.h"
namespace cuckoogpu { 

class cuckoo_hash {
public:
  u64 *cuckoo;

  cuckoo_hash() {
    cuckoo = new u64[CUCKOO_SIZE];
	memset(cuckoo, 0, CUCKOO_SIZE * sizeof(u64));
  }
  ~cuckoo_hash() {
    delete[] cuckoo;
  }
  void set(node_t u, node_t v) {
    u64 niew = (u64)u << NODEBITS | v;
    for (node_t ui = (u >> IDXSHIFT) & CUCKOO_MASK; ; ui = (ui+1) & CUCKOO_MASK) {
      u64 old = cuckoo[ui];
      if (old == 0 || (old >> NODEBITS) == (u & KEYMASK)) {
        cuckoo[ui] = niew;
        return;
      }
    }
  }
  node_t operator[](node_t u) const {
    for (node_t ui = (u >> IDXSHIFT) & CUCKOO_MASK; ; ui = (ui+1) & CUCKOO_MASK) {
      u64 cu = cuckoo[ui];
      if (!cu)
        return 0;
      if ((cu >> NODEBITS) == (u & KEYMASK)) {
        assert(((ui - (u >> IDXSHIFT)) & CUCKOO_MASK) < MAXDRIFT);
        return (node_t)(cu & NODEMASK);
      }
    }

  }
};

const static u32 MAXPATHLEN = 8 << ((NODEBITS+2)/3);

int nonce_cmp(const void *a, const void *b) {
  return *(u32 *)a - *(u32 *)b;
}

struct solver_ctx {
  edgetrimmer *trimmer;
  uint2 *edges;
  cuckoo_hash *cuckoo;
  uint2 soledges[PROOFSIZE];
  std::vector<u32> sols; // concatenation of all proof's indices
  u32 us[MAXPATHLEN];
  u32 vs[MAXPATHLEN];
  uint32_t device;

  solver_ctx(const trimparams tp, uint32_t _device = 0) {
    trimmer = new edgetrimmer(tp);
    edges   = new uint2[MAXEDGES];
    cuckoo  = new cuckoo_hash();
    device = _device;
  }

  void setheadernonce(char * const header,  const uint64_t nonce) {
    uint64_t littleEndianNonce = htole64(nonce);
    char headerBuf[40];
    memcpy(headerBuf, header, 32);
    memcpy(headerBuf + 32, static_cast<uint64_t*>(&littleEndianNonce), sizeof(nonce));
    setheader(headerBuf, 40, &trimmer->sipkeys);
    sols.clear();
  }

  ~solver_ctx() {
    delete cuckoo;
    delete[] edges;
    delete trimmer;
  }

  void recordedge(const u32 i, const u32 u2, const u32 v2) {
    soledges[i].x = u2/2;
    soledges[i].y = v2/2;
  }
//opencl
  void solution(const u32 *us, u32 nu, const u32 *vs, u32 nv) {
    u32 ni = 0;
    recordedge(ni++, *us, *vs);
    while (nu--)
      recordedge(ni++, us[(nu+1)&~1], us[nu|1]); // u's in even position; v's in odd
    while (nv--)
      recordedge(ni++, vs[nv|1], vs[(nv+1)&~1]); // u's in odd position; v's in even
    	assert(ni == PROOFSIZE);
    	sols.resize(sols.size() + PROOFSIZE);
	for(int i = 0; i < PROOFSIZE; i++){
		printf("<%u, %u>, ", soledges[i].x, soledges[i].y);
	}
	printf("\n");
    	hipMemset(trimmer->indexesE2, 0, trimmer->indexesSize);
    	hipMemcpy(trimmer->recoveredges, soledges, sizeof(soledges), hipMemcpyHostToDevice);

	Recovery<<<trimmer->tp.recover.blocks, trimmer->tp.recover.tpb>>>(trimmer->dipkeys, trimmer->bufferA, (int *)trimmer->indexesE2, trimmer->recoveredges);
    	hipMemcpy(&sols[sols.size()-PROOFSIZE], trimmer->indexesE2, PROOFSIZE * sizeof(u32), hipMemcpyDeviceToHost);
    	checkCudaErrors(hipDeviceSynchronize());
	fprintf(stderr, "Index: %d points: [", sols.size() / PROOFSIZE);
	for (uint32_t idx = 0; idx < PROOFSIZE; idx++) {
		fprintf(stderr, "<%zu,%zu>, ", soledges[idx].x, soledges[idx].y);
	}
	fprintf(stderr, "] solutions: [");
	for (uint32_t idx = 0; idx < PROOFSIZE; idx++) {
		fprintf(stderr, "%zu,", sols[sols.size() - PROOFSIZE + idx]);
	}
	fprintf(stderr, "]\n");
    	qsort(&sols[sols.size()-PROOFSIZE], PROOFSIZE, sizeof(u32), nonce_cmp);
  }

  u32 path(u32 u, u32 *us) {
    u32 nu, u0 = u;
	/* fprintf(stderr, "start %zu\n", u0); */
    for (nu = 0; u; u = (*cuckoo)[u]) {
      if (nu >= MAXPATHLEN) {
			/* fprintf(stderr, "nu: %zu, u: %zu, Maxpathlen: %zu\n", nu, u, MAXPATHLEN); */
        while (nu-- && us[nu] != u) ;
        if (~nu) {
          printf("illegal %4d-cycle from node %d\n", MAXPATHLEN-nu, u0);
          exit(0);
        }
        printf("maximum path length exceeded\n");
        return 0; // happens once in a million runs or so; signal trouble
      }
      us[nu++] = u;
    }
	/* fprintf(stderr, "path nu: %zu\n", nu); */
    return nu;
  }

  void addedge(uint2 edge) {
    const u32 u0 = edge.x << 1, v0 = (edge.y << 1) | 1;

    if (u0) {
      u32 nu = path(u0, us), nv = path(v0, vs);
      if (!nu-- || !nv--)
        return; // drop edge causing trouble

      if (us[nu] == vs[nv]) {
        const u32 min = nu < nv ? nu : nv;
        for (nu -= min, nv -= min; us[nu] != vs[nv]; nu++, nv++) ;
        const u32 len = nu + nv + 1;
        if (len == PROOFSIZE)
          solution(us, nu, vs, nv);
      } else if (nu < nv) {
        while (nu--)
          cuckoo->set(us[nu+1], us[nu]);
        cuckoo->set(u0, v0);
      } else {
        while (nv--)
          cuckoo->set(vs[nv+1], vs[nv]);
		cuckoo->set(v0, u0);
      }

    }
  }

  void findcycles(uint2 *edges, u32 nedges) {
    memset(cuckoo->cuckoo, 0, CUCKOO_SIZE * sizeof(u64));
    for (u32 i = 0; i < nedges; i++) {
      addedge(edges[i]);
	}
  }
  int solve() {
    u32 nedges = trimmer->trim(this->device);
    if (nedges > MAXEDGES) {
      fprintf(stderr, "OOPS; losing %d edges beyond MAXEDGES=%d\n", nedges-MAXEDGES, MAXEDGES);
      nedges = MAXEDGES;
    }

	nedges = nedges & CUCKOO_MASK;
    hipMemcpy(edges, trimmer->bufferB, nedges * 8, hipMemcpyDeviceToHost);
    findcycles(edges, nedges);
    return sols.size() / PROOFSIZE;
  }
};

}; // end of namespace cuckoogpu

cuckoogpu::solver_ctx* ctx = NULL;
int32_t CuckooFindSolutionsCuda(
        uint8_t *header,
        uint64_t nonce,
        result_t *result,
        uint32_t resultBuffSize,
        uint32_t *solLength,
        uint32_t *numSol)
{
    using namespace cuckoogpu;
    using std::vector;
    hipSetDevice(ctx->device);

    uint8_t tmpheader[32] = {3, 181, 241, 90, 114, 14, 82, 48, 238, 210, 214, 200, 40, 238, 92, 242, 246, 224, 171, 116, 220, 131, 19, 117, 176, 2, 253, 46, 114, 109, 164, 25};//{66, 178, 108, 246, 24, 92, 120, 111, 149, 32, 165, 229, 20, 16, 27, 216, 10, 250, 135, 182, 10, 198, 128, 20, 64, 141, 55, 205, 161, 38, 209, 177};
    nonce = 5882121833590555395;
	header = tmpheader;

    ctx->setheadernonce((char*)header, nonce); //TODO(tian)
    char headerInHex[65];
    for (uint32_t i = 0; i < 32; i++) {
        sprintf(headerInHex + 2 * i, "%02x", *((unsigned int8_t*)(header + i)));
    }
    headerInHex[64] = '\0';

    u32 nsols = ctx->solve();
    vector<vector<u32> > sols;
    vector<vector<u32> >* solutions = &sols;
    for (unsigned s = 0; s < nsols; s++) {
        u32* prf = &(ctx->sols[s * PROOFSIZE]);
        solutions->push_back(vector<u32>());
        vector<u32>& sol = solutions->back();
        for (uint32_t idx = 0; idx < PROOFSIZE; idx++) {
            sol.push_back(prf[idx]);
        }
    }
    *solLength = 0;
    *numSol = sols.size();
    if (sols.size() == 0)
        return 0;
    *solLength = uint32_t(sols[0].size());
    for (size_t n = 0; n < min(sols.size(), (size_t)resultBuffSize / (*solLength)); n++)
    {
        vector<u32>& sol = sols[n];
        for (size_t i = 0; i < sol.size(); i++) {
            result[i + n * (*solLength)] = sol[i];
        }
    }
    return nsols > 0;

}
void CuckooInitialize(uint32_t device) {
    printf("thread: %d\n", getpid());
    using namespace cuckoogpu;
    using std::vector;

    trimparams tp;
    int nDevices = 0;
    device = 0;
    //TODO(tian) make use of multiple gpu
    checkCudaErrors(hipGetDeviceCount(&nDevices));
    printf("ndevices = %d, device = %d\n", nDevices, device);
    assert(device < nDevices);
    hipSetDevice(device);
    hipDeviceProp_t prop;
    checkCudaErrors(hipGetDeviceProperties(&prop, device));
    assert(tp.genA.tpb <= prop.maxThreadsPerBlock);
    assert(tp.genB.tpb <= prop.maxThreadsPerBlock);
    assert(tp.trim.tpb <= prop.maxThreadsPerBlock);
    // assert(tp.tailblocks <= prop.threadDims[0]);
    assert(tp.tail.tpb <= prop.maxThreadsPerBlock);
    assert(tp.recover.tpb <= prop.maxThreadsPerBlock);
    ctx = new solver_ctx(tp, device);
    printf("50%% edges, %d*%d buckets, %d trims, and %d thread blocks.\n", NX, NY, tp.ntrims, NX);
    u64 bytes = ctx->trimmer->globalbytes();
    int unit;
    for (unit=0; bytes >= 10240; bytes>>=10,unit++);
    printf("Using %d%cB of global memory.\n", (u32)bytes, " KMGT"[unit]);
}
