#include "hip/hip_runtime.h"
#include "cuda_ops.h"
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <memory>
#include <string.h>
#include <iostream>
#include <string>
#include "nms.h"

// #define CVM_PRINT_CUDA_RESULT

void print_to_file(const int32_t *y, int32_t n, std::string filename){
#ifdef CVM_PRINT_CUDA_RESULT
  int32_t *y_data = new int32_t[n];
  hipMemcpy(y_data, y, sizeof(int32_t)*n, hipMemcpyDeviceToHost);

  FILE *fp = fopen(filename.c_str(), "a+");

  int32_t min = y_data[0], max= y_data[0];
  for(uint64_t i = 0; i < n; i++){
    min = min > y_data[i] ? y_data[i] : min;
    max = max < y_data[i] ? y_data[i] : max;
  }
  fprintf(fp, "%d %d\n", min, max);
  for(uint64_t i = 0; i < 1000 && i < n; i++){
    fprintf(fp, "%d ", y_data[i]);
  }
  fprintf(fp, "\n");
  fclose(fp);
  delete y_data;
#endif
}
inline int32_t getGridSize(const int64_t n, const int32_t blockSize){
  int64_t tg = (n + blockSize - 1) / blockSize;
  return tg > 4096 ? 4096 : tg;
}
inline int32_t getShareMemorySize(const int32_t device_id){
  static int32_t sharedMemPerBlock = 0;
  if(sharedMemPerBlock == 0){
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id);
    sharedMemPerBlock = prop.sharedMemPerBlock;
  }
  return sharedMemPerBlock;
}
const char* check_cuda_error(hipError_t error){
  if(error == hipSuccess) return NULL;
  else return hipGetErrorString(error);
}

__global__ void kernel_elemwise_add(int32_t *a, int32_t *b, int32_t *c, uint64_t n){
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(uint64_t i = tid; i < n; i += gridDim.x * blockDim.x){
    c[i] = a[i] + b[i];
  }
}

const char* cuda_elemwise_add(int32_t *a, int32_t *b, int32_t *c, uint64_t n, bool debug){
  int32_t *dev_a = a, *dev_b = b, *dev_c = c;
  size_t size = sizeof(int32_t) * n;
  if(debug){
    check_cuda_error(hipMalloc((void**)&dev_a, size));
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
  }
  int blockSize = 256;
  int gridSize = getGridSize(n, blockSize);//(n + blockSize - 1) / blockSize;
  kernel_elemwise_add<<<gridSize, blockSize>>>(dev_a, dev_b, dev_c, n);
  //    hipDeviceSynchronize();
  if(debug){
    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
  }
  return check_cuda_error(hipGetLastError());
}
__global__ void kernel_elemwise_sub(int32_t *a, int32_t *b, int32_t *c, uint64_t n){
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(uint64_t i = tid; i < n; i += gridDim.x*blockDim.x){
    c[i] = a[i] - b[i];
  }
}

const char* cuda_elemwise_sub(int32_t *a, int32_t *b, int32_t *c, uint64_t n){
  int blockSize = 256;
  int gridSize = getGridSize(n, blockSize);
  kernel_elemwise_sub<<<gridSize, blockSize>>>(a, b, c, n);
  return check_cuda_error(hipGetLastError());
}

#define BS 16
#define FS 8
//template<int F_H, int F_W, int STRIDE>
__global__ void kernel_conv2d(
    const int32_t * __restrict__ input, const int32_t i_n, const int32_t i_c, const int32_t i_h, const int32_t i_w,
    const int32_t * __restrict__ filter, const int32_t f_n, const int32_t f_c, const int32_t f_h, const int32_t f_w,
    const int32_t * __restrict__ bias,
    const int32_t padding_h, const int32_t padding_w,
    const int32_t stride_h, const int32_t stride_w,
    const int32_t dilation_h, const int32_t dilation_w,
    const int32_t groups,
    int32_t *output, const int32_t o_n, const int32_t o_c, const int32_t o_h, const int32_t o_w){
  //    int g_y = blockDim.y * blockIdx.y + threadIdx.y;
  int g_x = blockDim.x * blockIdx.x + threadIdx.x;
  int l_y = threadIdx.y; 
  int l_x = threadIdx.x;
  int tmp_f_h = (f_h - 1) * dilation_h + 1; // for dilation, to be optimized
  int tmp_f_w = (f_w - 1) * dilation_w + 1;
  int tmp_o_h = i_h + 2 * padding_h - tmp_f_h + 1; // for stride
  int tmp_o_w = i_w + 2 * padding_w - tmp_f_w + 1;
  int perBlockOneImageY = (tmp_o_h+BS-1) / BS;
  int perBlockOneImageX = (tmp_o_w+BS-1) / BS;
  int l_o_c = blockIdx.y / perBlockOneImageY;
  int n = l_o_c / ((o_c+FS-1)/FS);
  int nsize = n * i_c * i_h * i_w; 
  int l_f_n = l_o_c % ((o_c+FS-1)/FS);
  int l_o_hi = blockIdx.y % perBlockOneImageY;
  int l_o_wi = blockIdx.x % perBlockOneImageX;
  int l_o_h = l_o_hi * BS + l_y;
  //    int l_o_w = l_o_wi * BS + l_x;

  const int32_t F_H = f_h;
  const int32_t F_W = f_w;
  //    __shared__ int32_t shared_i[BS + F_H - 1][BS + F_W - 1];
  int32_t sih = BS + tmp_f_h - 1;
  int32_t siw = BS + tmp_f_w - 1;
  extern __shared__ int32_t  share[];
  int32_t *shared_i = (int32_t*)share; 
  int32_t *shared_f = &share[sih * siw];
  int32_t *shared_b = &shared_f[F_H*F_W*FS];

  int32_t sum[FS] = {0}; 
  int min_s_y = (l_o_hi+1) * BS <= tmp_o_h ? BS : tmp_o_h%BS;
  int min_s_x = (l_o_wi+1) * BS <= tmp_o_w ? BS : tmp_o_w%BS;

  //load bias to shared memory
  int lid = l_y * BS + l_x;
  for(int i = lid; bias != NULL && i < FS; i+=BS*BS){
    if(l_f_n*FS + i < o_c)
      shared_b[i] = bias[l_f_n*FS + i];
    else shared_b[i] = 0;
  }

  if(l_o_h >= tmp_o_h || g_x >= tmp_o_w) return;

  for(int c = 0; c < i_c; c++){
    //load input to shared
    int l_i_h = l_o_h - padding_h;
    int i_y = c * i_h + l_i_h;
    int i_x = g_x - padding_w;
    if(l_i_h < 0 || i_x < 0 || l_i_h >= i_h || i_x >= i_w)
      shared_i[l_y*siw + l_x] = 0;
    else
      shared_i[l_y*siw + l_x] = input[nsize + i_y * i_w + i_x];

    if(l_y < tmp_f_h-1){
      for(int i = l_y; i < tmp_f_h-1; i+=min_s_y){
        if(l_i_h+min_s_y+i-l_y < 0 || i_x < 0 || l_i_h+min_s_y+i-l_y >= i_h || i_x >= i_w)
          shared_i[(i+min_s_y)*siw + l_x] = 0;
        else
          shared_i[(i + min_s_y)*siw + l_x] = input[nsize + (i_y + min_s_y + i - l_y) * i_w + i_x];     
      }
    }
    if(l_x < tmp_f_w-1){
      for(int i = l_x; i < tmp_f_w-1; i+= min_s_x){
        if(l_i_h < 0 || i_x+min_s_x+i-l_x < 0 || l_i_h >= i_h || i_x+min_s_x+i-l_x >= i_w)
          shared_i[l_y * siw + i+min_s_x] = 0;
        else
          shared_i[l_y * siw + i + min_s_x] = input[nsize + i_y * i_w + i_x + min_s_x + i - l_x];
      }
    }
    if(l_y < tmp_f_h-1 && l_x < tmp_f_w-1){
      for(int i = l_y; i < tmp_f_h-1; i+=min_s_y){
        for(int j = l_x; j < tmp_f_w-1; j+=min_s_x){
          if(l_i_h+min_s_y+i-l_y < 0 || i_x+min_s_x+j-l_x < 0 || l_i_h+min_s_y+i-l_y >= i_h || i_x+min_s_x+j-l_x >= i_w)
            shared_i[(i+min_s_y) * siw + j+min_s_x] = 0;
          else
            shared_i[(i+min_s_y) * siw + j+min_s_x] = input[nsize + (i_y+min_s_y + i-l_y)*i_w + i_x + min_s_x + j - l_x];
        }
      }
    }

    //load filter to shared;
    if(l_y < F_H && l_x < F_W){
      for(int i = l_y; i < F_H; i+= min_s_y)
        for(int j = l_x; j < F_W; j+=min_s_x)
          for(int fc = 0; fc < FS; fc++){
            if(l_f_n * FS + fc < o_c)
              shared_f[fc * F_H*F_W + i*F_W + j] = filter[(l_f_n*FS+fc) * F_H * F_W * f_c + c * F_H * F_W + i * F_W + j];
            else shared_f[fc * F_H * F_W + i * F_W + j] = 0;
          }
    }
    __syncthreads();

    for(int fy = 0; fy < F_H; fy++){
      for(int fx = 0; fx < F_W; fx++){
        int32_t tmpx = shared_i[(l_y+fy*dilation_h)*siw + l_x+fx*dilation_w];
#pragma unroll
        for(int fc = 0; fc < FS; fc++){
          sum[fc] += tmpx * shared_f[fc*F_H*F_W + fy*F_W + fx];
        }
      }
    } 
    __syncthreads();
  }

  if(l_o_h % stride_h == 0 && g_x % stride_w == 0){ //TODO to be optimized
    //int oi = l_o_c * o_h * o_w + l_o_h * o_w + g_x;
    for(int fc = 0; fc < FS; fc++){
      if(l_f_n*FS + fc < o_c){
        int oi = n*o_c*o_h*o_w + (l_f_n*FS+fc) * o_h * o_w + l_o_h/stride_h * o_w + g_x/stride_w;
        output[oi] = sum[fc] + (bias != NULL ? shared_b[fc] : 0);
      }
    }
  }
}

__global__ void kernel_conv2d_no_shared(
    const int32_t * __restrict__ input, const int32_t i_n, const int32_t i_c, const int32_t i_h, const int32_t i_w,
    const int32_t * __restrict__ filter, const int32_t f_n, const int32_t f_c, const int32_t f_h, const int32_t f_w,
    const int32_t * __restrict__ bias,
    const int32_t padding_h, const int32_t padding_w,
    const int32_t stride_h, const int32_t stride_w,
    const int32_t dilation_h, const int32_t dilation_w,
    const int32_t groups,
    int32_t *output, const int32_t o_n, const int32_t o_c, const int32_t o_h, const int32_t o_w){
  int32_t gy = threadIdx.y + blockIdx.y * blockDim.y;
  int32_t gx = threadIdx.x + blockIdx.x * blockDim.x;
  int32_t l_o_h = gy % o_h;
  int32_t l_o_c = gy / o_h % o_c;
  int32_t l_o_n = gy / (o_h * o_c);
  if(gy < o_n * o_c * o_h && gx < o_w){
    int32_t sum = 0;
    for(int ic = 0; ic < i_c; ++ic){
      for(int fy = 0; fy < f_h; ++fy){
        for(int fx = 0; fx < f_w; ++fx){
          int32_t l_i_h = l_o_h * stride_h + fy * dilation_h - padding_h;
          int32_t l_i_w = gx * stride_w + fx * dilation_h - padding_w;
          int32_t x;
          if(l_i_h < 0 || l_i_w < 0 || l_i_h >= i_h || l_i_w >= i_w)
            x = 0;
          else x = input[l_o_n * i_c * i_h * i_w + ic * i_h * i_w + l_i_h * i_w + l_i_w];
          sum += x * filter[l_o_c * i_c * f_h * f_w + ic * f_h * f_w + fy * f_w + fx];
        }
      }
    }
    output[gy * o_w + gx] = sum + (bias != NULL ? bias[l_o_c] : 0);
  }
}
const char* cuda_conv2d(
    int32_t *input, int32_t i_n, int32_t i_c, int32_t i_h, int32_t i_w,
    int32_t *filter, int32_t f_n, int32_t f_c, const int32_t f_h, const int32_t f_w,
    int32_t *bias,
    const int32_t padding_h, const int32_t padding_w,
    const int32_t stride_h, const int32_t stride_w,
    const int32_t dilation_h, const int32_t dilation_w,
    int32_t groups,
    int32_t *output, int32_t o_n, int32_t o_c, int32_t o_h, int32_t o_w, 
    int32_t device_id,
    bool debug){
  if(i_n < 1 || i_c < 1 || i_h < 1 || i_w < 1 || f_n < 1 || f_c < 1 || f_h < 1 || f_w < 1 || 
      padding_h < 0 || padding_w < 0 || stride_h < 1 || stride_w < 1 || dilation_h < 1 || dilation_w < 1 ||
      o_n < 1 || o_c < 1 || o_h < 1 || o_w < 1){
    return "error args";
  }
  int32_t *dev_i = input, *dev_f = filter, *dev_o = output, *dev_b = bias;
  size_t s_i = i_n * i_c * i_h * i_w * sizeof(int32_t);
  size_t s_f = f_n * f_c * f_h * f_w * sizeof(int32_t);
  size_t s_b = o_c * sizeof(int32_t); 
  size_t s_o = o_n * o_c * o_h * o_w * sizeof(int32_t);
  hipEvent_t start, stop;
  if(debug){
    hipMalloc((void**)&dev_i, s_i);
    hipMalloc((void**)&dev_f, s_f);
    hipMalloc((void**)&dev_o, s_o);
    hipMemcpy(dev_i, input, s_i, hipMemcpyHostToDevice);
    hipMemcpy(dev_f, filter, s_f, hipMemcpyHostToDevice);
    if(bias != NULL){
      hipMalloc((void**)&dev_b, s_b);
      hipMemcpy(dev_b, bias, s_b, hipMemcpyHostToDevice);
    }
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
  }

  int tmp_f_h = (f_h - 1) * dilation_h + 1; // for dilation, to be optimized
  int tmp_f_w = (f_w - 1) * dilation_w + 1;
  int tmp_o_h = i_h + 2 * padding_h - tmp_f_h + 1; //for stride > 1 , TODO to be optimized
  int tmp_o_w = i_w + 2 * padding_w - tmp_f_w + 1;
  int32_t totalShareMemSize = getShareMemorySize(device_id);
  size_t share_size = ((BS + tmp_f_h - 1) * (BS + tmp_f_w - 1) + f_h * f_w * FS + FS) * sizeof(int32_t);
  if(share_size < totalShareMemSize){
    int b_h = BS;
    int b_w = BS;
    int32_t g_h = o_n * ((o_c + FS - 1) / FS) * ((tmp_o_h + b_h - 1) / b_h);
    int32_t g_w = (tmp_o_w + b_w - 1) / b_w;
    dim3 bDim(b_w, b_h, 1);
    dim3 gDim(g_w, g_h, 1);
    kernel_conv2d<<<gDim, bDim, share_size>>>(
        dev_i, i_n, i_c, i_h, i_w,
        dev_f, f_n, f_c, f_h, f_w,
        dev_b, 
        padding_h, padding_w,
        stride_h, stride_w,
        dilation_h, dilation_w,
        groups,
        dev_o, o_n, o_c, o_h, o_w);
  }else{
    int b_h = BS;
    int b_w = BS;
    int g_h = o_n * o_c * ((o_h + b_h - 1) / b_h);
    int g_w = (o_w + b_w - 1) / b_w;
    dim3 bDim(b_w, b_h, 1);
    dim3 gDim(g_w, g_h, 1);
    kernel_conv2d_no_shared<<<gDim, bDim>>>(
        dev_i, i_n, i_c, i_h, i_w,
        dev_f, f_n, f_c, f_h, f_w,
        dev_b, 
        padding_h, padding_w,
        stride_h, stride_w,
        dilation_h, dilation_w,
        groups,
        dev_o, o_n, o_c, o_h, o_w);
  }
  //    hipDeviceSynchronize();
  if(debug){
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    double ops = (double)((double)1.0*i_n * o_c * o_h * o_w * f_h * f_w * f_c * 3.0);
    printf("gpu cal time:%.4f, %f, %.4f\n", milliseconds, ops, ops / (milliseconds / 1000.0) / 1024.0/1024.0/1024.0);
    hipMemcpy(output, dev_o, s_o, hipMemcpyDeviceToHost);
    hipFree(dev_i);
    hipFree(dev_f);
    hipFree(dev_o);
    if(bias != NULL)
      hipFree(dev_b);
  }
  print_to_file(output, o_c * o_h * o_w, "/tmp/zkh/cuda_conv2d.txt");
  return check_cuda_error(hipGetLastError());
}
__global__ void kernel_depthwise_conv2d(
    const int32_t * __restrict__ input, int32_t i_n, int32_t i_c, int32_t i_h, int32_t i_w,
    const int32_t * __restrict__ filter, int32_t f_n, int32_t f_c, int32_t f_h, int32_t f_w,
    const int32_t * __restrict__ bias,
    int32_t padding_h, int32_t padding_w,
    int32_t stride_h, int32_t stride_w,
    int32_t dilation_h, int32_t dilation_w, 
    int32_t groups,
    int32_t *output, int32_t o_n, int32_t o_c, int32_t o_h, int32_t o_w)
{
  //    int g_y = blockDim.y * blockIdx.y + threadIdx.y;
  int g_x = blockDim.x * blockIdx.x + threadIdx.x;
  int l_y = threadIdx.y; 
  int l_x = threadIdx.x;
  int tmp_f_h = (f_h - 1) * dilation_h + 1; // for dilation, to be optimized
  int tmp_f_w = (f_w - 1) * dilation_w + 1;
  int tmp_o_h = i_h + 2 * padding_h - tmp_f_h + 1; // for stride
  int tmp_o_w = i_w + 2 * padding_w - tmp_f_w + 1;
  int perBlockOneImageY = (tmp_o_h+BS-1) / BS;
  int perBlockOneImageX = (tmp_o_w+BS-1) / BS;
  int l_o_c = blockIdx.y / perBlockOneImageY;
  int l_o_hi = blockIdx.y % perBlockOneImageY;
  int l_o_wi = blockIdx.x % perBlockOneImageX;
  int l_o_h = l_o_hi * BS + l_y;
  //    int l_o_w = l_o_wi * BS + l_x;
  if(l_o_h >= tmp_o_h || g_x >= tmp_o_w) return;

  const int32_t F_H = f_h;
  const int32_t F_W = f_w;
  //    __shared__ int32_t shared_i[BS + F_H - 1][BS + F_W - 1];
  int32_t sih = BS + tmp_f_h - 1;
  int32_t siw = BS + tmp_f_w - 1;
  extern __shared__ int32_t  share[];
  int32_t *shared_i = (int32_t*)share; 
  int32_t *shared_f = &share[sih * siw];

  int32_t sum = 0; 
  int min_s_y = (l_o_hi+1) * BS <= tmp_o_h ? BS : tmp_o_h%BS;
  int min_s_x = (l_o_wi+1) * BS <= tmp_o_w ? BS : tmp_o_w%BS;

  //load input to shared
  int l_i_h = l_o_h - padding_h;
  int i_y = l_o_c * i_h + l_i_h;
  int i_x = g_x - padding_w;
  // 0~2-> -1~1
  if(l_i_h < 0 || i_x < 0 || l_i_h >= i_h || i_x >= i_w)
    shared_i[l_y*siw + l_x] = 0;
  else
    shared_i[l_y*siw + l_x] = input[i_y * i_w + i_x];

  if(l_y < tmp_f_h-1){
    for(int i = l_y; i < tmp_f_h-1; i+=min_s_y){
      if(l_i_h+min_s_y+i-l_y < 0 || i_x < 0 || l_i_h+min_s_y+i-l_y >= i_h || i_x >= i_w)
        shared_i[(i+min_s_y)*siw + l_x] = 0;
      else
        shared_i[(i + min_s_y)*siw + l_x] = input[(i_y + min_s_y + i - l_y) * i_w + i_x]; 
    }
  }
  if(l_x < tmp_f_w-1){
    for(int i = l_x; i < tmp_f_w-1; i+= min_s_x){
      if(l_i_h < 0 || i_x+min_s_x+i-l_x < 0 || l_i_h >= i_h || i_x+min_s_x+i-l_x >= i_w)
        shared_i[l_y * siw + i+min_s_x] = 0;
      else
        shared_i[l_y * siw + i + min_s_x] = input[i_y * i_w + i_x + min_s_x + i - l_x];
    }
  }
  if(l_y < tmp_f_h-1 && l_x < tmp_f_w-1){
    for(int i = l_y; i < tmp_f_h-1; i+=min_s_y){
      for(int j = l_x; j < tmp_f_w-1; j+=min_s_x){
        if(l_i_h+min_s_y+i-l_y < 0 || i_x+min_s_x+j-l_x < 0 || l_i_h+min_s_y+i-l_y >= i_h || i_x+min_s_x+j-l_x >= i_w)
          shared_i[(i+min_s_y) * siw + j+min_s_x] = 0;
        else
          shared_i[(i+min_s_y) * siw + j+min_s_x] = input[(i_y+min_s_y + i-l_y)*i_w + i_x + min_s_x + j - l_x];
      }
    }
  }

  //load filter to shared;
  if(l_y < F_H && l_x < F_W){
    for(int i = l_y; i < F_H; i+= min_s_y)
      for(int j = l_x; j < F_W; j+=min_s_x)
        shared_f[i*F_W + j] = filter[l_o_c * F_H * F_W + i * F_W + j];
  }
  __syncthreads();

  for(int fy = 0; fy < F_H; fy++){
    for(int fx = 0; fx < F_W; fx++){
      sum += shared_i[(l_y+fy*dilation_h)*siw + l_x+fx*dilation_w] * shared_f[fy*F_W + fx];
    }
  } 
  __syncthreads();

  if(l_o_h % stride_h == 0 && g_x % stride_w == 0){
    //int oi = l_o_c * o_h * o_w + l_o_h * o_w + g_x;
    int oi = l_o_c * o_h * o_w + l_o_h/stride_h * o_w + g_x/stride_w;
    output[oi] = sum + (bias != NULL ? bias[l_o_c%o_c] : 0);
  }
}
__global__ void kernel_depthwise_conv2d_no_shared(
    int32_t *input, int32_t i_n, int32_t i_c, int32_t i_h, int32_t i_w,
    int32_t *filter, int32_t f_n, int32_t f_c, int32_t f_h, int32_t f_w,
    int32_t *bias,
    int32_t padding_h, int32_t padding_w,
    int32_t stride_h, int32_t stride_w,
    int32_t dilation_h, int32_t dilation_w, 
    int32_t groups,
    int32_t *output, int32_t o_n, int32_t o_c, int32_t o_h, int32_t o_w){
  int32_t gy = threadIdx.y + blockIdx.y * blockDim.y;
  int32_t gx = threadIdx.x + blockIdx.x * blockDim.x;
  int32_t l_o_h = gy % o_h;
  int32_t l_o_c = gy / o_h % o_c;
  int32_t l_o_n = gy / (o_h * o_c);
  if(gy < o_n * o_c * o_h && gx < o_w){
    int32_t sum = 0;
    for(int fy = 0; fy < f_h; ++fy){
      for(int fx = 0; fx < f_w; ++fx){
        int32_t l_i_h = l_o_h * stride_h + fy * dilation_h - padding_h;
        int32_t l_i_w = gx * stride_w + fx * dilation_h - padding_w;
        int32_t x;
        if(l_i_h < 0 || l_i_w < 0 || l_i_h >= i_h || l_i_w >= i_w)
          x = 0;
        else x = input[l_o_n * i_c * i_h * i_w + l_o_c * i_h * i_w + l_i_h * i_w + l_i_w];
        sum += x * filter[l_o_n * i_c * f_h * f_w + l_o_c * f_h * f_w + fy * f_w + fx];
      }
    }
    output[gy * o_w + gx] = sum + (bias != NULL ? bias[l_o_c] : 0);
  }
}
const char* cuda_depthwise_conv2d(
    int32_t *input, int32_t i_n, int32_t i_c, int32_t i_h, int32_t i_w,
    int32_t *filter, int32_t f_n, int32_t f_c, int32_t f_h, int32_t f_w,
    int32_t *bias,
    int32_t padding_h, int32_t padding_w,
    int32_t stride_h, int32_t stride_w,
    int32_t dilation_h, int32_t dilation_w,
    int32_t groups,
    int32_t *output, int32_t o_n, int32_t o_c, int32_t o_h, int32_t o_w, int32_t device_id, bool debug){
  int32_t *dev_i = input, *dev_f = filter, *dev_o = output, *dev_b = bias;
  size_t s_i = i_n * i_c * i_h * i_w * sizeof(int32_t);
  size_t s_f = f_n * f_c * f_h * f_w * sizeof(int32_t);
  size_t s_b = o_c * sizeof(int32_t); 
  size_t s_o = o_n * o_c * o_h * o_w * sizeof(int32_t);
  if(debug){
    hipMalloc((void**)&dev_i, s_i);
    hipMalloc((void**)&dev_f, s_f);
    hipMalloc((void**)&dev_o, s_o);
    hipMemcpy(dev_i, input, s_i, hipMemcpyHostToDevice);
    hipMemcpy(dev_f, filter, s_f, hipMemcpyHostToDevice);
    if(bias != NULL){
      hipMalloc((void**)&dev_b, s_b);
      hipMemcpy(dev_b, bias, s_b, hipMemcpyHostToDevice);
    }
  }
  //    clock_t start = clock();
  int b_h = BS;
  int b_w = BS;
  int tmp_f_h = (f_h - 1) * dilation_h + 1; // for dilation, to be optimized
  int tmp_f_w = (f_w - 1) * dilation_w + 1;
  int tmp_o_h = i_h + 2 * padding_h - tmp_f_h + 1; //for stride > 1
  int tmp_o_w = i_w + 2 * padding_w - tmp_f_w + 1;
  const int32_t totalShareMemSize = getShareMemorySize(device_id);
  size_t share_size = (BS + tmp_f_h - 1) * (BS + tmp_f_w - 1) * sizeof(int32_t) + f_h * f_w * sizeof(int32_t);
  if(share_size < totalShareMemSize){
    int32_t g_h = o_n * o_c * ((tmp_o_h + b_h - 1) / b_h);
    int32_t g_w = (tmp_o_w + b_w - 1) / b_w;
    dim3 bDim(b_w, b_h, 1);
    dim3 gDim(g_w, g_h, 1);
    kernel_depthwise_conv2d<<<gDim, bDim, share_size>>>(
        dev_i, i_n, i_c, i_h, i_w,
        dev_f, f_n, f_c, f_h, f_w,
        dev_b, 
        padding_h, padding_w,
        stride_h, stride_w,
        dilation_h, dilation_w,
        groups,
        dev_o, o_n, o_c, o_h, o_w);
  }else{
    int32_t g_h = o_n * o_c * ((o_h + b_h - 1) / b_h); 
    int32_t g_w = (o_w + b_w - 1) / b_w;
    dim3 bDim(b_w, b_h, 1);
    dim3 gDim(g_w, g_h, 1);
    kernel_depthwise_conv2d_no_shared<<<gDim, bDim>>>(
        dev_i, i_n, i_c, i_h, i_w,
        dev_f, f_n, f_c, f_h, f_w,
        dev_b, 
        padding_h, padding_w,
        stride_h, stride_w,
        dilation_h, dilation_w,
        groups,
        dev_o, o_n, o_c, o_h, o_w);
  }
  //hipDeviceSynchronize();
  //    clock_t end = clock();
  //    printf("gpu cal time: %d\n", end-start);
  if(debug){
    hipMemcpy(output, dev_o, s_o, hipMemcpyDeviceToHost);
    hipFree(dev_i);
    hipFree(dev_f);
    hipFree(dev_o);
    if(bias != NULL)
      hipFree(dev_b);
  }
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_max_pool(
    int32_t *input, int32_t i_n, int32_t i_c, int32_t i_h, int32_t i_w,
    int32_t f_h, int32_t f_w,
    int32_t padding_h, int32_t padding_w,
    int32_t stride_h, int32_t stride_w,
    int32_t *output, int32_t o_n, int32_t o_c, int32_t o_h, int32_t o_w){
  //    int g_y = blockDim.y * blockIdx.y + threadIdx.y;
  int g_x = blockDim.x * blockIdx.x + threadIdx.x;
  int l_y = threadIdx.y; 
  int l_x = threadIdx.x;
  int tmp_o_h = i_h + 2 * padding_h - f_h + 1; // for stride
  int tmp_o_w = i_w + 2 * padding_w - f_w + 1;
  int perBlockOneImageY = (tmp_o_h+BS-1) / BS;
  int perBlockOneImageX = (tmp_o_w+BS-1) / BS;
  int l_o_c = blockIdx.y / perBlockOneImageY;
  int l_o_hi = blockIdx.y % perBlockOneImageY;
  int l_o_wi = blockIdx.x % perBlockOneImageX;
  int l_o_h = l_o_hi * BS + l_y;
  //    int l_o_w = l_o_wi * BS + l_x;
  if(l_o_h >= tmp_o_h || g_x >= tmp_o_w) return;

  const int32_t F_H = f_h;
  const int32_t F_W = f_w;
  //    __shared__ int32_t shared_i[BS + F_H - 1][BS + F_W - 1];
  //    int32_t sih = BS + F_H - 1;
  int32_t siw = BS + F_W - 1;
  extern __shared__ int32_t  share[];
  int32_t *shared_i = (int32_t*)share; 

  int32_t max_elem = int(1)<<31; 
  int min_s_y = (l_o_hi+1) * BS <= tmp_o_h ? BS : tmp_o_h%BS;
  int min_s_x = (l_o_wi+1) * BS <= tmp_o_w ? BS : tmp_o_w%BS;

  //load input to shared
  int l_i_h = l_o_h - padding_h;
  int i_y = l_o_c * i_h + l_i_h;
  int i_x = g_x - padding_w;
  // 0~2-> -1~1
  if(l_i_h < 0 || i_x < 0 || l_i_h >= i_h || i_x >= i_w)
    shared_i[l_y*siw + l_x] = 0;
  else
    shared_i[l_y*siw + l_x] = input[i_y * i_w + i_x];

  if(l_y < F_H-1){
    for(int i = l_y; i < F_H-1; i+=min_s_y){
      if(l_i_h+min_s_y+i-l_y < 0 || i_x < 0 || l_i_h+min_s_y+i-l_y >= i_h || i_x >= i_w)
        shared_i[(i+min_s_y)*siw + l_x] = 0;
      else
        shared_i[(i + min_s_y)*siw + l_x] = input[(i_y + min_s_y + i - l_y) * i_w + i_x];     
    }
  }
  if(l_x < F_W-1){
    for(int i = l_x; i < F_W-1; i+= min_s_x){
      if(l_i_h < 0 || i_x+min_s_x+i-l_x < 0 || l_i_h >= i_h || i_x+min_s_x+i-l_x >= i_w)
        shared_i[l_y * siw + i+min_s_x] = 0;
      else
        shared_i[l_y * siw + i + min_s_x] = input[i_y * i_w + i_x + min_s_x + i - l_x];
    }
  }
  if(l_y < F_H-1 && l_x < F_W-1){
    for(int i = l_y; i < F_H-1; i+=min_s_y){
      for(int j = l_x; j < F_W-1; j+=min_s_x){
        if(l_i_h+min_s_y+i-l_y < 0 || i_x+min_s_x+j-l_x < 0 || l_i_h+min_s_y+i-l_y >= i_h || i_x+min_s_x+j-l_x >= i_w)
          shared_i[(i+min_s_y) * siw + j+min_s_x] = 0;
        else
          shared_i[(i+min_s_y) * siw + j+min_s_x] = input[(i_y+min_s_y + i-l_y)*i_w + i_x + min_s_x + j - l_x];
      }
    }
  }
  __syncthreads();

  for(int fy = 0; fy < F_H; fy++){
    for(int fx = 0; fx < F_W; fx++){
      int32_t tmp =  shared_i[(l_y+fy)*siw + l_x+fx];
      max_elem = max_elem < tmp ? tmp : max_elem;
    }
  } 
  __syncthreads();

  if(l_o_h % stride_h == 0 && g_x % stride_w == 0){
    //int oi = l_o_c * o_h * o_w + l_o_h * o_w + g_x;
    int oi = l_o_c * o_h * o_w + l_o_h/stride_h * o_w + g_x/stride_w;
    output[oi] = max_elem;
  }
}

__global__ void kernel_max_pool_no_shared(
    int32_t *input, int32_t i_n, int32_t i_c, int32_t i_h, int32_t i_w,
    int32_t f_h, int32_t f_w,
    int32_t padding_h, int32_t padding_w,
    int32_t stride_h, int32_t stride_w,
    int32_t *output, int32_t o_n, int32_t o_c, int32_t o_h, int32_t o_w){
  int32_t gy = threadIdx.y + blockIdx.y * blockDim.y;
  int32_t gx = threadIdx.x + blockIdx.x * blockDim.x;
  int32_t l_o_h = gy % o_h;
  int32_t l_o_c = gy / o_h % o_c;
  int32_t l_o_n = gy / (o_h * o_c);
  if(gy < o_n * o_c * o_h && gx < o_w){
    int32_t maxV = (int32_t)1 << 31;
    for(int fy = 0; fy < f_h; ++fy){
      for(int fx = 0; fx < f_w; ++fx){
        int32_t l_i_h = l_o_h * stride_h + fy  - padding_h;
        int32_t l_i_w = gx * stride_w + fx - padding_w;
        int32_t x;
        if(l_i_h < 0 || l_i_w < 0 || l_i_h >= i_h || l_i_w >= i_w)
          x = 0;
        else x = input[l_o_n * i_c * i_h * i_w + l_o_c * i_h * i_w + l_i_h * i_w + l_i_w];
        maxV = maxV < x ? x : maxV;
      }
    }
    output[gy * o_w + gx] = maxV;
  }
}
const char* cuda_max_pool(
    int32_t *input, int32_t i_n, int32_t i_c, int32_t i_h, int32_t i_w,
    const int32_t f_h, const int32_t f_w,
    int32_t padding_h, int32_t padding_w,
    int32_t stride_h, int32_t stride_w,
    int32_t *output, int32_t o_n, int32_t o_c, int32_t o_h, int32_t o_w, int32_t device_id, bool debug){
  int32_t *dev_i = input, *dev_o = output;
  size_t s_i = i_n * i_c * i_h * i_w * sizeof(int32_t);
  size_t s_o = o_n * o_c * o_h * o_w * sizeof(int32_t);
  if(debug){
    hipMalloc((void**)&dev_i, s_i);
    hipMalloc((void**)&dev_o, s_o);
    hipMemcpy(dev_i, input, s_i, hipMemcpyHostToDevice);
  }

  //    clock_t start = clock();
  const int32_t totalShareMemSize = getShareMemorySize(device_id);
  size_t share_size = (BS + f_h - 1) * (BS + f_w - 1) * sizeof(int32_t);
  int b_h = BS;
  int b_w = BS;
  int tmp_o_h = i_h + 2 * padding_h - f_h + 1; //for stride > 1
  int tmp_o_w = i_w + 2 * padding_w - f_w + 1;
  if(share_size < totalShareMemSize){
    int32_t g_h = o_n * o_c * ((tmp_o_h + b_h - 1) / b_h);
    int32_t g_w = (tmp_o_w + b_w - 1) / b_w;
    dim3 bDim(b_w, b_h, 1);
    dim3 gDim(g_w, g_h, 1);
    kernel_max_pool<<<gDim, bDim, share_size>>>(
        dev_i, i_n, i_c, i_h, i_w,
        f_h, f_w,
        padding_h, padding_w, 
        stride_h, stride_w,
        dev_o, o_n, o_c, o_h, o_w);
  }else{
    int32_t g_h = o_n * o_c * ((o_h + b_h - 1) / b_h); 
    int32_t g_w = (o_w + b_w - 1) / b_w;
    dim3 bDim(b_w, b_h, 1);
    dim3 gDim(g_w, g_h, 1);
    kernel_max_pool_no_shared<<<gDim, bDim>>>(
        dev_i, i_n, i_c, i_h, i_w,
        f_h, f_w,
        padding_h, padding_w, 
        stride_h, stride_w,
        dev_o, o_n, o_c, o_h, o_w);
  }
  //hipDeviceSynchronize();
  //    clock_t end = clock();
  //    printf("gpu cal time: %ld\n", end-start);
  if(debug){
    hipMemcpy(output, dev_o, s_o, hipMemcpyDeviceToHost);
    hipFree(dev_i);
    hipFree(dev_o);
  }
  return check_cuda_error(hipGetLastError());
}

#define TILE_WIDTH 16
__global__ void kernel_dense(
    int32_t *A, // m*k 
    int32_t *B, // was transposed, n*k
    int32_t *C, // m*n
    int32_t m, int32_t k, int32_t n, int32_t *bias, int32_t useBias){
  __shared__ int32_t sharedM[TILE_WIDTH][TILE_WIDTH];
  __shared__ int32_t sharedN[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int row = by*TILE_WIDTH + ty;
  int col = bx*TILE_WIDTH + tx;
  int sum = 0;

  for (int i = 0; i < (int)(ceil((float)k/TILE_WIDTH)); i++)
  {
    if (i*TILE_WIDTH + tx < k && row < m)//m*k
      sharedM[ty][tx] = A[row*k + i*TILE_WIDTH + tx];
    else
      sharedM[ty][tx] = 0;

    if(i*TILE_WIDTH + ty < k && col < n)//n*k
      sharedN[tx][ty] = B[col * k + i * TILE_WIDTH + ty];
    else
      sharedN[tx][ty] = 0;
    __syncthreads();

    for(int j = 0; j < TILE_WIDTH; j++)
      sum += sharedM[ty][j] * sharedN[tx][j];
    __syncthreads();
  }
  if (row < m && col < n){
    if(useBias == 1) sum += bias[col];
    C[row*n + col] = sum;
  }
}

const char* cuda_dense(
    int32_t *a,
    int32_t *b,
    int32_t *c,
    const int m, const int k, const int n, int32_t* bias, bool debug){
  int32_t *dev_a = a, *dev_b = b, *dev_c = c, *dev_bias = bias, useBias = 0;
  size_t s_a = sizeof(int32_t) * m * k;
  size_t s_b = sizeof(int32_t) * k * n;
  size_t s_c = sizeof(int32_t) * m * n;
  size_t s_bias = sizeof(int32_t) * n;
  if(debug){
    hipMalloc((void**)&dev_a, s_a);
    hipMalloc((void**)&dev_b, s_b);
    hipMalloc((void**)&dev_c, s_c);
    if(bias != NULL){
      hipMalloc((void**)&dev_bias, s_bias);
      hipMemcpy(dev_bias, bias, s_bias, hipMemcpyHostToDevice);
      useBias = 1;
    }
    hipMemcpy(dev_a, a, s_a, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, s_b, hipMemcpyHostToDevice);
  }
  if(bias != NULL) useBias = 1;

  dim3 bDim(TILE_WIDTH, TILE_WIDTH, 1);
  int gh = (m + TILE_WIDTH - 1) / TILE_WIDTH;
  int gw = (n + TILE_WIDTH - 1) / TILE_WIDTH;
  dim3 gDim(gw, gh, 1);
  kernel_dense<<<gDim, bDim>>>(dev_a, dev_b, dev_c, m, k, n, dev_bias, useBias);
  //hipDeviceSynchronize();
  if(debug){
    hipMemcpy(c, dev_c, s_c, hipMemcpyDeviceToHost);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipFree(dev_bias);
  }
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_clip(const int32_t *x, int32_t *y,
    const uint64_t n, const int32_t maxV, const int32_t minV){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for(uint64_t i = tid; i < n; i += gridDim.x*blockDim.x){
    y[i] = max(min(x[i], maxV), minV);
  }
}
const char* cuda_clip(const int32_t *x, int32_t *y, const uint64_t n, const int32_t max, const int32_t min, bool debug){
  const int32_t *dev_x = x;
  int32_t *tmp_x;
  int32_t *dev_y = y;
  if(debug) {
    hipMalloc((void**)&tmp_x, n*sizeof(int32_t));
    dev_x = tmp_x;
    hipMalloc((void**)&dev_y, n*sizeof(int32_t));
    hipMemcpy(tmp_x, x, sizeof(int32_t)*n, hipMemcpyHostToDevice);
  }

  int threadSize = 256;
  int blockSize = getGridSize(n, threadSize); //(n + threadSize - 1) / threadSize;
  kernel_clip<<<blockSize, threadSize>>>(dev_x, dev_y, n, max, min);
  // hipDeviceSynchronize();

  if(debug){
    hipMemcpy(y, dev_y, sizeof(int32_t)*n, hipMemcpyDeviceToHost);
    hipFree(tmp_x);
    hipFree(dev_y);
  }
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_relu(const int32_t *x, int32_t*y, const uint64_t n){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for(uint64_t i = tid; i < n; i += gridDim.x * blockDim.x){
    y[i] = max(x[i], 0);
  }
}
const char* cuda_relu(const int32_t *x, int32_t *y, const uint64_t n, bool debug){
  const int32_t *dev_x = x;
  int32_t *tmp_x;
  int32_t *dev_y = y;
  if(debug) {
    hipMalloc((void**)&tmp_x, n*sizeof(int32_t));
    dev_x = tmp_x;
    hipMalloc((void**)&dev_y, n*sizeof(int32_t));
    hipMemcpy(tmp_x, x, sizeof(int32_t)*n, hipMemcpyHostToDevice);
  }

  int threadSize = 256;
  int blockSize = getGridSize(n, threadSize);//(n + threadSize - 1) / threadSize;
  kernel_relu<<<blockSize, threadSize>>>(dev_x, dev_y, n);
  //hipDeviceSynchronize();

  if(debug){
    hipMemcpy(y, dev_y, sizeof(int32_t)*n, hipMemcpyDeviceToHost);
    hipFree(tmp_x);
    hipFree(dev_y);
  }
  return check_cuda_error(hipGetLastError());
}

const char* cuda_flatten(const int32_t *x, int32_t *y, const uint64_t n, bool debug){
  if(x == y) return NULL;
  hipMemcpy(y, x, n * sizeof(int32_t), hipMemcpyDeviceToDevice);
  return check_cuda_error(hipGetLastError());
}

inline __device__ int32_t broadcast_i_index(int64_t* oshape, int o_index, int64_t* ishape, int idim){
  int index = 0;
  int allIndex = 0;
  for(int i = 0; i < idim; i++){
    int idx = idim - 1 - i;
    int ovar = o_index % oshape[idx];
    if(ovar < ishape[idx]){
      index += i == 0 ? ovar : allIndex * ovar;
    }else if(ishape[idx] == 1){
    }else{
    }
    allIndex = (i == 0 ? ishape[idim-1] : allIndex * ishape[idx]);
    o_index /= oshape[idx];
  }
  return index;
}

__global__ void kernel_broadcast_add(const int32_t *a, const int32_t *b, int32_t*c, 
    const int64_t n,
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for(uint64_t i = tid; i < n; i += gridDim.x * blockDim.x){
    int ai = broadcast_i_index(cshape, i, ashape, adim);
    int bi = broadcast_i_index(cshape, i, bshape, bdim);
    c[i] = a[ai] + b[bi];
  }
}
const char* cuda_broadcast_add(const int32_t *a, const int32_t *b, int32_t* c, 
    const uint64_t n, 
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim,
    bool debug)
{
  const int32_t *dev_a = a, *dev_b = b;
  int32_t *tmp_a, *tmp_b;
  int32_t *dev_c = c;
  if(debug) {
    hipMalloc((void**)&tmp_a, n*sizeof(int32_t));
    dev_a = tmp_a;
    hipMalloc((void**)&tmp_b, sizeof(int32_t));
    dev_b = tmp_b;
    hipMalloc((void**)&dev_c, n*sizeof(int32_t));
    hipMemcpy(tmp_a, a, sizeof(int32_t)*n, hipMemcpyHostToDevice);
    hipMemcpy(tmp_b, b, sizeof(int32_t), hipMemcpyHostToDevice);
  }

  int64_t *dev_ashape, *dev_bshape, *dev_cshape;
  hipError_t status;
  status = hipMalloc((void**)&dev_ashape, sizeof(int64_t) * adim);
  if(status != hipSuccess){
    return check_cuda_error(status);
  }
  status = hipMalloc((void**)&dev_bshape, sizeof(int64_t) * bdim);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    return check_cuda_error(status);
  }
  status = hipMalloc((void**)&dev_cshape, sizeof(int64_t) * cdim);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_ashape, ashape, sizeof(int64_t) * adim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    hipFree(dev_cshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_bshape, bshape, sizeof(int64_t) * bdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    hipFree(dev_cshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_cshape, cshape, sizeof(int64_t) * cdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    hipFree(dev_cshape);
    return check_cuda_error(status);
  }
  int threadSize = 256;
  int blockSize = getGridSize(n, threadSize);//(n + threadSize - 1) / threadSize;
  kernel_broadcast_add<<<blockSize, threadSize>>>(dev_a, dev_b, dev_c, n, dev_ashape, adim, dev_bshape, bdim, dev_cshape, cdim);
  //hipDeviceSynchronize();

  if(debug){
    hipMemcpy(c, dev_c, sizeof(int32_t)*n, hipMemcpyDeviceToHost);
    hipFree(tmp_a);
    hipFree(dev_c);
    hipFree(tmp_b);
  }
  hipFree(dev_ashape);
  hipFree(dev_bshape);
  hipFree(dev_cshape);
  return check_cuda_error(hipGetLastError());
}
__global__ void kernel_broadcast_sub(const int32_t *a, const int32_t *b, int32_t*c, const uint64_t n,
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim
    ){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for(int i = tid; i < n; i += gridDim.x*blockDim.x){
    int32_t ai = broadcast_i_index(cshape, i, ashape, adim);
    int32_t bi = broadcast_i_index(cshape, i, bshape, bdim);
    c[i] = a[ai] - b[bi];
  }
}
const char* cuda_broadcast_sub(const int32_t *a, const int32_t *b, int32_t* c, const uint64_t n, 
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim,
    bool debug){
  const int32_t *dev_a = a, *dev_b = b;
  int32_t *tmp_a, *tmp_b;
  int32_t *dev_c = c;
  if(debug) {
    hipMalloc((void**)&tmp_a, n*sizeof(int32_t));
    dev_a = tmp_a;
    hipMalloc((void**)&tmp_b, sizeof(int32_t));
    dev_b = tmp_b;
    hipMalloc((void**)&dev_c, n*sizeof(int32_t));
    hipMemcpy(tmp_a, a, sizeof(int32_t)*n, hipMemcpyHostToDevice);
    hipMemcpy(tmp_b, b, sizeof(int32_t), hipMemcpyHostToDevice);
  }

  int64_t *dev_ashape, *dev_bshape, *dev_cshape;
  hipError_t status;
  status = hipMalloc((void**)&dev_ashape, sizeof(int64_t) * adim);
  if(status != hipSuccess){
    return check_cuda_error(status);
  }
  status = hipMalloc((void**)&dev_bshape, sizeof(int64_t) * bdim);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    return check_cuda_error(status);
  }
  status = hipMalloc((void**)&dev_cshape, sizeof(int64_t) * cdim);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_ashape, ashape, sizeof(int64_t) * adim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    hipFree(dev_cshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_bshape, bshape, sizeof(int64_t) * bdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    hipFree(dev_cshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_cshape, cshape, sizeof(int64_t) * cdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    hipFree(dev_cshape);
    return check_cuda_error(status);
  }
  int threadSize = 256;
  int blockSize = getGridSize(n, threadSize);//(n + threadSize - 1) / threadSize;
  kernel_broadcast_sub<<<blockSize, threadSize>>>(dev_a, dev_b, dev_c, n, dev_ashape, adim, dev_bshape, bdim, dev_cshape, cdim);
  //hipDeviceSynchronize();

  hipFree(dev_ashape);
  hipFree(dev_bshape);
  hipFree(dev_cshape);

  if(debug){
    hipMemcpy(c, dev_c, sizeof(int32_t)*n, hipMemcpyDeviceToHost);
    hipFree(tmp_a);
    hipFree(dev_c);
    hipFree(tmp_b);
  }
  return check_cuda_error(hipGetLastError());
}
__global__ void kernel_broadcast_mul(const int32_t *a, const int32_t *b, int32_t*c, const uint64_t n,
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim
    ){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for(int i = tid; i < n; i += gridDim.x*blockDim.x){
    int32_t ai = broadcast_i_index(cshape, i, ashape, adim);
    int32_t bi = broadcast_i_index(cshape, i, bshape, bdim);
    c[i] = a[ai] * b[bi];
  }
}
const char* cuda_broadcast_mul(const int32_t *a, const int32_t *b, int32_t* c, const uint64_t n, 
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim,
    bool debug){
  const int32_t *dev_a = a, *dev_b = b;
  int32_t *tmp_a, *tmp_b;
  int32_t *dev_c = c;
  if(debug) {
    hipMalloc((void**)&tmp_a, n*sizeof(int32_t));
    dev_a = tmp_a;
    hipMalloc((void**)&tmp_b, sizeof(int32_t));
    dev_b = tmp_b;
    hipMalloc((void**)&dev_c, n*sizeof(int32_t));
    hipMemcpy(tmp_a, a, sizeof(int32_t)*n, hipMemcpyHostToDevice);
    hipMemcpy(tmp_b, b, sizeof(int32_t), hipMemcpyHostToDevice);
  }

  int64_t *dev_ashape, *dev_bshape, *dev_cshape;
  hipError_t status;
  status = hipMalloc((void**)&dev_ashape, sizeof(int64_t) * adim);
  if(status != hipSuccess){
    return check_cuda_error(status);
  }
  status = hipMalloc((void**)&dev_bshape, sizeof(int64_t) * bdim);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    return check_cuda_error(status);
  }
  status = hipMalloc((void**)&dev_cshape, sizeof(int64_t) * cdim);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_ashape, ashape, sizeof(int64_t) * adim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    hipFree(dev_cshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_bshape, bshape, sizeof(int64_t) * bdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    hipFree(dev_cshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_cshape, cshape, sizeof(int64_t) * cdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    hipFree(dev_cshape);
    return check_cuda_error(status);
  }
  int threadSize = 256;
  int blockSize = getGridSize(n, threadSize);//(n + threadSize - 1) / threadSize;
  kernel_broadcast_mul<<<blockSize, threadSize>>>(dev_a, dev_b, dev_c, n, dev_ashape, adim, dev_bshape, bdim, dev_cshape, cdim);
  //hipDeviceSynchronize();

  hipFree(dev_ashape);
  hipFree(dev_bshape);
  hipFree(dev_cshape);
  if(debug){
    hipMemcpy(c, dev_c, sizeof(int32_t)*n, hipMemcpyDeviceToHost);
    hipFree(tmp_a);
    hipFree(dev_c);
    hipFree(tmp_b);
  }
  print_to_file(dev_c, n, "/tmp/zkh/cuda_mul.txt");
  return check_cuda_error(hipGetLastError());
}
__global__ void kernel_broadcast_div(const int32_t *a, const int32_t *b, int32_t*c, const uint64_t n,
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim
    ){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for(int i = tid; i < n; i += gridDim.x*blockDim.x){
    int32_t ai = broadcast_i_index(cshape, i, ashape, adim);
    int32_t bi = broadcast_i_index(cshape, i, bshape, bdim);
    c[i] = a[ai] / b[bi];
  }
}
const char* cuda_broadcast_div(const int32_t *a, const int32_t *b, int32_t* c, const uint64_t n,
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim,
    bool debug){
  const int32_t *dev_a = a, *dev_b = b;
  int32_t *tmp_a, *tmp_b;
  int32_t *dev_c = c;
  if(debug) {
    hipMalloc((void**)&tmp_a, n*sizeof(int32_t));
    dev_a = tmp_a;
    hipMalloc((void**)&tmp_b, sizeof(int32_t));
    dev_b = tmp_b;
    hipMalloc((void**)&dev_c, n*sizeof(int32_t));
    hipMemcpy(tmp_a, a, sizeof(int32_t)*n, hipMemcpyHostToDevice);
    hipMemcpy(tmp_b, b, sizeof(int32_t), hipMemcpyHostToDevice);
  }

  hipError_t status;
  int64_t bsize = 1;
  for(int i = 0; i < bdim; i++){
    bsize *= bshape[i];
  }
  int32_t* h_b = new int32_t[bsize];
  status = hipMemcpy(h_b, dev_b, sizeof(int32_t) * bsize, hipMemcpyDeviceToHost);
  if(status != hipSuccess){
    return check_cuda_error(status);
  }
  for(int i = 0; i < bsize; i++){
    if(h_b == 0){
      delete h_b;
      return "error: divide by zero";
    }
  }

  int64_t *dev_ashape, *dev_bshape, *dev_cshape;
  status = hipMalloc((void**)&dev_ashape, sizeof(int64_t) * adim);
  if(status != hipSuccess){
    delete h_b;
    return check_cuda_error(status);
  }
  status = hipMalloc((void**)&dev_bshape, sizeof(int64_t) * bdim);
  if(status != hipSuccess){
    delete h_b;
    hipFree(dev_ashape);
    return check_cuda_error(status);
  }
  status = hipMalloc((void**)&dev_cshape, sizeof(int64_t) * cdim);
  if(status != hipSuccess){
    delete h_b;
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_ashape, ashape, sizeof(int64_t) * adim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    delete h_b;
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    hipFree(dev_cshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_bshape, bshape, sizeof(int64_t) * bdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    delete h_b;
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    hipFree(dev_cshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_cshape, cshape, sizeof(int64_t) * cdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    delete h_b;
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    hipFree(dev_cshape);
    return check_cuda_error(status);
  }
  int threadSize = 256;
  int blockSize = getGridSize(n, threadSize);//(n + threadSize - 1) / threadSize;
  kernel_broadcast_div<<<blockSize, threadSize>>>(dev_a, dev_b, dev_c, n, dev_ashape, adim, dev_bshape, bdim, dev_cshape, cdim);
  //hipDeviceSynchronize();
  hipFree(dev_ashape);
  hipFree(dev_bshape);
  hipFree(dev_cshape);

  if(debug){
    hipMemcpy(c, dev_c, sizeof(int32_t)*n, hipMemcpyDeviceToHost);
    hipFree(tmp_a);
    hipFree(dev_c);
    hipFree(tmp_b);
  }
  return check_cuda_error(hipGetLastError());
}
__global__ void kernel_broadcast_right_shift(const int32_t *a, const int32_t *b, int32_t*c, const uint64_t n,
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim
    ){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for(int i = tid; i < n; i += gridDim.x * blockDim.x){
    int32_t ai = broadcast_i_index(cshape, i, ashape, adim);
    int32_t bi = broadcast_i_index(cshape, i, bshape, bdim);
    c[i] = a[ai] >> b[bi];
  }
}
const char* cuda_broadcast_right_shift(const int32_t *a, const int32_t* b, int32_t* c, const uint64_t n,
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim,
    bool debug){
  const int32_t *dev_a = a;
  const int32_t *dev_b = b;
  int32_t *tmp_a, *tmp_b;
  int32_t *dev_c = c;
  if(debug) {
    hipMalloc((void**)&tmp_a, n*sizeof(int32_t));
    dev_a = tmp_a;
    hipMalloc((void**)&tmp_b, sizeof(int32_t));
    dev_b = tmp_b;
    hipMalloc((void**)&dev_c, n*sizeof(int32_t));
    hipMemcpy(tmp_a, a, sizeof(int32_t)*n, hipMemcpyHostToDevice);
    hipMemcpy(tmp_b, b, sizeof(int32_t), hipMemcpyHostToDevice);
  }

  int64_t *dev_ashape, *dev_bshape, *dev_cshape;
  hipError_t status;
  status = hipMalloc((void**)&dev_ashape, sizeof(int64_t) * adim);
  if(status != hipSuccess){
    return check_cuda_error(status);
  }
  status = hipMalloc((void**)&dev_bshape, sizeof(int64_t) * bdim);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    return check_cuda_error(status);
  }
  status = hipMalloc((void**)&dev_cshape, sizeof(int64_t) * cdim);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_ashape, ashape, sizeof(int64_t) * adim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    hipFree(dev_cshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_bshape, bshape, sizeof(int64_t) * bdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    hipFree(dev_cshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_cshape, cshape, sizeof(int64_t) * cdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    hipFree(dev_cshape);
    return check_cuda_error(status);
  }
  int threadSize = 256;
  int blockSize = getGridSize(n, threadSize);//(n + threadSize - 1) / threadSize;
  kernel_broadcast_right_shift<<<blockSize, threadSize>>>(dev_a, dev_b, dev_c, n, dev_ashape, adim, dev_bshape, bdim, dev_cshape, cdim);
  //hipDeviceSynchronize();
  hipFree(dev_ashape);
  hipFree(dev_bshape);
  hipFree(dev_cshape);

  if(debug){
    hipMemcpy(c, dev_c, sizeof(int32_t)*n, hipMemcpyDeviceToHost);
    hipFree(tmp_a);
    hipFree(dev_c);
    hipFree(tmp_b);
  }
  return check_cuda_error(hipGetLastError());
}
__global__ void kernel_broadcast_left_shift(const int32_t *a, const int32_t *b, int32_t*c, const uint64_t n,
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim
    ){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for(int i = tid; i < n; i += gridDim.x * blockDim.x){
    int32_t ai = broadcast_i_index(cshape, i, ashape, adim);
    int32_t bi = broadcast_i_index(cshape, i, bshape, bdim);
    c[i] = a[ai] << b[bi];
  }
}
const char* cuda_broadcast_left_shift(const int32_t *a, const int32_t *b, int32_t* c, const uint64_t n, 
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim,
    bool debug){
  const int32_t *dev_a = a, *dev_b = b;
  int32_t *tmp_a, *tmp_b;
  int32_t *dev_c = c;
  if(debug) {
    hipMalloc((void**)&tmp_a, n*sizeof(int32_t));
    dev_a = tmp_a;
    hipMalloc((void**)&tmp_b, sizeof(int32_t));
    dev_b = tmp_b;
    hipMalloc((void**)&dev_c, n*sizeof(int32_t));
    hipMemcpy(tmp_a, a, sizeof(int32_t)*n, hipMemcpyHostToDevice);
    hipMemcpy(tmp_b, b, sizeof(int32_t), hipMemcpyHostToDevice);
  }

  int64_t *dev_ashape, *dev_bshape, *dev_cshape;
  hipError_t status;
  status = hipMalloc((void**)&dev_ashape, sizeof(int64_t) * adim);
  if(status != hipSuccess){
    return check_cuda_error(status);
  }
  status = hipMalloc((void**)&dev_bshape, sizeof(int64_t) * bdim);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    return check_cuda_error(status);
  }
  status = hipMalloc((void**)&dev_cshape, sizeof(int64_t) * cdim);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_ashape, ashape, sizeof(int64_t) * adim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    hipFree(dev_cshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_bshape, bshape, sizeof(int64_t) * bdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    hipFree(dev_cshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_cshape, cshape, sizeof(int64_t) * cdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    hipFree(dev_cshape);
    return check_cuda_error(status);
  }
  int threadSize = 256;
  int blockSize = getGridSize(n, threadSize);//(n + threadSize - 1) / threadSize;
  kernel_broadcast_left_shift<<<blockSize, threadSize>>>(dev_a, dev_b, dev_c, n, dev_ashape, adim, dev_bshape, bdim, dev_cshape, cdim);
  //hipDeviceSynchronize();
  hipFree(dev_ashape);
  hipFree(dev_bshape);
  hipFree(dev_cshape);

  if(debug){
    hipMemcpy(c, dev_c, sizeof(int32_t)*n, hipMemcpyDeviceToHost);
    hipFree(tmp_a);
    hipFree(dev_c);
    hipFree(tmp_b);
  }
  return check_cuda_error(hipGetLastError());
}
__global__ void kernel_broadcast_max(const int32_t *a, const int32_t *b, int32_t *c, const uint64_t n,
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim
    ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for(int i = tid; i < n; i += gridDim.x*blockDim.x){
    int32_t ai = broadcast_i_index(cshape, i, ashape, adim);
    int32_t bi = broadcast_i_index(cshape, i, bshape, bdim);
    c[i] = a[ai] > b[bi] ? a[ai] : b[bi];
  }
}
const char* cuda_broadcast_max(const int32_t *a, const int32_t *b, int32_t* c, const uint64_t n, 
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim,
    bool debug){
  const int32_t *dev_a = a, *dev_b = b;
  int32_t *tmp_a, *tmp_b;
  int32_t *dev_c = c;
  if(debug) {
    hipMalloc((void**)&tmp_a, n*sizeof(int32_t));
    dev_a = tmp_a;
    hipMalloc((void**)&tmp_b, sizeof(int32_t));
    dev_b = tmp_b;
    hipMalloc((void**)&dev_c, n*sizeof(int32_t));
    hipMemcpy(tmp_a, a, sizeof(int32_t)*n, hipMemcpyHostToDevice);
    hipMemcpy(tmp_b, b, sizeof(int32_t), hipMemcpyHostToDevice);
  }

  int64_t *dev_ashape, *dev_bshape, *dev_cshape;
  hipError_t status;
  status = hipMalloc((void**)&dev_ashape, sizeof(int64_t) * adim);
  if(status != hipSuccess){
    return check_cuda_error(status);
  }
  status = hipMalloc((void**)&dev_bshape, sizeof(int64_t) * bdim);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    return check_cuda_error(status);
  }
  status = hipMalloc((void**)&dev_cshape, sizeof(int64_t) * cdim);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_ashape, ashape, sizeof(int64_t) * adim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    hipFree(dev_cshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_bshape, bshape, sizeof(int64_t) * bdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    hipFree(dev_cshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_cshape, cshape, sizeof(int64_t) * cdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_ashape);
    hipFree(dev_bshape);
    hipFree(dev_cshape);
    return check_cuda_error(status);
  }
  int threadSize = 256;
  int blockSize = getGridSize(n, threadSize);//(n + threadSize - 1) / threadSize;
  kernel_broadcast_max<<<blockSize, threadSize>>>(dev_a, dev_b, dev_c, n, dev_ashape, adim, dev_bshape, bdim, dev_cshape, cdim);
  //hipDeviceSynchronize();
  hipFree(dev_ashape);
  hipFree(dev_bshape);
  hipFree(dev_cshape);

  if(debug){
    hipMemcpy(c, dev_c, sizeof(int32_t)*n, hipMemcpyDeviceToHost);
    hipFree(tmp_a);
    hipFree(dev_c);
    hipFree(tmp_b);
  }
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_sum(const int32_t *x, int32_t *y, int64_t n){
  __shared__ int32_t buf[256];
  int32_t tid = threadIdx.x;
  int32_t sum = 0;
  for (int i = tid; i < n; i += blockDim.x){
    sum += x[i];
  }

  buf[tid] = sum;
  __syncthreads();
  for(int s = 1; s < blockDim.x; s*=2){
    if((tid % (2*s)) == 0){
      int a = buf[tid];
      int b = buf[tid+s];
      buf[tid] = a + b;
    }
    __syncthreads();
  }

  if(tid == 0) y[0] = buf[0];
}

__global__ void kernel_sum_with_axis(const int32_t *x, int32_t *y, const int32_t *realAxis,
    const int64_t *xshape, const int64_t *yshape, const int32_t axis_ndim, const uint64_t *every_xdim_size,
    const int32_t xndim, const int32_t yndim, const int64_t ysize, const int32_t* flag, const int64_t axis_size){
  int32_t tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(uint64_t i =tid; i < ysize; i+= gridDim.x*blockDim.x){
    uint64_t in_i = 0, o_i = i;
    for(int j = yndim-1, xj = xndim-1; j>=0; j--){
      uint64_t col = o_i % yshape[j];
      o_i /= yshape[j];
      while(xj >= 0 && flag[xj--] == 1);
      in_i += col * every_xdim_size[xj+1];
    }
    //int32_t max = x[in_i];
    int32_t sum = 0;
    for(uint64_t xi = 0; xi < axis_size; xi++){
      uint64_t o_i = xi, tmp_in_i = 0;
      for(int j = axis_ndim - 1; j>=0; j--){
        uint64_t col = o_i % xshape[realAxis[j]];
        o_i /= xshape[realAxis[j]];
        tmp_in_i += col * every_xdim_size[realAxis[j]];
      }
      //if(max < x[in_i+tmp_in_i]) max = x[in_i+tmp_in_i];
      sum += x[in_i + tmp_in_i];
    }
    y[i] = sum;
  }
}
const char* cuda_sum(const int32_t *x, int32_t *y, const uint64_t xsize, const uint64_t ysize,
    const int64_t *xshape, const int64_t *yshape, const int32_t* realAxis, const int32_t* flag,
    const uint64_t *every_xdim_size, const int64_t axis_size,
    const int32_t xndim, const int32_t yndim, const int32_t axis_ndim){
  int64_t *dev_xshape = NULL, *dev_yshape = NULL;
  uint64_t *dev_every_xdim_size = NULL;
  int32_t *dev_flag = NULL, *dev_axis = NULL;
  if(axis_ndim == 0){
    kernel_sum<<<1, 256>>>(x, y, xsize);
  }else{
    hipError_t status;
    status = hipMalloc((void**)&dev_xshape, sizeof(int64_t)*xndim);
    if(status != hipSuccess){
      goto end;
    }
    status = hipMalloc((void**)&dev_yshape, sizeof(int64_t)*yndim);
    if(status != hipSuccess){
      goto end;
    }
    status = hipMalloc((void**)&dev_axis, sizeof(int32_t) * axis_ndim);
    if(status != hipSuccess){
      goto end;
    }
    status = hipMalloc((void**)&dev_every_xdim_size, sizeof(uint64_t) * xndim);
    if(status != hipSuccess){
      goto end;
    }
    status = hipMalloc((void**)&dev_flag, sizeof(int32_t)*xndim);
    if(status != hipSuccess){
      goto end;
    }
    status = hipMemcpy(dev_xshape, xshape, sizeof(int64_t)*xndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      goto end;
    }
    status = hipMemcpy(dev_yshape, yshape, sizeof(int64_t)*yndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      goto end;
    }
    status = hipMemcpy(dev_axis, realAxis, sizeof(int32_t)*axis_ndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      goto end;
    }
    status = hipMemcpy(dev_every_xdim_size, every_xdim_size, sizeof(uint64_t) * xndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      goto end;
    }
    status = hipMemcpy(dev_flag, flag, sizeof(int32_t)*xndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      goto end;
    }

    int bSize = 256;
    int gSize = getGridSize(ysize, bSize);//(ysize + bSize - 1) / bSize;
    kernel_sum_with_axis<<<gSize, bSize>>>(x, y, dev_axis, dev_xshape, dev_yshape, axis_ndim, 
        dev_every_xdim_size, xndim, yndim, ysize, dev_flag, axis_size);
    goto end;
  }
  print_to_file(y, ysize, "/tmp/zkh/cuda_max.txt");
end:
  if(dev_xshape != NULL) hipFree(dev_xshape);
  if(dev_yshape != NULL) hipFree(dev_yshape);
  if(dev_axis != NULL) hipFree(dev_axis);
  if(dev_every_xdim_size != NULL) hipFree(dev_every_xdim_size);
  if(dev_flag != NULL) hipFree(dev_flag);
  return check_cuda_error(hipGetLastError());
}

const char* cuda_reshape(const int32_t *x, int32_t *y, uint64_t n, bool debug){
  if(x == y) return NULL;
  if(debug)
    memcpy(y, x, n * sizeof(int32_t));
  else
    hipMemcpy(y, x, n*sizeof(int32_t), hipMemcpyDeviceToDevice);
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_log(const int32_t *x, int32_t *y){
  for(int i = 0; i < 64; i++){
    int64_t tmp = (int64_t)1 << i;
    if(x[0] <= tmp){
      y[0] = i;
      return;
    }
  }
  y[0] = 64;
}
const char* cuda_log(const int32_t *x, int32_t *y, const bool debug){
  const int32_t *dev_x = x;
  int32_t *tmp_x, *dev_y = y;
  if(debug){
    hipMalloc((void**)&tmp_x, sizeof(int32_t));
    dev_x = tmp_x;
    hipMemcpy(tmp_x, x, sizeof(int32_t), hipMemcpyHostToDevice);
  }

  int h_x;
  hipMemcpy(&h_x, dev_x, sizeof(int32_t), hipMemcpyDeviceToHost);
  if(h_x <= 0) return "error: log2 a no positive value";

  kernel_log<<<1,1>>>(dev_x, dev_y);

  if(debug){
    hipMemcpy(y, dev_y, sizeof(int32_t), hipMemcpyDeviceToHost);
    hipFree(tmp_x);
    hipFree(dev_y);
  }

  return check_cuda_error(hipGetLastError());
}
__global__ void kernel_abs(const int32_t *x, int32_t *y, const uint64_t n){
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(uint64_t i = tid; i < n; i += gridDim.x*blockDim.x){
    y[i] = abs(x[i]);
  }
}
const char* cuda_abs(const int32_t *x, int32_t *y, const uint64_t n, bool debug){
  const int32_t *dev_x = x;
  int32_t *tmp_x, *dev_y = y;
  if(debug){
    hipMalloc((void**)&tmp_x, sizeof(int32_t) * n);
    dev_x = tmp_x;
    hipMalloc((void**)&dev_y, sizeof(int32_t) * n);
    hipMemcpy(tmp_x, x, sizeof(int32_t) * n, hipMemcpyHostToDevice);
  }
  int bSize = 256;
  int gSize = getGridSize(n, bSize);//(n + bSize - 1) / bSize;
  kernel_abs<<<gSize, bSize>>>(dev_x, dev_y, n);
  if(debug){
    hipMemcpy(y, dev_y, sizeof(int32_t) * n, hipMemcpyDeviceToHost);
    hipFree(tmp_x);
    hipFree(dev_y);
  }
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_max(const int32_t *x, int32_t *y, int64_t n){
  __shared__ int32_t buf[256];
  int32_t tid = threadIdx.x;
  int32_t maxValue = (int32_t)1 << 31;
  for (int i = tid; i < n; i += blockDim.x){
    int32_t tmp = x[i];
    if(maxValue < tmp) maxValue = tmp;
  }

  buf[tid] = maxValue;
  __syncthreads();
  for(int s = 1; s < blockDim.x; s*=2){
    if((tid % (2*s)) == 0){
      int a = buf[tid];
      int b = buf[tid+s];
      buf[tid] = a > b ? a : b;
    }
    __syncthreads();
  }

  if(tid == 0) y[0] = buf[0];
}

__global__ void kernel_max_with_axis(const int32_t *x, int32_t *y, const int32_t *realAxis,
    const int64_t *xshape, const int64_t *yshape, const int32_t axis_ndim, const uint64_t *every_xdim_size,
    const int32_t xndim, const int32_t yndim, const int64_t ysize, const int32_t* flag, const int64_t axis_size){
  int32_t tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(uint64_t i =tid; i < ysize; i+= gridDim.x*blockDim.x){
    uint64_t in_i = 0, o_i = i;
    for(int j = yndim-1, xj = xndim-1; j>=0; j--){
      uint64_t col = o_i % yshape[j];
      o_i /= yshape[j];
      while(xj >= 0 && flag[xj--] == 1);
      in_i += col * every_xdim_size[xj+1];
    }
    int32_t max = x[in_i];
    for(uint64_t xi = 0; xi < axis_size; xi++){
      uint64_t o_i = xi, tmp_in_i = 0;
      for(int j = axis_ndim - 1; j>=0; j--){
        uint64_t col = o_i % xshape[realAxis[j]];
        o_i /= xshape[realAxis[j]];
        tmp_in_i += col * every_xdim_size[realAxis[j]];
      }
      if(max < x[in_i+tmp_in_i]) max = x[in_i+tmp_in_i];
    }
    y[i] = max;
  }
}
const char* cuda_max(const int32_t *x, int32_t *y, const uint64_t xsize, const uint64_t ysize,
    const int64_t *xshape, const int64_t *yshape, const int32_t* realAxis, const int32_t* flag, 
    const uint64_t *every_xdim_size, const int64_t axis_size,
    const int32_t xndim, const int32_t yndim, const int32_t axis_ndim){
  int64_t *dev_xshape = NULL, *dev_yshape = NULL;
  uint64_t *dev_every_xdim_size = NULL;
  int32_t *dev_flag = NULL, *dev_axis = NULL;
  if(axis_ndim == 0){
    kernel_max<<<1, 256>>>(x, y, xsize);
  }else{
    hipError_t status;
    status = hipMalloc((void**)&dev_xshape, sizeof(int64_t)*xndim);
    if(status != hipSuccess){
      goto end;
    }
    status = hipMalloc((void**)&dev_yshape, sizeof(int64_t)*yndim);
    if(status != hipSuccess){
      goto end;
    }
    status = hipMalloc((void**)&dev_axis, sizeof(int32_t) * axis_ndim);
    if(status != hipSuccess){
      goto end;
    }
    status = hipMalloc((void**)&dev_every_xdim_size, sizeof(uint64_t) * xndim);
    if(status != hipSuccess){
      goto end;
    }
    status = hipMalloc((void**)&dev_flag, sizeof(int32_t)*xndim);
    if(status != hipSuccess){
      goto end;
    }
    status = hipMemcpy(dev_xshape, xshape, sizeof(int64_t)*xndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      goto end;
    }
    status = hipMemcpy(dev_yshape, yshape, sizeof(int64_t)*yndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      goto end;
    }
    status = hipMemcpy(dev_axis, realAxis, sizeof(int32_t)*axis_ndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      goto end;
    }
    status = hipMemcpy(dev_every_xdim_size, every_xdim_size, sizeof(uint64_t) * xndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      goto end;
    }
    status = hipMemcpy(dev_flag, flag, sizeof(int32_t)*xndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      goto end;
    }

    int bSize = 256;
    int gSize = getGridSize(ysize, bSize);//(ysize + bSize - 1) / bSize;
    kernel_max_with_axis<<<gSize, bSize>>>(x, y, dev_axis, dev_xshape, dev_yshape, axis_ndim, 
        dev_every_xdim_size, xndim, yndim, ysize, dev_flag, axis_size);
    goto end;
  }
  print_to_file(y, ysize, "/tmp/zkh/cuda_max.txt");
end:
  if(dev_xshape != NULL) hipFree(dev_xshape);
  if(dev_yshape != NULL) hipFree(dev_yshape);
  if(dev_axis != NULL) hipFree(dev_axis);
  if(dev_every_xdim_size != NULL) hipFree(dev_every_xdim_size);
  if(dev_flag != NULL) hipFree(dev_flag);
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_cvm_clip(const int32_t *x, const int32_t precision, int32_t *y, const uint64_t n){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int minV = -((1 << (precision - 1)) - 1);
  int maxV = -minV;
  for(uint64_t i = tid; i < n; i += gridDim.x*blockDim.x){
    y[i] = max(min(x[i], maxV), minV);
  }
}
const char* cuda_cvm_clip(const int32_t* x, const int32_t precision, int32_t *y, const uint64_t n, bool debug){
  const int32_t *dev_x = x;
  int32_t *tmp_x, *dev_y = y;
  if(debug){
    hipMalloc((void**)&tmp_x, sizeof(int32_t) * n);
    hipMalloc((void**)&dev_y, sizeof(int32_t) * n);
    hipMemcpy(tmp_x, x, sizeof(int32_t) * n, hipMemcpyHostToDevice);
    dev_x = tmp_x;
  }
  int bSize = 256;
  int gSize = getGridSize(n, bSize); //(n + bSize - 1) / bSize;
  kernel_cvm_clip<<<gSize, bSize>>>(dev_x, precision, dev_y, n);
  if(debug){
    hipMemcpy(y, dev_y, sizeof(int32_t) * n, hipMemcpyDeviceToHost);
    hipFree(dev_y);
    hipFree(tmp_x);
  }

  print_to_file(y, n, "/tmp/zkh/cuda_cvm_clip.txt");
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_cvm_right_shift(const int32_t *a, const int32_t b, const int32_t precision, int32_t *c, const uint64_t n){
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int minV = -((1 << (precision - 1)) - 1);
  int maxV = -minV;
  for(uint64_t i = tid; i < n; i+= gridDim.x*blockDim.x){
    int shift_a = a[i];
    if(b == 0) c[i] = shift_a;
    else {
      shift_a = ((shift_a >> (b - 1)) + 1 ) >> 1;
      c[i] = max(min(shift_a, maxV), minV);
    } 
  }
}
const char* cuda_cvm_right_shift(const int32_t *a, const int32_t b, const int32_t precision, int32_t *c, const uint64_t n, bool debug){
  const int32_t *dev_a = a;
  int32_t *tmp_a, *dev_c = c;
  if(debug){
    hipMalloc((void**)&tmp_a, sizeof(int32_t) * n);
    hipMalloc((void**)&dev_c, sizeof(int32_t) * n);
    hipMemcpy(tmp_a, a, sizeof(int32_t) * n, hipMemcpyHostToDevice);
    dev_a = tmp_a;
  }

  int bSize = 256;
  int gSize = getGridSize(n, bSize); //(n + bSize - 1) / bSize;
  kernel_cvm_right_shift<<<gSize, bSize>>>(dev_a, b, precision, dev_c, n);
  if(debug){
    hipMemcpy(c, dev_c, sizeof(int32_t) * n, hipMemcpyDeviceToHost);
    hipFree(dev_c);
    hipFree(tmp_a);
  }
  print_to_file(dev_c, n, "/tmp/zkh/cuda_cvm_right_shft.txt");
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_cvm_left_shift(const int32_t *a, const int32_t b, const int32_t precision, int32_t *c, const uint64_t n){
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int minV = -((1 << (precision - 1)) - 1);
  int maxV = -minV;
  for(uint64_t i = tid; i < n; i += gridDim.x*blockDim.x){
    int shift_a = a[i];
    if(b == 0) c[i] = shift_a;
    else {
      shift_a = shift_a << b;
      c[i] = max(min(shift_a, maxV), minV);
    } 
  }
}
const char* cuda_cvm_left_shift(const int32_t *a, const int32_t b, const int32_t precision, int32_t *c, const uint64_t n, bool debug){
  const int32_t *dev_a = a;
  int32_t *tmp_a, *dev_c = c;
  if(debug){
    hipMalloc((void**)&tmp_a, sizeof(int32_t) * n);
    hipMalloc((void**)&dev_c, sizeof(int32_t) * n);
    hipMemcpy(tmp_a, a, sizeof(int32_t) * n, hipMemcpyHostToDevice);
    dev_a = tmp_a;
  }

  int bSize = 256;
  int gSize = getGridSize(n, bSize);//(n + bSize - 1) / bSize;
  kernel_cvm_left_shift<<<gSize, bSize>>>(dev_a, b, precision, dev_c, n);
  if(debug){
    hipMemcpy(c, dev_c, sizeof(int32_t) * n, hipMemcpyDeviceToHost);
    hipFree(dev_c);
    hipFree(tmp_a);
  }
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_concatenate(const int32_t *input, const int64_t *ishape, int32_t *output, 
    int64_t* oshape, const int32_t odim, const int64_t n,  
    const int64_t preShapeSize, const int64_t curShapeSize, const int32_t axis){
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(uint64_t i = tid; i < n; i += gridDim.x*blockDim.x){
    uint64_t o_i = i, in_i2 = 0, shapeSize = 0;
    bool flag = true;
    for(int j = odim-1; j >= 0; j--){
      uint64_t col = o_i % oshape[j];
      o_i /= oshape[j];
      uint64_t tmpcol = col;
      if(j == axis){
        if(col >= preShapeSize && col < curShapeSize) {
          tmpcol = col - preShapeSize;
        }else{
          flag = false;
          break;
        }
      }
      in_i2 += (j == odim-1 ? tmpcol : tmpcol * shapeSize);
      shapeSize = (j == odim-1 ? ishape[j] : shapeSize * ishape[j]);
    }
    if(flag)
    output[i] = input[in_i2];
  }
}
const char* cuda_concatenate(const int32_t *input, const int64_t *ishape, const int32_t idim, const uint64_t in, 
    int32_t *output, int64_t* oshape, const int32_t odim, const uint64_t on,  
    const int64_t preShapeSize, const int64_t curShapeSize, const int32_t axis, bool debug){
  const int32_t *dev_input = input;
  int32_t *tmp_input, *dev_output = output;
  if(debug){
    hipMalloc((void**)&tmp_input, sizeof(int32_t) * in);
    hipMalloc((void**)&dev_output, sizeof(int32_t) * on);
    hipMemcpy(tmp_input, input, sizeof(int32_t) * in, hipMemcpyHostToDevice);
    dev_input = tmp_input;
  }

  int64_t* dev_ishape, *dev_oshape;
  hipError_t status;
  status = hipMalloc((void**)&dev_ishape, sizeof(int64_t) * idim);
  if(status != hipSuccess){
    return check_cuda_error(status);
  }
  status = hipMalloc((void**)&dev_oshape, sizeof(int64_t) * odim);
  if(status != hipSuccess){
    hipFree(dev_ishape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_ishape, ishape, sizeof(int64_t)*idim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_ishape);
    hipFree(dev_oshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_oshape, oshape, sizeof(int64_t)*odim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_ishape);
    hipFree(dev_oshape);
    return check_cuda_error(status);
  }
  int bSize = 256;
  int gSize = getGridSize(on, bSize);//(on + bSize - 1) / bSize;
  kernel_concatenate<<<gSize, bSize>>>(dev_input, dev_ishape, dev_output, dev_oshape, odim, on,
      preShapeSize, curShapeSize, axis);
  hipDeviceSynchronize();

  hipFree(dev_ishape);
  hipFree(dev_oshape);

  if(debug){
    hipMemcpy(output, dev_output, sizeof(int32_t) * on, hipMemcpyDeviceToHost);
    hipFree(tmp_input);
    hipFree(dev_output);
  }
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_bias_add(const int32_t *x_data, const int32_t * bias_data, int32_t *y_data, 
    int64_t ysize, const int64_t *yshape, const int32_t ndim, const int32_t axis){
  int32_t i = threadIdx.x + blockDim.x * blockIdx.x;
  if(i < ysize){
    int32_t bV = 0;
    for(int32_t j = ndim - 1; j >= 0; j--){
      if(j == axis){
        bV = bias_data[axis];
        break;
      }
    }
    y_data[i] = x_data[i] + bV;
  }
}
const char* cuda_bias_add(const int32_t *x_data, const int32_t * bias_data, int32_t *y_data, 
    int64_t ysize, const int64_t *yshape, const int32_t ndim, const int32_t axis){
  int64_t *dev_yshape;
  hipMalloc((void**)&dev_yshape, sizeof(int64_t) * ndim);
  hipMemcpy(dev_yshape, yshape, sizeof(int64_t) * ndim, hipMemcpyHostToDevice);

  int bSize = 256;
  int gSize = (ysize + bSize - 1) / bSize;
  kernel_bias_add<<<gSize, bSize>>>(x_data, bias_data, y_data, ysize, dev_yshape, ndim, axis);

  hipFree(dev_yshape);
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_repeat(const int32_t *x_data, int32_t *y_data, const int64_t *xshape,
    const int64_t *yshape, const uint64_t ysize, const int32_t ndim, const int32_t axis, 
    const int32_t repeat){
  int32_t tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(uint64_t i = tid; i < ysize; i+=gridDim.x*blockDim.x){
    uint64_t o_i = i, in_i = 0, shapeSize = 0;
    for(int j = ndim-1; j >= 0; j--){
      uint64_t col = o_i % yshape[j];
      o_i /= yshape[j];
      if(j == axis) col = col / repeat;
      in_i += (j == ndim-1 ? col : col * shapeSize);
      shapeSize = (j == ndim-1 ? xshape[j] : shapeSize * xshape[j]);
    }
    y_data[i] = x_data[in_i];
  }
}
const char* cuda_repeat(const int32_t *x_data, int32_t *y_data, const int64_t *xshape,
    const int64_t *yshape, const uint64_t ysize, const int32_t xndim, const int32_t yndim, 
    const int32_t axis, const int32_t repeat){
  int64_t *dev_xshape, *dev_yshape;
  hipMalloc((void**)&dev_xshape, sizeof(int64_t) * xndim);
  hipMalloc((void**)&dev_yshape, sizeof(int64_t) * yndim);
  hipMemcpy(dev_xshape, xshape, sizeof(int64_t) * xndim, hipMemcpyHostToDevice);
  hipMemcpy(dev_xshape, xshape, sizeof(int64_t) * xndim, hipMemcpyHostToDevice);
  hipMemcpy(dev_yshape, yshape, sizeof(int64_t) * yndim, hipMemcpyHostToDevice);

  int bSize = 256;
  int gSize = getGridSize(ysize, bSize);//(ysize + bSize - 1) / bSize;
  kernel_repeat<<<gSize, bSize>>>(x_data, y_data, dev_xshape, dev_yshape, ysize, yndim, axis, repeat);

  hipFree(dev_xshape);
  hipFree(dev_yshape);
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_upsampling_nearest(const int32_t *x_data, int32_t *y_data, const uint32_t scale, const uint32_t ih, const uint32_t iw,
    const uint32_t oh, const uint32_t ow, const uint32_t channel){
  int ox = threadIdx.x;
  int oy = threadIdx.y;
  
  for(int b = blockIdx.x; b < channel; b+=gridDim.x){
    for(int r = oy; r < oh; r += blockDim.y){
      for(int c = ox; c < ow; c += blockDim.x){
        y_data[b * oh * ow + r * ow + c] = x_data[b * ih * iw + r/scale * iw + c/scale];
      }
    }
  }
}

const char* cuda_upsampling_nearest(const int32_t *x_data, int32_t *y_data, const uint32_t scale, const int32_t ih, const int32_t iw, 
    const uint32_t oh, const uint32_t ow, const uint32_t batch, const uint32_t channel){
  dim3 block(1, 32, 32);
  int grid = channel > 4096 ? 4096 : channel;

  for(int i = 0; i < batch; i++){
    kernel_upsampling_nearest<<<grid, block>>>(x_data + i*channel*ih*iw, 
        y_data + i*channel*oh*ow, 
        scale, ih, iw, oh, ow, channel);
  }
  print_to_file(y_data, batch * channel * oh * ow, "/tmp/zkh/cuda_upsampliing.txt");
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_negative(const int32_t *x_data, int32_t *y_data, uint64_t n){
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(uint64_t i = tid; i < n; i += gridDim.x*blockDim.x){
    y_data[i] = -x_data[i];
  }
}
const char* cuda_negative(const int32_t *x_data, int32_t *y_data, uint64_t n){
  int threadSize = 256;
  int blockSize = getGridSize(n, threadSize);//(n + threadSize - 1) / threadSize;
  kernel_negative<<<blockSize, threadSize>>>(x_data, y_data, n);
  return check_cuda_error(hipGetLastError());
}


__global__ void kernel_tile(const int32_t *x_data, int32_t *y_data, const uint64_t ysize, const int32_t yndim, const int32_t xndim,
    const int64_t *xshape, const int64_t *yshape){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for(uint64_t i = tid; i < ysize; i+=gridDim.x*blockDim.x){
    uint64_t o_i = i, in_i = 0, shapeSize = 1;
    for(int j = xndim-1; j >= 0; j--){
      int yj = j + yndim - xndim;
      int col = o_i % yshape[yj];
      o_i /= yshape[yj];
      col = col % xshape[j];
      in_i += col * shapeSize;
      shapeSize = shapeSize * xshape[j];
    }
    y_data[i] = x_data[in_i];
  }
}
const char* cuda_tile(const int32_t *x_data, int32_t *y_data, const uint64_t ysize, const int32_t yndim, const int32_t xndim,
    const int64_t *xshape, const int64_t *yshape){
  uint64_t tmp_y_size = 1;
  for(int i = 0; i < xndim; i++){
    tmp_y_size *= yshape[i + yndim - xndim];
  }

  int64_t *dev_xshape, *dev_yshape;
  hipError_t status;
  status = hipMalloc((void**)&dev_xshape, sizeof(int64_t) * xndim);
  if(status != hipSuccess){
    return check_cuda_error(status);
  }
  status = hipMalloc((void**)&dev_yshape, sizeof(int64_t) * yndim);
  if(status != hipSuccess){
    hipFree(dev_xshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_xshape, xshape, sizeof(int64_t) * xndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_xshape);
    hipFree(dev_yshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_yshape, yshape, sizeof(int64_t) * yndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_xshape);
    hipFree(dev_yshape);
    return check_cuda_error(status);
  }

  int threadSize = 256;
  int blockSize = getGridSize(tmp_y_size, threadSize);//(tmp_y_size + threadSize - 1) / threadSize;
  kernel_tile<<<blockSize, threadSize>>>(x_data, y_data, tmp_y_size, yndim, xndim, dev_xshape, dev_yshape);

  uint64_t othery = 1;
  for(int i = 0; i < yndim-xndim; i++){
    othery *= yshape[i];
  }
  for(size_t i = 1; i < othery; i++){
    status = hipMemcpy(y_data + i*tmp_y_size, y_data, tmp_y_size * sizeof(int32_t), hipMemcpyDeviceToDevice);
    if(status != hipSuccess){
      hipFree(dev_xshape);
      hipFree(dev_yshape);
      return check_cuda_error(status);
    }
  }
  hipFree(dev_xshape);
  hipFree(dev_yshape);
  return check_cuda_error(hipGetLastError());
}

const char *cuda_expand_dims(const int32_t *ishape_data, int32_t *oshape_data, const int32_t axis, const uint64_t n){
  if(oshape_data == ishape_data){
    return NULL;
  }
  hipMemcpy(oshape_data, ishape_data, sizeof(int32_t) * n, hipMemcpyDeviceToDevice);
  print_to_file(oshape_data, n, "/tmp/zkh/cuda_expand_dims.txt");
  return check_cuda_error(hipGetLastError());
}

const char *cuda_squeeze(const int32_t *ishape_data, int32_t *oshape_data, const uint64_t n){
  if(oshape_data == ishape_data){
    return NULL;
  }
  hipMemcpy(oshape_data, ishape_data, sizeof(int32_t) * n, hipMemcpyDeviceToDevice);
  print_to_file(oshape_data, n, "/tmp/zkh/cuda_squeeze.txt");
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_transpose(const int32_t *x_data, const int64_t *axes_data, int32_t *y_data, 
    const int64_t *xshape, const int64_t *yshape, const int32_t ndim, const int64_t ysize, 
    const int32_t axes_ndim){
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(uint64_t i = tid; i < ysize; i+=gridDim.x*blockDim.x){
    uint64_t in_i = 0, o_i = i;
    for(int j = ndim-1; j >= 0; j--){
      uint64_t col = o_i % yshape[j];
      o_i /= yshape[j];
      int xj = j;
      if(axes_ndim > 0){
        xj = axes_data[j];
      }else{
        xj = ndim - 1 - j;
      }
      int xi = 1;
      for(int tx = ndim-1; tx > xj; tx--){
        xi *= xshape[tx];
      }
      in_i += col * xi;
    }
    y_data[i] = x_data[in_i];
  }
}
const char* cuda_transpose(const int32_t *x_data, const int64_t *axes_data, int32_t *y_data, 
    const int64_t *xshape, const int64_t *yshape, const int32_t ndim, const uint64_t ysize,
    const int32_t axes_ndim){
  int64_t *dev_xshape, *dev_yshape, *dev_axes;
  hipError_t status;
  status = hipMalloc((void**)&dev_xshape, sizeof(int64_t) * ndim);
  if(status != hipSuccess){
    return check_cuda_error(status);
  }
  status = hipMalloc((void**)&dev_yshape, sizeof(int64_t) * ndim);
  if(status != hipSuccess){
    hipFree(dev_xshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_xshape, xshape, sizeof(int64_t) * ndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_xshape);
    hipFree(dev_yshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_yshape, yshape, sizeof(int64_t) * ndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_xshape);
    hipFree(dev_yshape);
    return check_cuda_error(status);
  }
  if(axes_ndim > 0){
    status = hipMalloc((void**)&dev_axes, sizeof(int64_t) * axes_ndim);
    if(status != hipSuccess){
      hipFree(dev_xshape);
      hipFree(dev_yshape);
      return check_cuda_error(status);
    }
    status = hipMemcpy(dev_axes, axes_data, sizeof(int64_t) * axes_ndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      hipFree(dev_xshape);
      hipFree(dev_yshape);
      hipFree(dev_axes);
      return check_cuda_error(status);
    }
  }

  int threadSize = 256;
  int blockSize = getGridSize(ysize, threadSize);//(ysize + threadSize - 1) / threadSize;
  kernel_transpose<<<blockSize, threadSize>>>(x_data, dev_axes, y_data, dev_xshape, dev_yshape, ndim, ysize, axes_ndim);
  hipFree(dev_xshape);
  hipFree(dev_yshape);
  if(axes_ndim > 0){
    hipFree(dev_axes);
  }
  print_to_file(y_data, ysize, "/tmp/zkh/cuda_transpose.txt");
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_stride_slice(const int32_t *x_data, int32_t *y_data, const int64_t *begin_data,
    const int32_t begin_ndim, const int64_t *step_data, const int64_t *xshape, const int64_t *yshape, 
    const int32_t step_ndim, const int32_t y_ndim, const uint64_t ysize){
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(uint64_t i = tid; i < ysize; i += gridDim.x*blockDim.x){
    uint64_t o_i = i, in_i = 0, shapeSize = 0;
    for(int j = y_ndim-1; j >= 0; j--){
      uint64_t col = o_i % yshape[j];
      o_i /= yshape[j];
      int64_t begin = begin_ndim > j ? begin_data[j] : 0;
      int64_t step = step_ndim > j ? step_data[j] : 1;
      col = begin + col * step;
      in_i += (j == y_ndim-1 ? col : col * shapeSize);
      shapeSize = (j == y_ndim-1 ? xshape[j] : shapeSize * xshape[j]);
    }
    y_data[i] = x_data[in_i];
  }
}
const char* cuda_stride_slice(const int32_t *x_data, int32_t *y_data, const int64_t *begin_data,
    const int32_t begin_ndim, const int64_t *step_data, const int64_t *xshape, const int64_t *yshape, 
    const int32_t step_ndim, const int32_t y_ndim, const uint64_t ysize, const int32_t x_ndim){
  int64_t *dev_xshape, *dev_yshape, *dev_begin, *dev_step;
  hipError_t status;
  status = hipMalloc((void**)&dev_xshape, sizeof(int64_t) * x_ndim);
  if(status != hipSuccess){
    return check_cuda_error(status);
  }
  status = hipMalloc((void**)&dev_yshape, sizeof(int64_t) * y_ndim);
  if(status != hipSuccess){
    hipFree(dev_xshape);
    return check_cuda_error(status);
  }
  status = hipMalloc((void**)&dev_begin, sizeof(int64_t) * begin_ndim);
  if(status != hipSuccess){
    hipFree(dev_xshape);
    hipFree(dev_yshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_xshape, xshape, sizeof(int64_t) * x_ndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_xshape);
    hipFree(dev_yshape);
    hipFree(dev_begin);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_yshape, yshape, sizeof(int64_t) * y_ndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_xshape);
    hipFree(dev_yshape);
    hipFree(dev_begin);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_begin, begin_data, sizeof(int64_t) * begin_ndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_xshape);
    hipFree(dev_yshape);
    hipFree(dev_begin);
    return check_cuda_error(status);
  }
  if(step_ndim > 0){
    status = hipMalloc((void**)&dev_step, sizeof(int64_t) * step_ndim);
    if(status != hipSuccess){
      hipFree(dev_xshape);
      hipFree(dev_yshape);
      hipFree(dev_begin);
      return check_cuda_error(status);
    }
    status = hipMemcpy(dev_step, step_data, sizeof(int64_t) * step_ndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      hipFree(dev_xshape);
      hipFree(dev_yshape);
      hipFree(dev_begin);
      hipFree(dev_step);
      return check_cuda_error(status);
    }
  }

  int threadSize = 256;
  int blockSize = getGridSize(ysize, threadSize);
  kernel_stride_slice<<<blockSize, threadSize>>>(x_data,  y_data, dev_begin, begin_ndim, dev_step, 
      dev_xshape, dev_yshape, step_ndim, y_ndim, ysize);
  hipFree(dev_xshape);
  hipFree(dev_yshape);
  hipFree(dev_begin);
  if(step_ndim > 0){
    hipFree(dev_step);
  }
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_slice_like(const int32_t *x_data, int32_t *y_data, const int64_t *xshape, const int64_t *yshape,
    const uint64_t ysize, const int32_t ndim){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for(uint64_t i = tid; i < ysize; i+=gridDim.x*blockDim.x){
    uint64_t o_i = i, in_i = 0, shapeSize = 1;
    for(int j = ndim-1; j >= 0; j--){
      int col = o_i % yshape[j];
      o_i /= yshape[j];
      in_i +=  col * shapeSize;
      shapeSize = shapeSize * xshape[j];
    }
    y_data[i] = x_data[in_i];
  }
}
const char* cuda_slice_like(const int32_t *x_data, int32_t *y_data, const int64_t *xshape, const int64_t *yshape,
    const uint64_t ysize, const int32_t ndim){
  int64_t *dev_xshape, *dev_yshape;
  hipError_t status;
  status = hipMalloc((void**)&dev_xshape, sizeof(int64_t) * ndim);
  if(status != hipSuccess){
    return check_cuda_error(status);
  }
  status = hipMalloc((void**)&dev_yshape, sizeof(int64_t) * ndim);
  if(status != hipSuccess){
    hipFree(dev_xshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_xshape, xshape, sizeof(int64_t) * ndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_xshape);
    hipFree(dev_yshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_yshape, yshape, sizeof(int64_t) * ndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_xshape);
    hipFree(dev_yshape);
    return check_cuda_error(status);
  }

  int threadSize = 256;
  int blockSize = getGridSize(ysize, threadSize);//(ysize + threadSize - 1) / threadSize;
  kernel_slice_like<<<blockSize, threadSize>>>(x_data, y_data, dev_xshape, dev_yshape, ysize, ndim);

  hipFree(dev_xshape);
  hipFree(dev_yshape);
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_get_valid_count(const int32_t *input, bool *saved, const int32_t n, const int32_t k, const int32_t score_threshold){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for(uint64_t j = tid; j < n; j+=gridDim.x*blockDim.x){
    const int32_t *row = input + j * k;
    saved[j] = row[1] > score_threshold ? 1 : 0;
  }
}
const char* cuda_get_valid_counts(const int32_t *x_data, int32_t *y_data, int32_t *valid_count_data,
    const int32_t n, const int32_t k,
    const int32_t score_threshold, const int32_t batchs){

  int32_t *host_count = (int32_t*)malloc(sizeof(int32_t) * batchs);//new int32_t[batchs];
  if(host_count == NULL){
    return "malloc error";
  }
  bool* saved = (bool*)malloc(sizeof(bool) * n);
  if(saved == NULL){
    free(host_count);
    return "malloc error";
  }
  bool *dev_saved;
  hipError_t status;
  status = hipMalloc((void**)&dev_saved, sizeof(bool)*n);
  if(status != hipSuccess){
    free(saved);
    return check_cuda_error(status);
  }

  for(int32_t i = 0; i < batchs; i++){
    int32_t y_index = 0;
    const int32_t *input = x_data + i * n * k;
    int32_t *output = y_data + i * n * k;

    int threadSize = 256;
    int blockSize = (n + threadSize - 1) / threadSize;
    kernel_get_valid_count<<<blockSize, threadSize>>>(input, dev_saved, n, k, score_threshold);
    status = hipMemcpy(saved, dev_saved, sizeof(bool) * n, hipMemcpyDeviceToHost);
    if(status != hipSuccess){
      free(host_count);
      free(saved);
      hipFree(dev_saved);
      return check_cuda_error(status);
    }

    for(int32_t j = 0; j < n; j++){
      const int32_t *row = input + j * k;
      if(saved[j]){
        status = hipMemcpy(&output[y_index * k], row, k * sizeof(int32_t), hipMemcpyDeviceToDevice);
        if(status != hipSuccess){
          free(host_count);
          free(saved);
          hipFree(dev_saved);
          return check_cuda_error(status);
        }
        y_index += 1;
      }
    }
    host_count[i] = y_index;
    //valid_count_data[i] = y_index;
    if(y_index < n){
      status = hipMemset(&output[y_index * k], -1, (n-y_index) * k * sizeof(int32_t));
      if(status != hipSuccess){
        free(host_count);
        free(saved);
        hipFree(dev_saved);
        return check_cuda_error(status);
      }
    }
  }
  hipMemcpy(valid_count_data, host_count, sizeof(int32_t) * batchs, hipMemcpyHostToDevice);
  hipFree(dev_saved);
  free(saved);
  free(host_count);

  /*
     int32_t *h_x = new int32_t[batchs * n * k];
     int32_t *h_vc = new int32_t[batchs];
     int32_t *h_y = new int32_t[batchs * n * k];
     hipMemcpy(h_x, x_data, batchs*n*k*sizeof(int32_t), hipMemcpyDeviceToHost);
     get_valid_count(h_x, h_y, h_vc, batchs, n, k, score_threshold);
     hipMemcpy(y_data, h_y, batchs*n*k*sizeof(int32_t), hipMemcpyHostToDevice);
     hipMemcpy(valid_count_data, h_vc, batchs*sizeof(int32_t), hipMemcpyHostToDevice);
     delete h_x;
     delete h_vc;
     delete h_y;
   */
  return check_cuda_error(hipGetLastError());
}

const char *cuda_non_max_suppression(int32_t *d_x_data, const int32_t *d_valid_count_data, int32_t *d_y_data, const int32_t batchs, const int32_t n, const int32_t k,
    const int32_t max_output_size, const int32_t iou_threshold, const int32_t topk, 
    const int32_t coord_start, const int32_t score_index, const int32_t id_index, const bool force_suppress){
  int32_t *x_data = NULL, *valid_count_data = NULL, *y_data = NULL;
  x_data = (int32_t*)malloc(sizeof(int32_t) * batchs*n*k);//new int32_t[batchs * n * k];
  valid_count_data = (int32_t*)malloc(sizeof(int32_t)*batchs);//new int32_t[batchs];
  y_data = (int32_t*)malloc(sizeof(int32_t) *batchs*n*k);//new int32_t[batchs * n * k];
  if(x_data == NULL || valid_count_data == NULL || y_data == NULL){
    goto end;
  }
  hipError_t status;
  status = hipMemcpy(x_data, d_x_data, batchs*n*k*sizeof(int32_t), hipMemcpyDeviceToHost);
  if(status != hipSuccess){
    goto end;
  }
  status = hipMemcpy(valid_count_data, d_valid_count_data, batchs*sizeof(int32_t), hipMemcpyDeviceToHost);
  if(status != hipSuccess){
    goto end;
  }

  non_max_suppression(
      x_data, valid_count_data, y_data, batchs, n, k,
      max_output_size, iou_threshold, topk, coord_start, score_index, id_index, force_suppress);

  hipMemcpy(d_y_data, y_data, batchs * n * k * sizeof(int32_t), hipMemcpyHostToDevice);
end:
  if(x_data != NULL)
    free(x_data);
  if(valid_count_data != NULL)
    free(valid_count_data);
  if(y_data != NULL)
    free(y_data);
  return check_cuda_error(hipGetLastError());
}


__global__ void kernel_take(const int32_t *x_data, const int32_t *indices_data, int32_t *y_data, 
    const int64_t *xshape, const int64_t *yshape, const int64_t *indices_shape, const int32_t yndim,
    const int32_t xndim, const int32_t indices_ndim, const uint64_t ysize, const int32_t axis){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for(uint64_t i = tid; i < ysize; i += gridDim.x*blockDim.x){
    uint64_t o_i = i, x_i = 0, indices_i = 0, x_shape_size = 0, indices_shape_size = 0;
    for(int32_t j = yndim - 1, k = indices_ndim-1; j>=axis; j--){
      uint64_t col = o_i % yshape[j];
      o_i /= yshape[j];
      if(j < axis + indices_ndim){
        indices_i += (indices_shape_size == 0 ? col : col * indices_shape_size);
        indices_shape_size = (indices_shape_size == 0 ? indices_shape[k]
            : indices_shape_size * indices_shape[k]);
        --k;
      }
    }

    o_i = i;
    int32_t k = xndim - 1;
    for(int32_t j = yndim - 1; j >= axis + indices_ndim; j--, k--){
      uint64_t col = o_i % yshape[j];
      o_i /= yshape[j];
      x_i += (j == yndim-1 ? col : col * x_shape_size);
      x_shape_size = (j == yndim-1 ? xshape[k] : x_shape_size * xshape[k]);
    }

    uint64_t x_indices_i = min(max(indices_data[indices_i], 0), (int32_t)xshape[k]-1);
    x_i += (x_shape_size == 0 ? x_indices_i : x_indices_i * x_shape_size);
    x_shape_size = (x_shape_size == 0 ? xshape[k] : x_shape_size * xshape[k]);
    --k;

    o_i = i;
    for(int32_t j = yndim - 1; j>=0 && k >= 0; j--){
      uint64_t col = o_i % yshape[j];
      o_i /= yshape[j];
      if(j < axis){
        x_i += x_shape_size == 0 ? col : col * x_shape_size;
        x_shape_size = x_shape_size == 0 ? xshape[k] : x_shape_size * xshape[k];
        --k;
      }
    }
    y_data[i] = x_data[x_i];
  }
}
const char* cuda_take(const int32_t *x_data, const int32_t *indices_data, int32_t *y_data, 
    const int64_t *xshape, const int64_t *yshape, const int64_t *indices_shape, const int32_t yndim,
    const int32_t xndim, const int32_t indices_ndim, const uint64_t ysize, const int32_t axis){
  int64_t *dev_xshape, *dev_yshape, *dev_indices_shape;
  hipError_t status;
  status = hipMalloc((void**)&dev_xshape, sizeof(int64_t) * xndim);
  if(status != hipSuccess){
    return check_cuda_error(status);
  }
  status = hipMalloc((void**)&dev_yshape, sizeof(int64_t) * yndim);
  if(status != hipSuccess){
    hipFree(dev_xshape);
    return check_cuda_error(status);
  }
  status = hipMalloc((void**)&dev_indices_shape, sizeof(int64_t) * indices_ndim);
  if(status != hipSuccess){
    hipFree(dev_xshape);
    hipFree(dev_yshape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_xshape, xshape, sizeof(int64_t)*xndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_xshape);
    hipFree(dev_yshape);
    hipFree(dev_indices_shape);
    return check_cuda_error(status);
  }

  status = hipMemcpy(dev_yshape, yshape, sizeof(int64_t)*yndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_xshape);
    hipFree(dev_yshape);
    hipFree(dev_indices_shape);
    return check_cuda_error(status);
  }
  status = hipMemcpy(dev_indices_shape, indices_shape, sizeof(int64_t)*indices_ndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    hipFree(dev_xshape);
    hipFree(dev_yshape);
    hipFree(dev_indices_shape);
    return check_cuda_error(status);
  }

  int threadSize = 256;
  int blockSize = getGridSize(ysize, threadSize);//(ysize + threadSize - 1) / threadSize;
  kernel_take<<<blockSize, threadSize>>>(x_data, indices_data, y_data, dev_xshape, dev_yshape, dev_indices_shape,
      yndim, xndim, indices_ndim, ysize, axis);

  hipFree(dev_xshape);
  hipFree(dev_yshape);
  hipFree(dev_indices_shape);
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_take_noaxis(const int32_t *x_data, const int32_t *indices_data, int32_t *y_data, const uint64_t ysize, const uint64_t xsize){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for(uint64_t i = tid; i < ysize; i+=gridDim.x*blockDim.x){
    int32_t in_i = min((uint64_t)max(indices_data[i], 0), xsize-1); 
    y_data[i] = x_data[in_i];
  }
}
const char* cuda_take(const int32_t *x_data, const int32_t *indices_data, int32_t *y_data, const uint64_t ysize, const uint64_t xsize){
  int threadSize = 256;
  int blockSize = getGridSize(ysize, threadSize);//(ysize + threadSize - 1) / threadSize;
  kernel_take_noaxis<<<blockSize, threadSize>>>(x_data, indices_data, y_data, ysize, xsize);
  print_to_file(y_data, ysize, "/tmp/cu_take.log");
  return check_cuda_error(hipGetLastError());
}
