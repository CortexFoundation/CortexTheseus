#include "hip/hip_runtime.h"
#include "cuda_ops.h"
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <memory>
#include <string.h>
#include <iostream>
#include <string>
#include "nms.h"

// #define CVM_PRINT_CUDA_RESULT

void print_to_file(const int32_t *y, int32_t n, std::string filename){
#ifdef CVM_PRINT_CUDA_RESULT
  int32_t *y_data = new int32_t[n];
  hipMemcpy(y_data, y, sizeof(int32_t)*n, hipMemcpyDeviceToHost);

  FILE *fp = fopen(filename.c_str(), "a+");

  int32_t min = y_data[0], max= y_data[0];
  for(uint64_t i = 0; i < n; i++){
    min = min > y_data[i] ? y_data[i] : min;
    max = max < y_data[i] ? y_data[i] : max;
  }
  fprintf(fp, "%d %d\n", min, max);
  for(uint64_t i = 0; i < 1000 && i < n; i++){
    fprintf(fp, "%d ", y_data[i]);
  }
  fprintf(fp, "\n");
  fclose(fp);
  delete y_data;
#endif
}
inline int32_t getGridSize(const int64_t n, const int32_t blockSize){
  int64_t tg = (n + blockSize - 1) / blockSize;
  return tg > 4096 ? 4096 : tg;
}
inline int32_t getShareMemorySize(const int32_t device_id, int&error_code){
  static int32_t sharedMemPerBlock = 0;
  if(sharedMemPerBlock == 0){
    hipDeviceProp_t prop;
    hipError_t status = hipGetDeviceProperties(&prop, device_id);
    if(status != hipSuccess){
        error_code = ERROR_GET_PROPERTIES;
        return -1;
    }
    sharedMemPerBlock = prop.sharedMemPerBlock;
  }
  return sharedMemPerBlock;
}
inline int32_t getFreeMemorySize(const int32_t device_id, int&error_code){
  size_t freeSize = 0, totalSize = 0;
  hipError_t status = hipMemGetInfo(&freeSize, &totalSize);
  if(status != hipSuccess){
    error_code = ERROR_GET_PROPERTIES;
    return -1;
  }
  return freeSize;
}

const char* check_cuda_error(hipError_t error){
  if(error == hipSuccess) return NULL;
  else return hipGetErrorString(error);
}

__global__ void kernel_elemwise_add(int32_t *a, int32_t *b, int32_t *c, uint64_t n){
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(uint64_t i = tid; i < n; i += gridDim.x * blockDim.x){
    c[i] = a[i] + b[i];
  }
}

const char* cuda_elemwise_add(int32_t *a, int32_t *b, int32_t *c, uint64_t n, int& error_code){
  int32_t *dev_a = a, *dev_b = b, *dev_c = c;
  int blockSize = 256;
  int gridSize = getGridSize(n, blockSize);
  kernel_elemwise_add<<<gridSize, blockSize>>>(dev_a, dev_b, dev_c, n);
  hipError_t error = hipGetLastError();
  if(error != hipSuccess){
    error_code = ERROR_KERNEL;
  }
  print_to_file(dev_c, n, "/tmp/zkh/trec/gpu/elemwise_add.txt");
  return check_cuda_error(error);
}
__global__ void kernel_elemwise_sub(int32_t *a, int32_t *b, int32_t *c, uint64_t n){
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(uint64_t i = tid; i < n; i += gridDim.x*blockDim.x){
    c[i] = a[i] - b[i];
  }
}

const char* cuda_elemwise_sub(int32_t *a, int32_t *b, int32_t *c, uint64_t n, int& error_code){
  int blockSize = 256;
  int gridSize = getGridSize(n, blockSize);
  kernel_elemwise_sub<<<gridSize, blockSize>>>(a, b, c, n);
  hipError_t error = hipGetLastError();
  if(error != hipSuccess){
    error_code = ERROR_KERNEL;
  }
  return check_cuda_error(error);
}

#define BS 16
#define FS 8
__global__ void kernel_conv2d(
    const int32_t * __restrict__ input, const int32_t i_n, const int32_t i_c, const int32_t i_h, const int32_t i_w,
    const int32_t * __restrict__ filter, const int32_t f_n, const int32_t f_c, const int32_t f_h, const int32_t f_w,
    const int32_t * __restrict__ bias,
    const int32_t padding_h, const int32_t padding_w,
    const int32_t stride_h, const int32_t stride_w,
    const int32_t dilation_h, const int32_t dilation_w,
    const int32_t groups,
    int32_t *output, const int32_t o_n, const int32_t o_c, const int32_t o_h, const int32_t o_w){
  int g_x = blockDim.x * blockIdx.x + threadIdx.x;
  int l_y = threadIdx.y; 
  int l_x = threadIdx.x;
  int tmp_f_h = (f_h - 1) * dilation_h + 1; // for dilation, to be optimized
  int tmp_f_w = (f_w - 1) * dilation_w + 1;
  int tmp_o_h = i_h + 2 * padding_h - tmp_f_h + 1; // for stride
  int tmp_o_w = i_w + 2 * padding_w - tmp_f_w + 1;
  int perBlockOneImageY = (tmp_o_h+BS-1) / BS;
  int perBlockOneImageX = (tmp_o_w+BS-1) / BS;
  int l_o_c = blockIdx.y / perBlockOneImageY;
  int n = l_o_c / ((o_c+FS-1)/FS);
  int nsize = n * i_c * i_h * i_w; 
  int l_f_n = l_o_c % ((o_c+FS-1)/FS);
  int l_o_hi = blockIdx.y % perBlockOneImageY;
  int l_o_wi = blockIdx.x % perBlockOneImageX;
  int l_o_h = l_o_hi * BS + l_y;
  //    int l_o_w = l_o_wi * BS + l_x;

  const int32_t F_H = f_h;
  const int32_t F_W = f_w;
  //    __shared__ int32_t shared_i[BS + F_H - 1][BS + F_W - 1];
  int32_t sih = BS + tmp_f_h - 1;
  int32_t siw = BS + tmp_f_w - 1;
  extern __shared__ int32_t  share[];
  int32_t *shared_i = (int32_t*)share; 
  int32_t *shared_f = &share[sih * siw];
  int32_t *shared_b = &shared_f[F_H*F_W*FS];

  int32_t sum[FS] = {0}; 
  int min_s_y = (l_o_hi+1) * BS <= tmp_o_h ? BS : tmp_o_h%BS;
  int min_s_x = (l_o_wi+1) * BS <= tmp_o_w ? BS : tmp_o_w%BS;

  //load bias to shared memory
  int lid = l_y * BS + l_x;
  for(int i = lid; bias != NULL && i < FS; i+=BS*BS){
    if(l_f_n*FS + i < o_c)
      shared_b[i] = bias[l_f_n*FS + i];
    else shared_b[i] = 0;
  }

  if(l_o_h >= tmp_o_h || g_x >= tmp_o_w) return;

  for(int c = 0; c < i_c; c++){
    //load input to shared
    int l_i_h = l_o_h - padding_h;
    int i_y = c * i_h + l_i_h;
    int i_x = g_x - padding_w;
    if(l_i_h < 0 || i_x < 0 || l_i_h >= i_h || i_x >= i_w)
      shared_i[l_y*siw + l_x] = 0;
    else
      shared_i[l_y*siw + l_x] = input[nsize + i_y * i_w + i_x];

    if(l_y < tmp_f_h-1){
      for(int i = l_y; i < tmp_f_h-1; i+=min_s_y){
        if(l_i_h+min_s_y+i-l_y < 0 || i_x < 0 || l_i_h+min_s_y+i-l_y >= i_h || i_x >= i_w)
          shared_i[(i+min_s_y)*siw + l_x] = 0;
        else
          shared_i[(i + min_s_y)*siw + l_x] = input[nsize + (i_y + min_s_y + i - l_y) * i_w + i_x];     
      }
    }
    if(l_x < tmp_f_w-1){
      for(int i = l_x; i < tmp_f_w-1; i+= min_s_x){
        if(l_i_h < 0 || i_x+min_s_x+i-l_x < 0 || l_i_h >= i_h || i_x+min_s_x+i-l_x >= i_w)
          shared_i[l_y * siw + i+min_s_x] = 0;
        else
          shared_i[l_y * siw + i + min_s_x] = input[nsize + i_y * i_w + i_x + min_s_x + i - l_x];
      }
    }
    if(l_y < tmp_f_h-1 && l_x < tmp_f_w-1){
      for(int i = l_y; i < tmp_f_h-1; i+=min_s_y){
        for(int j = l_x; j < tmp_f_w-1; j+=min_s_x){
          if(l_i_h+min_s_y+i-l_y < 0 || i_x+min_s_x+j-l_x < 0 || l_i_h+min_s_y+i-l_y >= i_h || i_x+min_s_x+j-l_x >= i_w)
            shared_i[(i+min_s_y) * siw + j+min_s_x] = 0;
          else
            shared_i[(i+min_s_y) * siw + j+min_s_x] = input[nsize + (i_y+min_s_y + i-l_y)*i_w + i_x + min_s_x + j - l_x];
        }
      }
    }

    //load filter to shared;
    if(l_y < F_H && l_x < F_W){
      for(int i = l_y; i < F_H; i+= min_s_y)
        for(int j = l_x; j < F_W; j+=min_s_x)
          for(int fc = 0; fc < FS; fc++){
            if(l_f_n * FS + fc < o_c)
              shared_f[fc * F_H*F_W + i*F_W + j] = filter[(l_f_n*FS+fc) * F_H * F_W * f_c + c * F_H * F_W + i * F_W + j];
            else shared_f[fc * F_H * F_W + i * F_W + j] = 0;
          }
    }
    __syncthreads();

    for(int fy = 0; fy < F_H; fy++){
      for(int fx = 0; fx < F_W; fx++){
        int32_t tmpx = shared_i[(l_y+fy*dilation_h)*siw + l_x+fx*dilation_w];
#pragma unroll
        for(int fc = 0; fc < FS; fc++){
          sum[fc] += tmpx * shared_f[fc*F_H*F_W + fy*F_W + fx];
        }
      }
    } 
    __syncthreads();
  }

  if(l_o_h % stride_h == 0 && g_x % stride_w == 0){ //TODO to be optimized
    //int oi = l_o_c * o_h * o_w + l_o_h * o_w + g_x;
    for(int fc = 0; fc < FS; fc++){
      if(l_f_n*FS + fc < o_c){
        int oi = n*o_c*o_h*o_w + (l_f_n*FS+fc) * o_h * o_w + l_o_h/stride_h * o_w + g_x/stride_w;
        output[oi] = sum[fc] + (bias != NULL ? shared_b[fc] : 0);
      }
    }
  }
}

__global__ void kernel_conv2d_no_shared(
    const int32_t * __restrict__ input, const int32_t i_n, const int32_t i_c, const int32_t i_h, const int32_t i_w,
    const int32_t * __restrict__ filter, const int32_t f_n, const int32_t f_c, const int32_t f_h, const int32_t f_w,
    const int32_t * __restrict__ bias,
    const int32_t padding_h, const int32_t padding_w,
    const int32_t stride_h, const int32_t stride_w,
    const int32_t dilation_h, const int32_t dilation_w,
    const int32_t groups,
    int32_t *output, const int32_t o_n, const int32_t o_c, const int32_t o_h, const int32_t o_w){
  int32_t gy = threadIdx.y + blockIdx.y * blockDim.y;
  int32_t gx = threadIdx.x + blockIdx.x * blockDim.x;
  int32_t l_o_h = gy % o_h;
  int32_t l_o_c = gy / o_h % o_c;
  int32_t l_o_n = gy / (o_h * o_c);
  if(gy < o_n * o_c * o_h && gx < o_w){
    int32_t sum = 0;
    for(int ic = 0; ic < i_c; ++ic){
      for(int fy = 0; fy < f_h; ++fy){
        for(int fx = 0; fx < f_w; ++fx){
          int32_t l_i_h = l_o_h * stride_h + fy * dilation_h - padding_h;
          int32_t l_i_w = gx * stride_w + fx * dilation_h - padding_w;
          int32_t x;
          if(l_i_h < 0 || l_i_w < 0 || l_i_h >= i_h || l_i_w >= i_w)
            x = 0;
          else x = input[l_o_n * i_c * i_h * i_w + ic * i_h * i_w + l_i_h * i_w + l_i_w];
          sum += x * filter[l_o_c * i_c * f_h * f_w + ic * f_h * f_w + fy * f_w + fx];
        }
      }
    }
    output[gy * o_w + gx] = sum + (bias != NULL ? bias[l_o_c] : 0);
  }
}
__global__ void kernel_int32_to_int8(const int32_t *in_data, int8_t *out_data, const int n){
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    for(int64_t i = tid; i < n; i+= gridDim.x * blockDim.x){
        out_data[i] = static_cast<int8_t>(in_data[i]);
    }
}

__global__ void im2col_gpu_kernel(const int n, const int32_t* data_im,
        const int height, const int width, const int kernel_h, const int kernel_w,
        const int pad_h, const int pad_w,
        const int stride_h, const int stride_w,
        const int dilation_h, const int dilation_w,
        const int height_col, const int width_col,
        int8_t* data_col) {
//    CUDA_KERNEL_LOOP(index, n) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(int64_t index = tid; index < n; index += gridDim.x*blockDim.x){
        const int h_index = index / width_col;
        const int h_col = h_index % height_col;
        const int w_col = index % width_col;
        const int c_im = h_index / height_col;
        const int c_col = c_im * kernel_h * kernel_w;
        const int h_offset = h_col * stride_h - pad_h;
        const int w_offset = w_col * stride_w - pad_w;
        int8_t* data_col_ptr = data_col;
        data_col_ptr += (c_col * height_col + h_col) * width_col + w_col;
        const int32_t* data_im_ptr = data_im;
        data_im_ptr += (c_im * height + h_offset) * width + w_offset;
        for (int i = 0; i < kernel_h; ++i) {
            for (int j = 0; j < kernel_w; ++j) {
                int h_im = h_offset + i * dilation_h;
                int w_im = w_offset + j * dilation_w;
                *data_col_ptr =
                    (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) ?
                    static_cast<int8_t>(data_im_ptr[i * dilation_h * width + j * dilation_w]) : 0;
                data_col_ptr += height_col * width_col;
            }
        }
    }
}

#define TILE_WIDTH 16
__global__ void kernel_matrix_mul(
    int8_t *a, // m*k 
    int8_t *b, // k*n
    int32_t *c, // m*n
    int32_t m, int32_t k, int32_t n, int32_t *bias){
  __shared__ int8_t sharedm[TILE_WIDTH][TILE_WIDTH];
  __shared__ int8_t sharedn[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int row = by*TILE_WIDTH + ty;
  int col = bx*TILE_WIDTH + tx;
  int sum = 0;

  for (int i = 0; i < (int)(ceil((float)k/TILE_WIDTH)); i++)
  {
    if (i*TILE_WIDTH + tx < k && row < m)//m*k
      sharedm[ty][tx] = a[row*k + i*TILE_WIDTH + tx];
    else
      sharedm[ty][tx] = 0;

    if(i*TILE_WIDTH + ty < k && col < n)//k*n
      sharedn[ty][tx] =b[(i*TILE_WIDTH + ty) * n + col] ;//b[col * k + i * TILE_WIDTH + ty];
    else
      sharedn[ty][tx] = 0;
    __syncthreads();

    for(int j = 0; j < TILE_WIDTH; j++)
      sum += static_cast<int32_t>(sharedm[ty][j]) * sharedn[j][tx];
    __syncthreads();
  }
  if (row < m && col < n){
    if(bias != NULL) sum += bias[row];
    c[row*n + col] = sum;
  }
}
inline void im2col_gpu(const int32_t* data_im, const int channels,
        const int height, const int width, const int kernel_h, const int kernel_w,
        const int pad_h, const int pad_w,
        const int stride_h, const int stride_w,
        const int dilation_h, const int dilation_w,
        int8_t* data_col) {
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int height_col = (height + 2 * pad_h -
            (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    int width_col = (width + 2 * pad_w -
            (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
    int num_kernels = channels * height_col * width_col;
    int threads = 256;
    int blocks = (num_kernels + threads - 1) / threads;
    im2col_gpu_kernel<<<blocks, threads>>>(
                num_kernels, data_im, height, width, kernel_h, kernel_w, pad_h,
                pad_w, stride_h, stride_w, dilation_h, dilation_w, height_col,
                width_col, data_col);
}
const char* cuda_conv2d(
    int32_t *input, int32_t i_n, int32_t i_c, int32_t i_h, int32_t i_w,
    int32_t *filter, int32_t f_n, int32_t f_c, const int32_t f_h, const int32_t f_w,
    int32_t *bias,
    const int32_t padding_h, const int32_t padding_w,
    const int32_t stride_h, const int32_t stride_w,
    const int32_t dilation_h, const int32_t dilation_w,
    int32_t groups,
    int32_t *output, int32_t o_n, int32_t o_c, int32_t o_h, int32_t o_w, 
    int32_t device_id,
    int& error_code){
  if(i_n < 1 || i_c < 1 || i_h < 1 || i_w < 1 || f_n < 1 || f_c < 1 || f_h < 1 || f_w < 1 || 
      padding_h < 0 || padding_w < 0 || stride_h < 1 || stride_w < 1 || dilation_h < 1 || dilation_w < 1 ||
      o_n < 1 || o_c < 1 || o_h < 1 || o_w < 1){
    error_code = ERROR_PARAMS;
    return "error args";
  }
  int32_t *dev_i = input, *dev_f = filter, *dev_o = output, *dev_b = bias;

  int tmp_f_h = (f_h - 1) * dilation_h + 1; // for dilation, to be optimized
  int tmp_f_w = (f_w - 1) * dilation_w + 1;
  int tmp_o_h = i_h + 2 * padding_h - tmp_f_h + 1; //for stride > 1 , TODO to be optimized
  int tmp_o_w = i_w + 2 * padding_w - tmp_f_w + 1;
  int32_t totalShareMemSize = getShareMemorySize(device_id, error_code);
  if(error_code != NON_ERROR){
    return check_cuda_error(hipGetLastError());
  }
  size_t share_size = ((BS + tmp_f_h - 1) * (BS + tmp_f_w - 1) + f_h * f_w * FS + FS) * sizeof(int32_t);
  if(share_size < totalShareMemSize){
    size_t freeSize = getFreeMemorySize(device_id, error_code);
    size_t tmp_filter_size = o_c * i_c * f_h * f_w * sizeof(int8_t);
    size_t tmp_input_size = i_c * f_h * f_w * o_h * o_w * sizeof(int8_t);
    if(tmp_filter_size + tmp_input_size >= freeSize){
      int b_h = BS;
      int b_w = BS;
      int32_t g_h = o_n * ((o_c + FS - 1) / FS) * ((tmp_o_h + b_h - 1) / b_h);
      int32_t g_w = (tmp_o_w + b_w - 1) / b_w;
      dim3 bDim(b_w, b_h, 1);
      dim3 gDim(g_w, g_h, 1);
      kernel_conv2d<<<gDim, bDim, share_size>>>(
          dev_i, i_n, i_c, i_h, i_w,
          dev_f, f_n, f_c, f_h, f_w,
          dev_b, 
          padding_h, padding_w,
          stride_h, stride_w,
          dilation_h, dilation_w,
          groups,
          dev_o, o_n, o_c, o_h, o_w);
    }else{
      int32_t fn = o_c * i_c * f_h * f_w;
      const int M = o_c;
      const int K = i_c * f_h * f_w;
      const int N = o_h * o_w;
      dim3 bDim(TILE_WIDTH, TILE_WIDTH, 1);
      int gh = (M + TILE_WIDTH - 1) / TILE_WIDTH;
      int gw = (N + TILE_WIDTH - 1) / TILE_WIDTH;
      dim3 gDim(gw, gh, 1);

      int8_t *d_f, *d_col;
      hipError_t status = hipMalloc((void**)&d_f, fn * sizeof(int8_t));
      if(status != hipSuccess){
        error_code = ERROR_MALLOC;
        return check_cuda_error(status);
      }
      int blockSize = 256;
      int gridSize = getGridSize(fn, blockSize);
      kernel_int32_to_int8<<<gridSize, blockSize>>>(dev_f, d_f, fn);

      status = hipMalloc((void**)&d_col, sizeof(int8_t) * i_c * f_h * f_w * o_h * o_w);
      if(status != hipSuccess){
        hipFree(d_f);
        error_code = ERROR_MALLOC;
        return check_cuda_error(status);
      }
      
      for(int i = 0; i < o_n; i++){
        im2col_gpu(dev_i + i * i_c * i_h * i_w,
            i_c, i_h, i_w, f_h, f_w, padding_h, padding_w, stride_h, stride_w, 
            dilation_h, dilation_w, d_col);
        kernel_matrix_mul<<<gDim, bDim>>>(d_f, d_col, dev_o + i * o_c * o_h * o_w, M, K, N, dev_b);
      }
      hipFree(d_f);
      hipFree(d_col);
    }
  }else{
    int b_h = BS;
    int b_w = BS;
    int g_h = o_n * o_c * ((o_h + b_h - 1) / b_h);
    int g_w = (o_w + b_w - 1) / b_w;
    dim3 bDim(b_w, b_h, 1);
    dim3 gDim(g_w, g_h, 1);
    kernel_conv2d_no_shared<<<gDim, bDim>>>(
        dev_i, i_n, i_c, i_h, i_w,
        dev_f, f_n, f_c, f_h, f_w,
        dev_b, 
        padding_h, padding_w,
        stride_h, stride_w,
        dilation_h, dilation_w,
        groups,
        dev_o, o_n, o_c, o_h, o_w);
  }
  hipError_t error = hipGetLastError();
  if(hipSuccess != error){
    error_code = ERROR_KERNEL;
  }

  print_to_file(dev_o, o_n * o_c * o_h * o_w, "/tmp/zkh/trec/gpu/conv2d.txt");
  return check_cuda_error(error);
}
__global__ void kernel_depthwise_conv2d(
    const int32_t * __restrict__ input, int32_t i_n, int32_t i_c, int32_t i_h, int32_t i_w,
    const int32_t * __restrict__ filter, int32_t f_n, int32_t f_c, int32_t f_h, int32_t f_w,
    const int32_t * __restrict__ bias,
    int32_t padding_h, int32_t padding_w,
    int32_t stride_h, int32_t stride_w,
    int32_t dilation_h, int32_t dilation_w, 
    int32_t groups,
    int32_t *output, int32_t o_n, int32_t o_c, int32_t o_h, int32_t o_w)
{
  int g_x = blockDim.x * blockIdx.x + threadIdx.x;
  int l_y = threadIdx.y; 
  int l_x = threadIdx.x;
  int tmp_f_h = (f_h - 1) * dilation_h + 1; // for dilation, to be optimized
  int tmp_f_w = (f_w - 1) * dilation_w + 1;
  int tmp_o_h = i_h + 2 * padding_h - tmp_f_h + 1; // for stride
  int tmp_o_w = i_w + 2 * padding_w - tmp_f_w + 1;
  int perBlockOneImageY = (tmp_o_h+BS-1) / BS;
  int perBlockOneImageX = (tmp_o_w+BS-1) / BS;
  int l_o_c = blockIdx.y / perBlockOneImageY;
  int l_f_c = l_o_c % o_c;
  int l_o_hi = blockIdx.y % perBlockOneImageY;
  int l_o_wi = blockIdx.x % perBlockOneImageX;
  int l_o_h = l_o_hi * BS + l_y;
  //    int l_o_w = l_o_wi * BS + l_x;
  if(l_o_h >= tmp_o_h || g_x >= tmp_o_w) return;

  const int32_t F_H = f_h;
  const int32_t F_W = f_w;
  //    __shared__ int32_t shared_i[BS + F_H - 1][BS + F_W - 1];
  int32_t sih = BS + tmp_f_h - 1;
  int32_t siw = BS + tmp_f_w - 1;
  extern __shared__ int32_t  share[];
  int32_t *shared_i = (int32_t*)share; 
  int32_t *shared_f = &share[sih * siw];

  int32_t sum = 0; 
  int min_s_y = (l_o_hi+1) * BS <= tmp_o_h ? BS : tmp_o_h%BS;
  int min_s_x = (l_o_wi+1) * BS <= tmp_o_w ? BS : tmp_o_w%BS;

  //load input to shared
  int l_i_h = l_o_h - padding_h;
  int i_y = l_o_c * i_h + l_i_h;
  int i_x = g_x - padding_w;
  // 0~2-> -1~1
  if(l_i_h < 0 || i_x < 0 || l_i_h >= i_h || i_x >= i_w)
    shared_i[l_y*siw + l_x] = 0;
  else
    shared_i[l_y*siw + l_x] = input[i_y * i_w + i_x];

  if(l_y < tmp_f_h-1){
    for(int i = l_y; i < tmp_f_h-1; i+=min_s_y){
      if(l_i_h+min_s_y+i-l_y < 0 || i_x < 0 || l_i_h+min_s_y+i-l_y >= i_h || i_x >= i_w)
        shared_i[(i+min_s_y)*siw + l_x] = 0;
      else
        shared_i[(i + min_s_y)*siw + l_x] = input[(i_y + min_s_y + i - l_y) * i_w + i_x]; 
    }
  }
  if(l_x < tmp_f_w-1){
    for(int i = l_x; i < tmp_f_w-1; i+= min_s_x){
      if(l_i_h < 0 || i_x+min_s_x+i-l_x < 0 || l_i_h >= i_h || i_x+min_s_x+i-l_x >= i_w)
        shared_i[l_y * siw + i+min_s_x] = 0;
      else
        shared_i[l_y * siw + i + min_s_x] = input[i_y * i_w + i_x + min_s_x + i - l_x];
    }
  }
  if(l_y < tmp_f_h-1 && l_x < tmp_f_w-1){
    for(int i = l_y; i < tmp_f_h-1; i+=min_s_y){
      for(int j = l_x; j < tmp_f_w-1; j+=min_s_x){
        if(l_i_h+min_s_y+i-l_y < 0 || i_x+min_s_x+j-l_x < 0 || l_i_h+min_s_y+i-l_y >= i_h || i_x+min_s_x+j-l_x >= i_w)
          shared_i[(i+min_s_y) * siw + j+min_s_x] = 0;
        else
          shared_i[(i+min_s_y) * siw + j+min_s_x] = input[(i_y+min_s_y + i-l_y)*i_w + i_x + min_s_x + j - l_x];
      }
    }
  }

  //load filter to shared;
  if(l_y < F_H && l_x < F_W){
    for(int i = l_y; i < F_H; i+= min_s_y)
      for(int j = l_x; j < F_W; j+=min_s_x)
        shared_f[i*F_W + j] = filter[l_f_c * F_H * F_W + i * F_W + j];
  }
  __syncthreads();

  for(int fy = 0; fy < F_H; fy++){
    for(int fx = 0; fx < F_W; fx++){
      sum += shared_i[(l_y+fy*dilation_h)*siw + l_x+fx*dilation_w] * shared_f[fy*F_W + fx];
    }
  } 
  __syncthreads();

  if(l_o_h % stride_h == 0 && g_x % stride_w == 0){
    //int oi = l_o_c * o_h * o_w + l_o_h * o_w + g_x;
    int oi = l_o_c * o_h * o_w + l_o_h/stride_h * o_w + g_x/stride_w;
    output[oi] = sum + (bias != NULL ? bias[l_o_c%o_c] : 0);
  }
}
__global__ void kernel_depthwise_conv2d_no_shared(
    int32_t *input, int32_t i_n, int32_t i_c, int32_t i_h, int32_t i_w,
    int32_t *filter, int32_t f_n, int32_t f_c, int32_t f_h, int32_t f_w,
    int32_t *bias,
    int32_t padding_h, int32_t padding_w,
    int32_t stride_h, int32_t stride_w,
    int32_t dilation_h, int32_t dilation_w, 
    int32_t groups,
    int32_t *output, int32_t o_n, int32_t o_c, int32_t o_h, int32_t o_w){
  int32_t gy = threadIdx.y + blockIdx.y * blockDim.y;
  int32_t gx = threadIdx.x + blockIdx.x * blockDim.x;
  int32_t l_o_h = gy % o_h;
  int32_t l_o_c = gy / o_h % o_c;
  int32_t l_o_n = gy / (o_h * o_c);
  if(gy < o_n * o_c * o_h && gx < o_w){
    int32_t sum = 0;
    for(int fy = 0; fy < f_h; ++fy){
      for(int fx = 0; fx < f_w; ++fx){
        int32_t l_i_h = l_o_h * stride_h + fy * dilation_h - padding_h;
        int32_t l_i_w = gx * stride_w + fx * dilation_w - padding_w;
        int32_t x;
        if(l_i_h < 0 || l_i_w < 0 || l_i_h >= i_h || l_i_w >= i_w)
          //x = 0;
          continue;
        x = input[l_o_n * i_c * i_h * i_w + l_o_c * i_h * i_w + l_i_h * i_w + l_i_w];
        sum += x * filter[l_o_c * f_h * f_w + fy * f_w + fx];
      }
    }
    output[gy * o_w + gx] = sum + (bias != NULL ? bias[l_o_c] : 0);
  }
}
const char* cuda_depthwise_conv2d(
    int32_t *input, int32_t i_n, int32_t i_c, int32_t i_h, int32_t i_w,
    int32_t *filter, int32_t f_n, int32_t f_c, int32_t f_h, int32_t f_w,
    int32_t *bias,
    int32_t padding_h, int32_t padding_w,
    int32_t stride_h, int32_t stride_w,
    int32_t dilation_h, int32_t dilation_w,
    int32_t groups,
    int32_t *output, int32_t o_n, int32_t o_c, int32_t o_h, int32_t o_w, int32_t device_id, int& error_code){
  int32_t *dev_i = input, *dev_f = filter, *dev_o = output, *dev_b = bias;

  int b_h = BS;
  int b_w = BS;
  int tmp_f_h = (f_h - 1) * dilation_h + 1; // for dilation, to be optimized
  int tmp_f_w = (f_w - 1) * dilation_w + 1;
  int tmp_o_h = i_h + 2 * padding_h - tmp_f_h + 1; //for stride > 1
  int tmp_o_w = i_w + 2 * padding_w - tmp_f_w + 1;
  const int32_t totalShareMemSize = getShareMemorySize(device_id, error_code);
  if(error_code != NON_ERROR){
    return check_cuda_error(hipGetLastError());
  }
  size_t share_size = (BS + tmp_f_h - 1) * (BS + tmp_f_w - 1) * sizeof(int32_t) + f_h * f_w * sizeof(int32_t);
  if(false){//(share_size < totalShareMemSize){
    int32_t g_h = o_n * o_c * ((tmp_o_h + b_h - 1) / b_h);
    int32_t g_w = (tmp_o_w + b_w - 1) / b_w;
    dim3 bDim(b_w, b_h, 1);
    dim3 gDim(g_w, g_h, 1);
    kernel_depthwise_conv2d<<<gDim, bDim, share_size>>>(
        dev_i, i_n, i_c, i_h, i_w,
        dev_f, f_n, f_c, f_h, f_w,
        dev_b, 
        padding_h, padding_w,
        stride_h, stride_w,
        dilation_h, dilation_w,
        groups,
        dev_o, o_n, o_c, o_h, o_w);
  }else{
    int32_t g_h = o_n * o_c * ((tmp_o_h + b_h - 1) / b_h); 
    int32_t g_w = (tmp_o_w + b_w - 1) / b_w;
    dim3 bDim(b_w, b_h, 1);
    dim3 gDim(g_w, g_h, 1);
    kernel_depthwise_conv2d_no_shared<<<gDim, bDim>>>(
        dev_i, i_n, i_c, i_h, i_w,
        dev_f, f_n, f_c, f_h, f_w,
        dev_b, 
        padding_h, padding_w,
        stride_h, stride_w,
        dilation_h, dilation_w,
        groups,
        dev_o, o_n, o_c, o_h, o_w);
  }
  hipError_t error = hipGetLastError();
  if(hipSuccess != error){
    error_code = ERROR_KERNEL;
  }
  return check_cuda_error(error);
}

__global__ void kernel_max_pool(
    int32_t *input, int32_t i_n, int32_t i_c, int32_t i_h, int32_t i_w,
    int32_t f_h, int32_t f_w,
    int32_t padding_h, int32_t padding_w,
    int32_t stride_h, int32_t stride_w,
    int32_t *output, int32_t o_n, int32_t o_c, int32_t o_h, int32_t o_w){
  int g_x = blockDim.x * blockIdx.x + threadIdx.x;
  int l_y = threadIdx.y; 
  int l_x = threadIdx.x;
  int tmp_o_h = i_h + 2 * padding_h - f_h + 1; // for stride
  int tmp_o_w = i_w + 2 * padding_w - f_w + 1;
  int perBlockOneImageY = (tmp_o_h+BS-1) / BS;
  int perBlockOneImageX = (tmp_o_w+BS-1) / BS;
  int l_o_c = blockIdx.y / perBlockOneImageY;
  int l_o_hi = blockIdx.y % perBlockOneImageY;
  int l_o_wi = blockIdx.x % perBlockOneImageX;
  int l_o_h = l_o_hi * BS + l_y;
  //    int l_o_w = l_o_wi * BS + l_x;
  if(l_o_h >= tmp_o_h || g_x >= tmp_o_w) return;

  const int32_t F_H = f_h;
  const int32_t F_W = f_w;
  //    __shared__ int32_t shared_i[BS + F_H - 1][BS + F_W - 1];
  //    int32_t sih = BS + F_H - 1;
  int32_t siw = BS + F_W - 1;
  extern __shared__ int32_t  share[];
  int32_t *shared_i = (int32_t*)share; 

  int32_t max_elem = int(1)<<31; 
  int min_s_y = (l_o_hi+1) * BS <= tmp_o_h ? BS : tmp_o_h%BS;
  int min_s_x = (l_o_wi+1) * BS <= tmp_o_w ? BS : tmp_o_w%BS;

  //load input to shared
  int l_i_h = l_o_h - padding_h;
  int i_y = l_o_c * i_h + l_i_h;
  int i_x = g_x - padding_w;
  // 0~2-> -1~1
  if(l_i_h < 0 || i_x < 0 || l_i_h >= i_h || i_x >= i_w)
    shared_i[l_y*siw + l_x] = 0;
  else
    shared_i[l_y*siw + l_x] = input[i_y * i_w + i_x];

  if(l_y < F_H-1){
    for(int i = l_y; i < F_H-1; i+=min_s_y){
      if(l_i_h+min_s_y+i-l_y < 0 || i_x < 0 || l_i_h+min_s_y+i-l_y >= i_h || i_x >= i_w)
        shared_i[(i+min_s_y)*siw + l_x] = 0;
      else
        shared_i[(i + min_s_y)*siw + l_x] = input[(i_y + min_s_y + i - l_y) * i_w + i_x];     
    }
  }
  if(l_x < F_W-1){
    for(int i = l_x; i < F_W-1; i+= min_s_x){
      if(l_i_h < 0 || i_x+min_s_x+i-l_x < 0 || l_i_h >= i_h || i_x+min_s_x+i-l_x >= i_w)
        shared_i[l_y * siw + i+min_s_x] = 0;
      else
        shared_i[l_y * siw + i + min_s_x] = input[i_y * i_w + i_x + min_s_x + i - l_x];
    }
  }
  if(l_y < F_H-1 && l_x < F_W-1){
    for(int i = l_y; i < F_H-1; i+=min_s_y){
      for(int j = l_x; j < F_W-1; j+=min_s_x){
        if(l_i_h+min_s_y+i-l_y < 0 || i_x+min_s_x+j-l_x < 0 || l_i_h+min_s_y+i-l_y >= i_h || i_x+min_s_x+j-l_x >= i_w)
          shared_i[(i+min_s_y) * siw + j+min_s_x] = 0;
        else
          shared_i[(i+min_s_y) * siw + j+min_s_x] = input[(i_y+min_s_y + i-l_y)*i_w + i_x + min_s_x + j - l_x];
      }
    }
  }
  __syncthreads();

  for(int fy = 0; fy < F_H; fy++){
    for(int fx = 0; fx < F_W; fx++){
      int32_t tmp =  shared_i[(l_y+fy)*siw + l_x+fx];
      max_elem = max_elem < tmp ? tmp : max_elem;
    }
  } 
  __syncthreads();

  if(l_o_h % stride_h == 0 && g_x % stride_w == 0){
    //int oi = l_o_c * o_h * o_w + l_o_h * o_w + g_x;
    int oi = l_o_c * o_h * o_w + l_o_h/stride_h * o_w + g_x/stride_w;
    output[oi] = max_elem;
  }
}

__global__ void kernel_max_pool_no_shared(
    int32_t *input, int32_t i_n, int32_t i_c, int32_t i_h, int32_t i_w,
    int32_t f_h, int32_t f_w,
    int32_t padding_h, int32_t padding_w,
    int32_t stride_h, int32_t stride_w,
    int32_t *output, int32_t o_n, int32_t o_c, int32_t o_h, int32_t o_w){
  int32_t gy = threadIdx.y + blockIdx.y * blockDim.y;
  int32_t gx = threadIdx.x + blockIdx.x * blockDim.x;
  int32_t l_o_h = gy % o_h;
  int32_t l_o_c = gy / o_h % o_c;
  int32_t l_o_n = gy / (o_h * o_c);
  if(gy < o_n * o_c * o_h && gx < o_w){
    int32_t maxV = (int32_t)1 << 31;
    for(int fy = 0; fy < f_h; ++fy){
      for(int fx = 0; fx < f_w; ++fx){
        int32_t l_i_h = l_o_h * stride_h + fy  - padding_h;
        int32_t l_i_w = gx * stride_w + fx - padding_w;
        int32_t x;
        if(l_i_h < 0 || l_i_w < 0 || l_i_h >= i_h || l_i_w >= i_w)
          x = 0;
        else x = input[l_o_n * i_c * i_h * i_w + l_o_c * i_h * i_w + l_i_h * i_w + l_i_w];
        maxV = maxV < x ? x : maxV;
      }
    }
    output[gy * o_w + gx] = maxV;
  }
}
const char* cuda_max_pool(
    int32_t *input, int32_t i_n, int32_t i_c, int32_t i_h, int32_t i_w,
    const int32_t f_h, const int32_t f_w,
    int32_t padding_h, int32_t padding_w,
    int32_t stride_h, int32_t stride_w,
    int32_t *output, int32_t o_n, int32_t o_c, int32_t o_h, int32_t o_w, int32_t device_id, int& error_code){
  int32_t *dev_i = input, *dev_o = output;

  const int32_t totalShareMemSize = getShareMemorySize(device_id, error_code);
  if(error_code != NON_ERROR){
    return check_cuda_error(hipGetLastError());
  }
  size_t share_size = (BS + f_h - 1) * (BS + f_w - 1) * sizeof(int32_t);
  int b_h = BS;
  int b_w = BS;
  int tmp_o_h = i_h + 2 * padding_h - f_h + 1; //for stride > 1
  int tmp_o_w = i_w + 2 * padding_w - f_w + 1;
  if(false){//(share_size < totalShareMemSize){
    int32_t g_h = o_n * o_c * ((tmp_o_h + b_h - 1) / b_h);
    int32_t g_w = (tmp_o_w + b_w - 1) / b_w;
    dim3 bDim(b_w, b_h, 1);
    dim3 gDim(g_w, g_h, 1);
    kernel_max_pool<<<gDim, bDim, share_size>>>(
        dev_i, i_n, i_c, i_h, i_w,
        f_h, f_w,
        padding_h, padding_w, 
        stride_h, stride_w,
        dev_o, o_n, o_c, o_h, o_w);
  }else{
    int32_t g_h = o_n * o_c * ((o_h + b_h - 1) / b_h); 
    int32_t g_w = (o_w + b_w - 1) / b_w;
    dim3 bDim(b_w, b_h, 1);
    dim3 gDim(g_w, g_h, 1);
    kernel_max_pool_no_shared<<<gDim, bDim>>>(
        dev_i, i_n, i_c, i_h, i_w,
        f_h, f_w,
        padding_h, padding_w, 
        stride_h, stride_w,
        dev_o, o_n, o_c, o_h, o_w);
  }
  hipError_t error = hipGetLastError();
  if(hipSuccess != error){
    error_code = ERROR_KERNEL;
  }
  return check_cuda_error(error);
}

//#define TILE_WIDTH 16
__global__ void kernel_dense(
    int32_t *A, // m*k 
    int32_t *B, // was transposed, n*k
    int32_t *C, // m*n
    int32_t m, int32_t k, int32_t n, int32_t *bias, int32_t useBias){
  __shared__ int32_t sharedM[TILE_WIDTH][TILE_WIDTH];
  __shared__ int32_t sharedN[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int row = by*TILE_WIDTH + ty;
  int col = bx*TILE_WIDTH + tx;
  int sum = 0;

  for (int i = 0; i < (int)(ceil((float)k/TILE_WIDTH)); i++)
  {
    if (i*TILE_WIDTH + tx < k && row < m)//m*k
      sharedM[ty][tx] = A[row*k + i*TILE_WIDTH + tx];
    else
      sharedM[ty][tx] = 0;

    if(i*TILE_WIDTH + ty < k && col < n)//n*k
      sharedN[tx][ty] = B[col * k + i * TILE_WIDTH + ty];
    else
      sharedN[tx][ty] = 0;
    __syncthreads();

    for(int j = 0; j < TILE_WIDTH; j++)
      sum += sharedM[ty][j] * sharedN[tx][j];
    __syncthreads();
  }
  if (row < m && col < n){
    if(useBias == 1) sum += bias[col];
    C[row*n + col] = sum;
  }
}

const char* cuda_dense(
    int32_t *a,
    int32_t *b,
    int32_t *c,
    const int m, const int k, const int n, int32_t* bias, int& error_code){
  int32_t *dev_a = a, *dev_b = b, *dev_c = c, *dev_bias = bias, useBias = 0;
  if(bias != NULL) useBias = 1;

  dim3 bDim(TILE_WIDTH, TILE_WIDTH, 1);
  int gh = (m + TILE_WIDTH - 1) / TILE_WIDTH;
  int gw = (n + TILE_WIDTH - 1) / TILE_WIDTH;
  dim3 gDim(gw, gh, 1);
  kernel_dense<<<gDim, bDim>>>(dev_a, dev_b, dev_c, m, k, n, dev_bias, useBias);

  hipError_t error = hipGetLastError();
  if(hipSuccess != error){
    error_code = ERROR_KERNEL;
  }
  print_to_file(dev_c, m*n, "/tmp/zkh/trec/gpu/dense.txt");
  return check_cuda_error(error);
}

__global__ void kernel_clip(const int32_t *x, int32_t *y,
    const uint64_t n, const int32_t maxV, const int32_t minV){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for(uint64_t i = tid; i < n; i += gridDim.x*blockDim.x){
    y[i] = max(min(x[i], maxV), minV);
  }
}
const char* cuda_clip(const int32_t *x, int32_t *y, const uint64_t n, const int32_t max, const int32_t min, int& error_code){
  const int32_t *dev_x = x;
  int32_t *dev_y = y;

  int threadSize = 256;
  int blockSize = getGridSize(n, threadSize); //(n + threadSize - 1) / threadSize;
  kernel_clip<<<blockSize, threadSize>>>(dev_x, dev_y, n, max, min);
  hipError_t error = hipGetLastError();
  if(hipSuccess != error){
    error_code = ERROR_KERNEL;
  }
  return check_cuda_error(error);
}

__global__ void kernel_relu(const int32_t *x, int32_t*y, const uint64_t n){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for(uint64_t i = tid; i < n; i += gridDim.x * blockDim.x){
    y[i] = max(x[i], 0);
  }
}
const char* cuda_relu(const int32_t *x, int32_t *y, const uint64_t n, int& error_code){
  const int32_t *dev_x = x;
  int32_t *dev_y = y;

  int threadSize = 256;
  int blockSize = getGridSize(n, threadSize);//(n + threadSize - 1) / threadSize;
  kernel_relu<<<blockSize, threadSize>>>(dev_x, dev_y, n);

  hipError_t error = hipGetLastError();
  if(hipSuccess != error){
    error_code = ERROR_KERNEL;
  }
  return check_cuda_error(error);
}

const char* cuda_flatten(const int32_t *x, int32_t *y, const uint64_t n, int& error_code){
  if(x == y) return NULL;
  hipMemcpy(y, x, n * sizeof(int32_t), hipMemcpyDeviceToDevice);
  hipError_t error = hipGetLastError();
  if(hipSuccess != error){
    error_code = ERROR_MEMCPY;
  }
  return check_cuda_error(error);
}

inline __device__ int32_t broadcast_i_index(int64_t* oshape, int o_index, int64_t* ishape, int idim){
  int index = 0;
  int allIndex = 0;
  for(int i = 0; i < idim; i++){
    int idx = idim - 1 - i;
    int ovar = o_index % oshape[idx];
    if(ovar < ishape[idx]){
      index += i == 0 ? ovar : allIndex * ovar;
    }else if(ishape[idx] == 1){
    }else{
    }
    allIndex = (i == 0 ? ishape[idim-1] : allIndex * ishape[idx]);
    o_index /= oshape[idx];
  }
  return index;
}

__global__ void kernel_broadcast_add(const int32_t *a, const int32_t *b, int32_t*c, 
    const int64_t n,
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for(uint64_t i = tid; i < n; i += gridDim.x * blockDim.x){
    int ai = broadcast_i_index(cshape, i, ashape, adim);
    int bi = broadcast_i_index(cshape, i, bshape, bdim);
    c[i] = a[ai] + b[bi];
  }
}
const char* cuda_broadcast_add(const int32_t *a, const int32_t *b, int32_t* c, 
    const uint64_t n, 
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim,
    int& error_code)
{
  const int32_t *dev_a = a, *dev_b = b;
  int32_t *dev_c = c;
  int threadSize = 256;
  int blockSize = getGridSize(n, threadSize);

  int64_t *dev_ashape = NULL, *dev_bshape = NULL, *dev_cshape = NULL;
  hipError_t status;
  status = hipMalloc((void**)&dev_ashape, sizeof(int64_t) * adim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_bshape, sizeof(int64_t) * bdim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_cshape, sizeof(int64_t) * cdim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMemcpy(dev_ashape, ashape, sizeof(int64_t) * adim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMemcpy(dev_bshape, bshape, sizeof(int64_t) * bdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  status = hipMemcpy(dev_cshape, cshape, sizeof(int64_t) * cdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  kernel_broadcast_add<<<blockSize, threadSize>>>(dev_a, dev_b, dev_c, n, dev_ashape, adim, dev_bshape, bdim, dev_cshape, cdim);
  //hipDeviceSynchronize();

  status = hipGetLastError();
  if(hipSuccess != status){
    error_code = ERROR_KERNEL;
  }
end:
  if(dev_ashape != NULL) hipFree(dev_ashape);
  if(dev_bshape != NULL) hipFree(dev_bshape);
  if(dev_cshape != NULL) hipFree(dev_cshape);
  return check_cuda_error(hipGetLastError());
}
__global__ void kernel_broadcast_sub(const int32_t *a, const int32_t *b, int32_t*c, const uint64_t n,
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim
    ){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for(int i = tid; i < n; i += gridDim.x*blockDim.x){
    int32_t ai = broadcast_i_index(cshape, i, ashape, adim);
    int32_t bi = broadcast_i_index(cshape, i, bshape, bdim);
    c[i] = a[ai] - b[bi];
  }
}
const char* cuda_broadcast_sub(const int32_t *a, const int32_t *b, int32_t* c, const uint64_t n, 
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim,
    int& error_code){
  const int32_t *dev_a = a, *dev_b = b;
  int32_t *dev_c = c;

  int threadSize = 256;
  int blockSize = getGridSize(n, threadSize);
  int64_t *dev_ashape = NULL, *dev_bshape = NULL, *dev_cshape = NULL;
  hipError_t status;
  status = hipMalloc((void**)&dev_ashape, sizeof(int64_t) * adim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_bshape, sizeof(int64_t) * bdim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_cshape, sizeof(int64_t) * cdim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMemcpy(dev_ashape, ashape, sizeof(int64_t) * adim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  status = hipMemcpy(dev_bshape, bshape, sizeof(int64_t) * bdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  status = hipMemcpy(dev_cshape, cshape, sizeof(int64_t) * cdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  kernel_broadcast_sub<<<blockSize, threadSize>>>(dev_a, dev_b, dev_c, n, dev_ashape, adim, dev_bshape, bdim, dev_cshape, cdim);
  //hipDeviceSynchronize();
  if(hipGetLastError() != hipSuccess){
    error_code = ERROR_KERNEL;
  }
end:
  if(dev_ashape != NULL) hipFree(dev_ashape);
  if(dev_bshape != NULL) hipFree(dev_bshape);
  if(dev_cshape != NULL) hipFree(dev_cshape);
  return check_cuda_error(hipGetLastError());
}
__global__ void kernel_broadcast_mul(const int32_t *a, const int32_t *b, int32_t*c, const uint64_t n,
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim
    ){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for(int i = tid; i < n; i += gridDim.x*blockDim.x){
    int32_t ai = broadcast_i_index(cshape, i, ashape, adim);
    int32_t bi = broadcast_i_index(cshape, i, bshape, bdim);
    c[i] = a[ai] * b[bi];
  }
}
const char* cuda_broadcast_mul(const int32_t *a, const int32_t *b, int32_t* c, const uint64_t n, 
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim,
    int& error_code){
  const int32_t *dev_a = a, *dev_b = b;
  int32_t *dev_c = c;

  int threadSize = 256;
  int blockSize = getGridSize(n, threadSize);
  int64_t *dev_ashape = NULL, *dev_bshape = NULL, *dev_cshape = NULL;
  hipError_t status;
  status = hipMalloc((void**)&dev_ashape, sizeof(int64_t) * adim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_bshape, sizeof(int64_t) * bdim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_cshape, sizeof(int64_t) * cdim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMemcpy(dev_ashape, ashape, sizeof(int64_t) * adim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  status = hipMemcpy(dev_bshape, bshape, sizeof(int64_t) * bdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  status = hipMemcpy(dev_cshape, cshape, sizeof(int64_t) * cdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  kernel_broadcast_mul<<<blockSize, threadSize>>>(dev_a, dev_b, dev_c, n, dev_ashape, adim, dev_bshape, bdim, dev_cshape, cdim);
  //hipDeviceSynchronize();
  if(hipGetLastError() != hipSuccess){
    error_code = ERROR_KERNEL;
  }
  print_to_file(dev_c, n, "/tmp/zkh/trec/gpu/broadcast_mul.txt");
end:
  if(dev_ashape != NULL) hipFree(dev_ashape);
  if(dev_bshape != NULL) hipFree(dev_bshape);
  if(dev_cshape != NULL) hipFree(dev_cshape);
  return check_cuda_error(hipGetLastError());
}
__global__ void kernel_broadcast_div(const int32_t *a, const int32_t *b, int32_t*c, const uint64_t n,
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim
    ){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for(int i = tid; i < n; i += gridDim.x*blockDim.x){
    int32_t ai = broadcast_i_index(cshape, i, ashape, adim);
    int32_t bi = broadcast_i_index(cshape, i, bshape, bdim);
    c[i] = a[ai] / b[bi];
  }
}
const char* cuda_broadcast_div(const int32_t *a, const int32_t *b, int32_t* c, const uint64_t n,
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim,
    int& error_code){
  const int32_t *dev_a = a, *dev_b = b;
  int32_t *dev_c = c;

  hipError_t status;
  int64_t bsize = 1;
  for(int i = 0; i < bdim; i++){
    bsize *= bshape[i];
  }
  //int32_t* h_b = new int32_t[bsize];
  int32_t *h_b = (int32_t*)malloc(sizeof(int32_t) * bsize);
  if(h_b == NULL){
    error_code = ERROR_MALLOC;
    return "malloc failed";
  }
  status = hipMemcpy(h_b, dev_b, sizeof(int32_t) * bsize, hipMemcpyDeviceToHost);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    return check_cuda_error(status);
  }
  for(int i = 0; i < bsize; i++){
    if(h_b[i] == 0){
      free(h_b);
      error_code = ERROR_DIV_0;
      return "error: divide by zero";
    }
  }

  int threadSize = 256;
  int blockSize = getGridSize(n, threadSize);
  int64_t *dev_ashape = NULL, *dev_bshape = NULL, *dev_cshape = NULL;
  status = hipMalloc((void**)&dev_ashape, sizeof(int64_t) * adim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_bshape, sizeof(int64_t) * bdim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_cshape, sizeof(int64_t) * cdim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMemcpy(dev_ashape, ashape, sizeof(int64_t) * adim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  status = hipMemcpy(dev_bshape, bshape, sizeof(int64_t) * bdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  status = hipMemcpy(dev_cshape, cshape, sizeof(int64_t) * cdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  kernel_broadcast_div<<<blockSize, threadSize>>>(dev_a, dev_b, dev_c, n, dev_ashape, adim, dev_bshape, bdim, dev_cshape, cdim);
  //hipDeviceSynchronize();
  if(hipSuccess != hipGetLastError()){
    error_code = ERROR_KERNEL;
  }

end:
  if(dev_ashape != NULL) hipFree(dev_ashape);
  if(dev_bshape != NULL) hipFree(dev_bshape);
  if(dev_cshape != NULL) hipFree(dev_cshape);
  if(h_b != NULL) free(h_b);

  return check_cuda_error(hipGetLastError());
}
__global__ void kernel_broadcast_right_shift(const int32_t *a, const int32_t *b, int32_t*c, const uint64_t n,
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim
    ){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for(int i = tid; i < n; i += gridDim.x * blockDim.x){
    int32_t ai = broadcast_i_index(cshape, i, ashape, adim);
    int32_t bi = broadcast_i_index(cshape, i, bshape, bdim);
    c[i] = a[ai] >> b[bi];
  }
}
const char* cuda_broadcast_right_shift(const int32_t *a, const int32_t* b, int32_t* c, const uint64_t n,
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim,
    int& error_code){
  const int32_t *dev_a = a;
  const int32_t *dev_b = b;
  int32_t *dev_c = c;

  int threadSize = 256;
  int blockSize = getGridSize(n, threadSize);
  int64_t *dev_ashape = NULL, *dev_bshape = NULL, *dev_cshape = NULL;
  hipError_t status;
  status = hipMalloc((void**)&dev_ashape, sizeof(int64_t) * adim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_bshape, sizeof(int64_t) * bdim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_cshape, sizeof(int64_t) * cdim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMemcpy(dev_ashape, ashape, sizeof(int64_t) * adim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  status = hipMemcpy(dev_bshape, bshape, sizeof(int64_t) * bdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  status = hipMemcpy(dev_cshape, cshape, sizeof(int64_t) * cdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  kernel_broadcast_right_shift<<<blockSize, threadSize>>>(dev_a, dev_b, dev_c, n, dev_ashape, adim, dev_bshape, bdim, dev_cshape, cdim);
  //hipDeviceSynchronize();
end:
  if(dev_ashape != NULL) hipFree(dev_ashape);
  if(dev_bshape != NULL) hipFree(dev_bshape);
  if(dev_cshape != NULL) hipFree(dev_cshape);
  return check_cuda_error(hipGetLastError());
}
__global__ void kernel_broadcast_left_shift(const int32_t *a, const int32_t *b, int32_t*c, const uint64_t n,
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim
    ){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for(int i = tid; i < n; i += gridDim.x * blockDim.x){
    int32_t ai = broadcast_i_index(cshape, i, ashape, adim);
    int32_t bi = broadcast_i_index(cshape, i, bshape, bdim);
    c[i] = a[ai] << b[bi];
  }
}
const char* cuda_broadcast_left_shift(const int32_t *a, const int32_t *b, int32_t* c, const uint64_t n, 
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim,
    int& error_code){
  const int32_t *dev_a = a, *dev_b = b;
  int32_t *dev_c = c;

  int threadSize = 256;
  int blockSize = getGridSize(n, threadSize);
  int64_t *dev_ashape = NULL, *dev_bshape = NULL, *dev_cshape = NULL;
  hipError_t status;
  status = hipMalloc((void**)&dev_ashape, sizeof(int64_t) * adim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_bshape, sizeof(int64_t) * bdim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_cshape, sizeof(int64_t) * cdim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMemcpy(dev_ashape, ashape, sizeof(int64_t) * adim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  status = hipMemcpy(dev_bshape, bshape, sizeof(int64_t) * bdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  status = hipMemcpy(dev_cshape, cshape, sizeof(int64_t) * cdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  kernel_broadcast_left_shift<<<blockSize, threadSize>>>(dev_a, dev_b, dev_c, n, dev_ashape, adim, dev_bshape, bdim, dev_cshape, cdim);
  //hipDeviceSynchronize();
  if(hipSuccess != hipGetLastError()){
    error_code = ERROR_KERNEL;
  }
end:
  if(dev_ashape != NULL) hipFree(dev_ashape);
  if(dev_bshape != NULL) hipFree(dev_bshape);
  if(dev_cshape != NULL) hipFree(dev_cshape);

  return check_cuda_error(hipGetLastError());
}
__global__ void kernel_broadcast_max(const int32_t *a, const int32_t *b, int32_t *c, const uint64_t n,
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim
    ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for(int i = tid; i < n; i += gridDim.x*blockDim.x){
    int32_t ai = broadcast_i_index(cshape, i, ashape, adim);
    int32_t bi = broadcast_i_index(cshape, i, bshape, bdim);
    c[i] = a[ai] > b[bi] ? a[ai] : b[bi];
  }
}
const char* cuda_broadcast_max(const int32_t *a, const int32_t *b, int32_t* c, const uint64_t n, 
    int64_t *ashape, int32_t adim,
    int64_t *bshape, int32_t bdim,
    int64_t *cshape, int32_t cdim,
    int& error_code){
  const int32_t *dev_a = a, *dev_b = b;
  int32_t *dev_c = c;

  int threadSize = 256;
  int blockSize = getGridSize(n, threadSize);//(n + threadSize - 1) / threadSize;
  int64_t *dev_ashape = NULL, *dev_bshape = NULL, *dev_cshape = NULL;
  hipError_t status;
  status = hipMalloc((void**)&dev_ashape, sizeof(int64_t) * adim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_bshape, sizeof(int64_t) * bdim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_cshape, sizeof(int64_t) * cdim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMemcpy(dev_ashape, ashape, sizeof(int64_t) * adim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  status = hipMemcpy(dev_bshape, bshape, sizeof(int64_t) * bdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  status = hipMemcpy(dev_cshape, cshape, sizeof(int64_t) * cdim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  kernel_broadcast_max<<<blockSize, threadSize>>>(dev_a, dev_b, dev_c, n, dev_ashape, adim, dev_bshape, bdim, dev_cshape, cdim);
  //hipDeviceSynchronize();
  if(hipSuccess != hipGetLastError()){
    error_code = ERROR_KERNEL;
  }
end:
  if(dev_ashape != NULL) hipFree(dev_ashape);
  if(dev_bshape != NULL) hipFree(dev_bshape);
  if(dev_cshape != NULL) hipFree(dev_cshape);

  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_sum(const int32_t *x, int32_t *y, int64_t n){
  __shared__ int32_t buf[256];
  int32_t tid = threadIdx.x;
  int32_t sum = 0;
  for (int i = tid; i < n; i += blockDim.x){
    sum += x[i];
  }

  buf[tid] = sum;
  __syncthreads();
  for(int s = 1; s < blockDim.x; s*=2){
    if((tid % (2*s)) == 0){
      int a = buf[tid];
      int b = buf[tid+s];
      buf[tid] = a + b;
    }
    __syncthreads();
  }

  if(tid == 0) y[0] = buf[0];
}

__global__ void kernel_sum_with_axis(const int32_t *x, int32_t *y, const int32_t *realAxis,
    const int64_t *xshape, const int64_t *yshape, const int32_t axis_ndim, const uint64_t *every_xdim_size,
    const int32_t xndim, const int32_t yndim, const int64_t ysize, const int32_t* flag, const int64_t axis_size){
  int32_t tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(uint64_t i =tid; i < ysize; i+= gridDim.x*blockDim.x){
    uint64_t in_i = 0, o_i = i;
    for(int j = yndim-1, xj = xndim-1; j>=0; j--){
      uint64_t col = o_i % yshape[j];
      o_i /= yshape[j];
      while(xj >= 0 && flag[xj--] == 1);
      in_i += col * every_xdim_size[xj+1];
    }
    int32_t sum = 0;
    for(uint64_t xi = 0; xi < axis_size; xi++){
      uint64_t o_i = xi, tmp_in_i = 0;
      for(int j = axis_ndim - 1; j>=0; j--){
        uint64_t col = o_i % xshape[realAxis[j]];
        o_i /= xshape[realAxis[j]];
        tmp_in_i += col * every_xdim_size[realAxis[j]];
      }
      sum += x[in_i + tmp_in_i];
    }
    y[i] = sum;
  }
}
const char* cuda_sum(const int32_t *x, int32_t *y, const uint64_t xsize, const uint64_t ysize,
    const int64_t *xshape, const int64_t *yshape, const int32_t* realAxis, const int32_t* flag,
    const uint64_t *every_xdim_size, const int64_t axis_size,
    const int32_t xndim, const int32_t yndim, const int32_t axis_ndim, int& error_code){
  int64_t *dev_xshape = NULL, *dev_yshape = NULL;
  uint64_t *dev_every_xdim_size = NULL;
  int32_t *dev_flag = NULL, *dev_axis = NULL;
  if(axis_ndim == 0){
    kernel_sum<<<1, 256>>>(x, y, xsize);
    int error = hipGetLastError();
    if(error != hipSuccess){
        error_code = ERROR_KERNEL;
    }
  }else{
    int bSize = 256;
    int gSize = getGridSize(ysize, bSize);//(ysize + bSize - 1) / bSize;
    hipError_t status;
    status = hipMalloc((void**)&dev_xshape, sizeof(int64_t)*xndim);
    if(status != hipSuccess){
      error_code = ERROR_MALLOC;
      goto end;
    }
    status = hipMalloc((void**)&dev_yshape, sizeof(int64_t)*yndim);
    if(status != hipSuccess){
      error_code = ERROR_MALLOC;
      goto end;
    }
    status = hipMalloc((void**)&dev_axis, sizeof(int32_t) * axis_ndim);
    if(status != hipSuccess){
      error_code = ERROR_MALLOC;
      goto end;
    }
    status = hipMalloc((void**)&dev_every_xdim_size, sizeof(uint64_t) * xndim);
    if(status != hipSuccess){
      error_code = ERROR_MALLOC;
      goto end;
    }
    status = hipMalloc((void**)&dev_flag, sizeof(int32_t)*xndim);
    if(status != hipSuccess){
      error_code = ERROR_MALLOC;
      goto end;
    }
    status = hipMemcpy(dev_xshape, xshape, sizeof(int64_t)*xndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      error_code = ERROR_MEMCPY;
      goto end;
    }
    status = hipMemcpy(dev_yshape, yshape, sizeof(int64_t)*yndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      error_code = ERROR_MEMCPY;
      goto end;
    }
    status = hipMemcpy(dev_axis, realAxis, sizeof(int32_t)*axis_ndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      error_code = ERROR_MEMCPY;
      goto end;
    }
    status = hipMemcpy(dev_every_xdim_size, every_xdim_size, sizeof(uint64_t) * xndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      error_code = ERROR_MEMCPY;
      goto end;
    }
    status = hipMemcpy(dev_flag, flag, sizeof(int32_t)*xndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      error_code = ERROR_MEMCPY;
      goto end;
    }

    kernel_sum_with_axis<<<gSize, bSize>>>(x, y, dev_axis, dev_xshape, dev_yshape, axis_ndim, 
        dev_every_xdim_size, xndim, yndim, ysize, dev_flag, axis_size);
    if(hipSuccess != hipGetLastError()){
        error_code = ERROR_KERNEL;
    }
  }
  print_to_file(y, ysize, "/tmp/zkh/trec/gpu/sum.txt");

end:
  if(dev_xshape != NULL) hipFree(dev_xshape);
  if(dev_yshape != NULL) hipFree(dev_yshape);
  if(dev_axis != NULL) hipFree(dev_axis);
  if(dev_every_xdim_size != NULL) hipFree(dev_every_xdim_size);
  if(dev_flag != NULL) hipFree(dev_flag);
  return check_cuda_error(hipGetLastError());
}

const char* cuda_reshape(const int32_t *x, int32_t *y, uint64_t n, int& error_code){
  if(x == y) return NULL;
  hipMemcpy(y, x, n*sizeof(int32_t), hipMemcpyDeviceToDevice);
  hipError_t error = hipGetLastError();
  if(error != hipSuccess){
    error_code = ERROR_MEMCPY;
  }
  return check_cuda_error(error);
}

__global__ void kernel_log(const int32_t *x, int32_t *y){
  for(int i = 0; i < 64; i++){
    int64_t tmp = (int64_t)1 << i;
    if(x[0] <= tmp){
      y[0] = i;
      return;
    }
  }
  y[0] = 64;
}
const char* cuda_log(const int32_t *x, int32_t *y, int& error_code){
  const int32_t *dev_x = x;
  int32_t *dev_y = y;

  int h_x;
  hipError_t status = hipMemcpy(&h_x, dev_x, sizeof(int32_t), hipMemcpyDeviceToHost);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    return check_cuda_error(hipGetLastError());
  }
  if(h_x <= 0){
    error_code = ERROR_LOG_0;
    return "error: log2 a no positive value";
  }

  kernel_log<<<1,1>>>(dev_x, dev_y);

  hipError_t error = hipGetLastError();
  if(hipSuccess != error){
    error_code = ERROR_KERNEL;
  }
  return check_cuda_error(error);
}
__global__ void kernel_abs(const int32_t *x, int32_t *y, const uint64_t n){
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(uint64_t i = tid; i < n; i += gridDim.x*blockDim.x){
    y[i] = abs(x[i]);
  }
}
const char* cuda_abs(const int32_t *x, int32_t *y, const uint64_t n, int& error_code){
  const int32_t *dev_x = x;
  int32_t *dev_y = y;
  int bSize = 256;
  int gSize = getGridSize(n, bSize);//(n + bSize - 1) / bSize;
  kernel_abs<<<gSize, bSize>>>(dev_x, dev_y, n);
  hipError_t error = hipGetLastError();
  if(hipSuccess != error){
    error_code = ERROR_KERNEL;
  }
  return check_cuda_error(error);
}

__global__ void kernel_max(const int32_t *x, int32_t *y, int64_t n){
  __shared__ int32_t buf[256];
  int32_t tid = threadIdx.x;
  int32_t maxValue = (int32_t)1 << 31;
  for (int i = tid; i < n; i += blockDim.x){
    int32_t tmp = x[i];
    if(maxValue < tmp) maxValue = tmp;
  }

  buf[tid] = maxValue;
  __syncthreads();
  for(int s = 1; s < blockDim.x; s*=2){
    if((tid % (2*s)) == 0){
      int a = buf[tid];
      int b = buf[tid+s];
      buf[tid] = a > b ? a : b;
    }
    __syncthreads();
  }

  if(tid == 0) y[0] = buf[0];
}

__global__ void kernel_max_with_axis(const int32_t *x, int32_t *y, const int32_t *realAxis,
    const int64_t *xshape, const int64_t *yshape, const int32_t axis_ndim, const uint64_t *every_xdim_size,
    const int32_t xndim, const int32_t yndim, const int64_t ysize, const int32_t* flag, const int64_t axis_size){
  int32_t tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(uint64_t i =tid; i < ysize; i+= gridDim.x*blockDim.x){
    uint64_t in_i = 0, o_i = i;
    for(int j = yndim-1, xj = xndim-1; j>=0; j--){
      uint64_t col = o_i % yshape[j];
      o_i /= yshape[j];
      while(xj >= 0 && flag[xj--] == 1);
      in_i += col * every_xdim_size[xj+1];
    }
    int32_t max = x[in_i];
    for(uint64_t xi = 0; xi < axis_size; xi++){
      uint64_t o_i = xi, tmp_in_i = 0;
      for(int j = axis_ndim - 1; j>=0; j--){
        uint64_t col = o_i % xshape[realAxis[j]];
        o_i /= xshape[realAxis[j]];
        tmp_in_i += col * every_xdim_size[realAxis[j]];
      }
      if(max < x[in_i+tmp_in_i]) max = x[in_i+tmp_in_i];
    }
    y[i] = max;
  }
}
const char* cuda_max(const int32_t *x, int32_t *y, const uint64_t xsize, const uint64_t ysize,
    const int64_t *xshape, const int64_t *yshape, const int32_t* realAxis, const int32_t* flag, 
    const uint64_t *every_xdim_size, const int64_t axis_size,
    const int32_t xndim, const int32_t yndim, const int32_t axis_ndim, int& error_code){
  int64_t *dev_xshape = NULL, *dev_yshape = NULL;
  uint64_t *dev_every_xdim_size = NULL;
  int32_t *dev_flag = NULL, *dev_axis = NULL;
  if(axis_ndim == 0){
    kernel_max<<<1, 256>>>(x, y, xsize);
    int error = hipGetLastError();
    if(hipSuccess != error){
      error_code = ERROR_KERNEL;
    }
  }else{
    int bSize = 256;
    int gSize = getGridSize(ysize, bSize);//(ysize + bSize - 1) / bSize;
    hipError_t status;
    status = hipMalloc((void**)&dev_xshape, sizeof(int64_t)*xndim);
    if(status != hipSuccess){
      error_code = ERROR_MALLOC;
      goto end;
    }
    status = hipMalloc((void**)&dev_yshape, sizeof(int64_t)*yndim);
    if(status != hipSuccess){
      error_code = ERROR_MALLOC;
      goto end;
    }
    status = hipMalloc((void**)&dev_axis, sizeof(int32_t) * axis_ndim);
    if(status != hipSuccess){
      error_code = ERROR_MALLOC;
      goto end;
    }
    status = hipMalloc((void**)&dev_every_xdim_size, sizeof(uint64_t) * xndim);
    if(status != hipSuccess){
      error_code = ERROR_MALLOC;
      goto end;
    }
    status = hipMalloc((void**)&dev_flag, sizeof(int32_t)*xndim);
    if(status != hipSuccess){
      error_code = ERROR_MALLOC;
      goto end;
    }
    status = hipMemcpy(dev_xshape, xshape, sizeof(int64_t)*xndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      error_code = ERROR_MEMCPY;
      goto end;
    }
    status = hipMemcpy(dev_yshape, yshape, sizeof(int64_t)*yndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      error_code = ERROR_MEMCPY;
      goto end;
    }
    status = hipMemcpy(dev_axis, realAxis, sizeof(int32_t)*axis_ndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      error_code = ERROR_MEMCPY;
      goto end;
    }
    status = hipMemcpy(dev_every_xdim_size, every_xdim_size, sizeof(uint64_t) * xndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      error_code = ERROR_MEMCPY;
      goto end;
    }
    status = hipMemcpy(dev_flag, flag, sizeof(int32_t)*xndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      error_code = ERROR_MEMCPY;
      goto end;
    }

    kernel_max_with_axis<<<gSize, bSize>>>(x, y, dev_axis, dev_xshape, dev_yshape, axis_ndim, 
        dev_every_xdim_size, xndim, yndim, ysize, dev_flag, axis_size);
    if(hipSuccess != hipGetLastError()){
        error_code = ERROR_KERNEL;
    }
  }
end:
  if(dev_xshape != NULL) hipFree(dev_xshape);
  if(dev_yshape != NULL) hipFree(dev_yshape);
  if(dev_axis != NULL) hipFree(dev_axis);
  if(dev_every_xdim_size != NULL) hipFree(dev_every_xdim_size);
  if(dev_flag != NULL) hipFree(dev_flag);
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_cvm_clip(const int32_t *x, const int32_t precision, int32_t *y, const uint64_t n){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int minV = -((1 << (precision - 1)) - 1);
  int maxV = -minV;
  for(uint64_t i = tid; i < n; i += gridDim.x*blockDim.x){
    y[i] = max(min(x[i], maxV), minV);
  }
}
const char* cuda_cvm_clip(const int32_t* x, const int32_t precision, int32_t *y, const uint64_t n, int& error_code){
  const int32_t *dev_x = x;
  int32_t *dev_y = y;
  int bSize = 256;
  int gSize = getGridSize(n, bSize); //(n + bSize - 1) / bSize;
  kernel_cvm_clip<<<gSize, bSize>>>(dev_x, precision, dev_y, n);
  hipError_t error = hipGetLastError();
  if(hipSuccess != error){
    error_code = ERROR_KERNEL;
  }
  print_to_file(dev_y, n, "/tmp/zkh/trec/gpu/cvm_clip.txt");
  return check_cuda_error(error);
}

__global__ void kernel_cvm_right_shift(const int32_t *a, const int32_t b, const int32_t precision, int32_t *c, const uint64_t n){
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int minV = -((1 << (precision - 1)) - 1);
  int maxV = -minV;
  for(uint64_t i = tid; i < n; i+= gridDim.x*blockDim.x){
    int shift_a = a[i];
    if(b == 0) c[i] = shift_a;
    else {
      shift_a = ((shift_a >> (b - 1)) + 1 ) >> 1;
      c[i] = max(min(shift_a, maxV), minV);
    } 
  }
}
const char* cuda_cvm_right_shift(const int32_t *a, const int32_t b, const int32_t precision, int32_t *c, const uint64_t n, int& error_code){
  const int32_t *dev_a = a;
  int32_t *dev_c = c;

  int bSize = 256;
  int gSize = getGridSize(n, bSize); //(n + bSize - 1) / bSize;
  kernel_cvm_right_shift<<<gSize, bSize>>>(dev_a, b, precision, dev_c, n);
  hipError_t error = hipGetLastError();
  if(hipSuccess != error){
    error_code = ERROR_KERNEL;
  }
  print_to_file(dev_c, n, "/tmp/zkh/trec/gpu/cvm_right_shift.txt");
  return check_cuda_error(error);
}

__global__ void kernel_cvm_left_shift(const int32_t *a, const int32_t b, const int32_t precision, int32_t *c, const uint64_t n){
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int minV = -((1 << (precision - 1)) - 1);
  int maxV = -minV;
  for(uint64_t i = tid; i < n; i += gridDim.x*blockDim.x){
    int shift_a = a[i];
    if(b == 0) c[i] = shift_a;
    else {
      shift_a = shift_a << b;
      c[i] = max(min(shift_a, maxV), minV);
    } 
  }
}
const char* cuda_cvm_left_shift(const int32_t *a, const int32_t b, const int32_t precision, int32_t *c, const uint64_t n, int& error_code){
  const int32_t *dev_a = a;
  int32_t *dev_c = c;

  int bSize = 256;
  int gSize = getGridSize(n, bSize);//(n + bSize - 1) / bSize;
  kernel_cvm_left_shift<<<gSize, bSize>>>(dev_a, b, precision, dev_c, n);
  hipError_t error = hipGetLastError();
  if(hipSuccess != error){
    error_code = ERROR_KERNEL;
  }
  return check_cuda_error(error);
}

__global__ void kernel_concatenate(const int32_t *input, const int64_t *ishape, int32_t *output, 
    int64_t* oshape, const int32_t odim, const int64_t n,  
    const int64_t preShapeSize, const int64_t curShapeSize, const int32_t axis){
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(uint64_t i = tid; i < n; i += gridDim.x*blockDim.x){
    uint64_t o_i = i, in_i2 = 0, shapeSize = 0;
    bool flag = true;
    for(int j = odim-1; j >= 0; j--){
      uint64_t col = o_i % oshape[j];
      o_i /= oshape[j];
      uint64_t tmpcol = col;
      if(j == axis){
        if(col >= preShapeSize && col < curShapeSize) {
          tmpcol = col - preShapeSize;
        }else{
          flag = false;
          break;
        }
      }
      in_i2 += (j == odim-1 ? tmpcol : tmpcol * shapeSize);
      shapeSize = (j == odim-1 ? ishape[j] : shapeSize * ishape[j]);
    }
    if(flag)
    output[i] = input[in_i2];
  }
}
const char* cuda_concatenate(const int32_t *input, const int64_t *ishape, const int32_t idim, const uint64_t in, 
    int32_t *output, int64_t* oshape, const int32_t odim, const uint64_t on,  
    const int64_t preShapeSize, const int64_t curShapeSize, const int32_t axis, int& error_code){
  const int32_t *dev_input = input;
  int32_t *dev_output = output;
  int bSize = 256;
  int gSize = getGridSize(on, bSize);//(on + bSize - 1) / bSize;

  int64_t* dev_ishape = NULL, *dev_oshape = NULL;
  hipError_t status;
  status = hipMalloc((void**)&dev_ishape, sizeof(int64_t) * idim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_oshape, sizeof(int64_t) * odim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMemcpy(dev_ishape, ishape, sizeof(int64_t)*idim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  status = hipMemcpy(dev_oshape, oshape, sizeof(int64_t)*odim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  kernel_concatenate<<<gSize, bSize>>>(dev_input, dev_ishape, dev_output, dev_oshape, odim, on,
      preShapeSize, curShapeSize, axis);

  if(hipSuccess != hipGetLastError()){
    error_code = ERROR_KERNEL;
  }
end:
  if(dev_ishape != NULL) hipFree(dev_ishape);
  if(dev_oshape != NULL) hipFree(dev_oshape);
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_bias_add(const int32_t *x_data, const int32_t * bias_data, int32_t *y_data, 
    int64_t ysize, const int64_t *yshape, const int32_t ndim, const int32_t axis){
  int32_t i = threadIdx.x + blockDim.x * blockIdx.x;
  if(i < ysize){
    int32_t bV = 0;
    for(int32_t j = ndim - 1; j >= 0; j--){
      if(j == axis){
        bV = bias_data[axis];
        break;
      }
    }
    y_data[i] = x_data[i] + bV;
  }
}
const char* cuda_bias_add(const int32_t *x_data, const int32_t * bias_data, int32_t *y_data, 
    int64_t ysize, const int64_t *yshape, const int32_t ndim, const int32_t axis, int& error_code){
  int64_t *dev_yshape;
  hipMalloc((void**)&dev_yshape, sizeof(int64_t) * ndim);
  hipMemcpy(dev_yshape, yshape, sizeof(int64_t) * ndim, hipMemcpyHostToDevice);

  int bSize = 256;
  int gSize = (ysize + bSize - 1) / bSize;
  kernel_bias_add<<<gSize, bSize>>>(x_data, bias_data, y_data, ysize, dev_yshape, ndim, axis);

  hipFree(dev_yshape);
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_repeat(const int32_t *x_data, int32_t *y_data, const int64_t *xshape,
    const int64_t *yshape, const uint64_t ysize, const int32_t ndim, const int32_t axis, 
    const int32_t repeat){
  int32_t tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(uint64_t i = tid; i < ysize; i+=gridDim.x*blockDim.x){
    uint64_t o_i = i, in_i = 0, shapeSize = 0;
    for(int j = ndim-1; j >= 0; j--){
      uint64_t col = o_i % yshape[j];
      o_i /= yshape[j];
      if(j == axis) col = col / repeat;
      in_i += (j == ndim-1 ? col : col * shapeSize);
      shapeSize = (j == ndim-1 ? xshape[j] : shapeSize * xshape[j]);
    }
    y_data[i] = x_data[in_i];
  }
}
const char* cuda_repeat(const int32_t *x_data, int32_t *y_data, const int64_t *xshape,
    const int64_t *yshape, const uint64_t ysize, const int32_t xndim, const int32_t yndim, 
    const int32_t axis, const int32_t repeat, int& error_code){
  int bSize = 256;
  int gSize = getGridSize(ysize, bSize);//(ysize + bSize - 1) / bSize;
  int64_t *dev_xshape = NULL, *dev_yshape = NULL;
  hipError_t status;
  status = hipMalloc((void**)&dev_xshape, sizeof(int64_t) * xndim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_yshape, sizeof(int64_t) * yndim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMemcpy(dev_xshape, xshape, sizeof(int64_t) * xndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  status = hipMemcpy(dev_yshape, yshape, sizeof(int64_t) * yndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }

  kernel_repeat<<<gSize, bSize>>>(x_data, y_data, dev_xshape, dev_yshape, ysize, yndim, axis, repeat);

  if(hipSuccess != hipGetLastError()){
    error_code = ERROR_KERNEL;
  }
end:
  if(dev_xshape != NULL) hipFree(dev_xshape);
  if(dev_yshape != NULL) hipFree(dev_yshape);
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_upsampling_nearest(const int32_t *x_data, int32_t *y_data, const uint32_t scale, const uint32_t ih, const uint32_t iw,
    const uint32_t oh, const uint32_t ow, const uint32_t channel){
  int ox = threadIdx.x;
  int oy = threadIdx.y;
  
  for(int b = blockIdx.x; b < channel; b+=gridDim.x){
    for(int r = oy; r < oh; r += blockDim.y){
      for(int c = ox; c < ow; c += blockDim.x){
        y_data[b * oh * ow + r * ow + c] = x_data[b * ih * iw + r/scale * iw + c/scale];
      }
    }
  }
}

const char* cuda_upsampling_nearest(const int32_t *x_data, int32_t *y_data, const uint32_t scale, const int32_t ih, const int32_t iw, 
    const uint32_t oh, const uint32_t ow, const uint32_t batch, const uint32_t channel, int& error_code){
  dim3 block(1, 32, 32);
  int grid = channel > 4096 ? 4096 : channel;

  for(int i = 0; i < batch; i++){
    kernel_upsampling_nearest<<<grid, block>>>(x_data + i*channel*ih*iw, 
        y_data + i*channel*oh*ow, 
        scale, ih, iw, oh, ow, channel);
    if(hipSuccess != hipGetLastError()){
        error_code = ERROR_KERNEL;
        return check_cuda_error(hipGetLastError());
    }
  }
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_negative(const int32_t *x_data, int32_t *y_data, uint64_t n){
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(uint64_t i = tid; i < n; i += gridDim.x*blockDim.x){
    y_data[i] = -x_data[i];
  }
}
const char* cuda_negative(const int32_t *x_data, int32_t *y_data, uint64_t n, int& error_code){
  int threadSize = 256;
  int blockSize = getGridSize(n, threadSize);
  kernel_negative<<<blockSize, threadSize>>>(x_data, y_data, n);
  hipError_t error = hipGetLastError();
  if(hipSuccess != error){
    error_code = ERROR_KERNEL;
  }
  return check_cuda_error(error);
}


__global__ void kernel_tile(const int32_t *x_data, int32_t *y_data, const uint64_t ysize, const int32_t yndim, const int32_t xndim,
    const int64_t *xshape, const int64_t *yshape){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for(uint64_t i = tid; i < ysize; i+=gridDim.x*blockDim.x){
    uint64_t o_i = i, in_i = 0, shapeSize = 1;
    for(int j = xndim-1; j >= 0; j--){
      int yj = j + yndim - xndim;
      int col = o_i % yshape[yj];
      o_i /= yshape[yj];
      col = col % xshape[j];
      in_i += col * shapeSize;
      shapeSize = shapeSize * xshape[j];
    }
    y_data[i] = x_data[in_i];
  }
}
const char* cuda_tile(const int32_t *x_data, int32_t *y_data, const uint64_t ysize, const int32_t yndim, const int32_t xndim,
    const int64_t *xshape, const int64_t *yshape, int& error_code){
  uint64_t tmp_y_size = 1;
  for(int i = 0; i < xndim; i++){
    tmp_y_size *= yshape[i + yndim - xndim];
  }

  int threadSize = 256;
  int blockSize = getGridSize(tmp_y_size, threadSize);//(tmp_y_size + threadSize - 1) / threadSize;
  uint64_t othery = 1;
  int64_t *dev_xshape = NULL, *dev_yshape = NULL;
  hipError_t status;
  status = hipMalloc((void**)&dev_xshape, sizeof(int64_t) * xndim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_yshape, sizeof(int64_t) * yndim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMemcpy(dev_xshape, xshape, sizeof(int64_t) * xndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  status = hipMemcpy(dev_yshape, yshape, sizeof(int64_t) * yndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }

  kernel_tile<<<blockSize, threadSize>>>(x_data, y_data, tmp_y_size, yndim, xndim, dev_xshape, dev_yshape);

  if(hipSuccess != hipGetLastError()){
    error_code = ERROR_KERNEL;
    goto end;
  }
  for(int i = 0; i < yndim-xndim; i++){
    othery *= yshape[i];
  }
  for(size_t i = 1; i < othery; i++){
    status = hipMemcpy(y_data + i*tmp_y_size, y_data, tmp_y_size * sizeof(int32_t), hipMemcpyDeviceToDevice);
    if(status != hipSuccess){
      error_code = ERROR_MEMCPY;
      goto end;
    }
  }

end:
  if(dev_xshape != NULL) hipFree(dev_xshape);
  if(dev_yshape != NULL) hipFree(dev_yshape);
  return check_cuda_error(hipGetLastError());
}

const char *cuda_expand_dims(const int32_t *ishape_data, int32_t *oshape_data, const int32_t axis, const uint64_t n, int& error_code){
  if(oshape_data == ishape_data){
    return NULL;
  }
  hipError_t status = hipMemcpy(oshape_data, ishape_data, sizeof(int32_t) * n, hipMemcpyDeviceToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
  }
  return check_cuda_error(status);
}

const char *cuda_squeeze(const int32_t *ishape_data, int32_t *oshape_data, const uint64_t n, int& error_code){
  if(oshape_data == ishape_data){
    return NULL;
  }
  hipError_t status = hipMemcpy(oshape_data, ishape_data, sizeof(int32_t) * n, hipMemcpyDeviceToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
  }
  return check_cuda_error(status);
}

__global__ void kernel_transpose(const int32_t *x_data, const int64_t *axes_data, int32_t *y_data, 
    const int64_t *xshape, const int64_t *yshape, const int32_t ndim, const int64_t ysize, 
    const int32_t axes_ndim){
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(uint64_t i = tid; i < ysize; i+=gridDim.x*blockDim.x){
    uint64_t in_i = 0, o_i = i;
    for(int j = ndim-1; j >= 0; j--){
      uint64_t col = o_i % yshape[j];
      o_i /= yshape[j];
      int xj = j;
      if(axes_ndim > 0){
        xj = axes_data[j];
      }else{
        xj = ndim - 1 - j;
      }
      int xi = 1;
      for(int tx = ndim-1; tx > xj; tx--){
        xi *= xshape[tx];
      }
      in_i += col * xi;
    }
    y_data[i] = x_data[in_i];
  }
}
const char* cuda_transpose(const int32_t *x_data, const int64_t *axes_data, int32_t *y_data, 
    const int64_t *xshape, const int64_t *yshape, const int32_t ndim, const uint64_t ysize,
    const int32_t axes_ndim, int& error_code){
  int threadSize = 256;
  int blockSize = getGridSize(ysize, threadSize);//(ysize + threadSize - 1) / threadSize;
  int64_t *dev_xshape = NULL, *dev_yshape = NULL, *dev_axes = NULL;
  hipError_t status;
  status = hipMalloc((void**)&dev_xshape, sizeof(int64_t) * ndim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_yshape, sizeof(int64_t) * ndim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMemcpy(dev_xshape, xshape, sizeof(int64_t) * ndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  status = hipMemcpy(dev_yshape, yshape, sizeof(int64_t) * ndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  if(axes_ndim > 0){
    status = hipMalloc((void**)&dev_axes, sizeof(int64_t) * axes_ndim);
    if(status != hipSuccess){
      error_code = ERROR_MALLOC;
      goto end;
    }
    status = hipMemcpy(dev_axes, axes_data, sizeof(int64_t) * axes_ndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      error_code = ERROR_MEMCPY;
      goto end;
    }
  }

  kernel_transpose<<<blockSize, threadSize>>>(x_data, dev_axes, y_data, dev_xshape, dev_yshape, ndim, ysize, axes_ndim);
  if(hipSuccess != hipGetLastError()){
    error_code = ERROR_KERNEL;
  }

end:
  if(dev_xshape != NULL) hipFree(dev_xshape);
  if(dev_yshape != NULL) hipFree(dev_yshape);
  if(axes_ndim > 0){
    if(dev_axes != NULL) hipFree(dev_axes);
  }
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_stride_slice(const int32_t *x_data, int32_t *y_data, const int64_t *begin_data,
    const int32_t begin_ndim, const int64_t *step_data, const int64_t *xshape, const int64_t *yshape, 
    const int32_t step_ndim, const int32_t y_ndim, const uint64_t ysize){
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  for(uint64_t i = tid; i < ysize; i += gridDim.x*blockDim.x){
    uint64_t o_i = i, in_i = 0, shapeSize = 0;
    for(int j = y_ndim-1; j >= 0; j--){
      uint64_t col = o_i % yshape[j];
      o_i /= yshape[j];
      int64_t begin = begin_ndim > j ? begin_data[j] : 0;
      int64_t step = step_ndim > j ? step_data[j] : 1;
      col = begin + col * step;
      in_i += (j == y_ndim-1 ? col : col * shapeSize);
      shapeSize = (j == y_ndim-1 ? xshape[j] : shapeSize * xshape[j]);
    }
    y_data[i] = x_data[in_i];
  }
}
const char* cuda_stride_slice(const int32_t *x_data, int32_t *y_data, const int64_t *begin_data,
    const int32_t begin_ndim, const int64_t *step_data, const int64_t *xshape, const int64_t *yshape, 
    const int32_t step_ndim, const int32_t y_ndim, const uint64_t ysize, const int32_t x_ndim, int& error_code){
  int threadSize = 256;
  int blockSize = getGridSize(ysize, threadSize);
  int64_t *dev_xshape = NULL, *dev_yshape = NULL, *dev_begin = NULL, *dev_step = NULL;
  hipError_t status;
  status = hipMalloc((void**)&dev_xshape, sizeof(int64_t) * x_ndim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_yshape, sizeof(int64_t) * y_ndim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_begin, sizeof(int64_t) * begin_ndim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMemcpy(dev_xshape, xshape, sizeof(int64_t) * x_ndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  status = hipMemcpy(dev_yshape, yshape, sizeof(int64_t) * y_ndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  status = hipMemcpy(dev_begin, begin_data, sizeof(int64_t) * begin_ndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  if(step_ndim > 0){
    status = hipMalloc((void**)&dev_step, sizeof(int64_t) * step_ndim);
    if(status != hipSuccess){
      error_code = ERROR_MALLOC;
      goto end;
    }
    status = hipMemcpy(dev_step, step_data, sizeof(int64_t) * step_ndim, hipMemcpyHostToDevice);
    if(status != hipSuccess){
      error_code = ERROR_MEMCPY;
      goto end;
    }
  }

  kernel_stride_slice<<<blockSize, threadSize>>>(x_data,  y_data, dev_begin, begin_ndim, dev_step, 
      dev_xshape, dev_yshape, step_ndim, y_ndim, ysize);
  if(hipSuccess != hipGetLastError()){
    error_code = ERROR_KERNEL;
  }

end:
  if(dev_xshape != NULL) hipFree(dev_xshape);
  if(dev_yshape != NULL) hipFree(dev_yshape);
  if(dev_begin != NULL) hipFree(dev_begin);
  if(step_ndim > 0){
    if(dev_step != NULL) hipFree(dev_step);
  }
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_slice_like(const int32_t *x_data, int32_t *y_data, const int64_t *xshape, const int64_t *yshape,
    const uint64_t ysize, const int32_t ndim){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for(uint64_t i = tid; i < ysize; i+=gridDim.x*blockDim.x){
    uint64_t o_i = i, in_i = 0, shapeSize = 1;
    for(int j = ndim-1; j >= 0; j--){
      int col = o_i % yshape[j];
      o_i /= yshape[j];
      in_i +=  col * shapeSize;
      shapeSize = shapeSize * xshape[j];
    }
    y_data[i] = x_data[in_i];
  }
}
const char* cuda_slice_like(const int32_t *x_data, int32_t *y_data, const int64_t *xshape, const int64_t *yshape,
    const uint64_t ysize, const int32_t ndim, int& error_code){
  int threadSize = 256;
  int blockSize = getGridSize(ysize, threadSize);//(ysize + threadSize - 1) / threadSize;
  int64_t *dev_xshape = NULL, *dev_yshape = NULL;
  hipError_t status;
  status = hipMalloc((void**)&dev_xshape, sizeof(int64_t) * ndim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_yshape, sizeof(int64_t) * ndim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMemcpy(dev_xshape, xshape, sizeof(int64_t) * ndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  status = hipMemcpy(dev_yshape, yshape, sizeof(int64_t) * ndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }

  kernel_slice_like<<<blockSize, threadSize>>>(x_data, y_data, dev_xshape, dev_yshape, ysize, ndim);
  if(hipSuccess != hipGetLastError()){
    error_code = ERROR_KERNEL;
  }
end:
  if(dev_xshape != NULL) hipFree(dev_xshape);
  if(dev_yshape != NULL) hipFree(dev_yshape);
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_get_valid_count(const int32_t *input, bool *saved, const int32_t n, const int32_t k, const int32_t score_threshold){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for(uint64_t j = tid; j < n; j+=gridDim.x*blockDim.x){
    const int32_t *row = input + j * k;
    saved[j] = row[1] > score_threshold ? 1 : 0;
  }
}
const char* cuda_get_valid_counts(const int32_t *x_data, int32_t *y_data, int32_t *valid_count_data,
    const int32_t n, const int32_t k,
    const int32_t score_threshold, const int32_t batchs, int& error_code){

  int32_t *host_count = (int32_t*)malloc(sizeof(int32_t) * batchs);//new int32_t[batchs];
  if(host_count == NULL){
    error_code = ERROR_MALLOC;
    return "malloc error";
  }
  bool *dev_saved = NULL;
  bool* saved = (bool*)malloc(sizeof(bool) * n);
  if(saved == NULL){
    error_code = ERROR_MALLOC;
    goto end;
  }
  hipError_t status;
  status = hipMalloc((void**)&dev_saved, sizeof(bool)*n);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }

  for(int32_t i = 0; i < batchs; i++){
    int32_t y_index = 0;
    const int32_t *input = x_data + i * n * k;
    int32_t *output = y_data + i * n * k;

    int threadSize = 256;
    int blockSize = (n + threadSize - 1) / threadSize;
    kernel_get_valid_count<<<blockSize, threadSize>>>(input, dev_saved, n, k, score_threshold);
    status = hipMemcpy(saved, dev_saved, sizeof(bool) * n, hipMemcpyDeviceToHost);
    if(status != hipSuccess){
      error_code = ERROR_MEMCPY;
      goto end;
    }

    for(int32_t j = 0; j < n; j++){
      const int32_t *row = input + j * k;
      if(saved[j]){
        status = hipMemcpy(&output[y_index * k], row, k * sizeof(int32_t), hipMemcpyDeviceToDevice);
        if(status != hipSuccess){
          error_code = ERROR_MEMCPY;
          goto end;
        }
        y_index += 1;
      }
    }
    host_count[i] = y_index;
    //valid_count_data[i] = y_index;
    if(y_index < n){
      status = hipMemset(&output[y_index * k], -1, (n-y_index) * k * sizeof(int32_t));
      if(status != hipSuccess){
        error_code = ERROR_MEMCPY;
        goto end;
      }
    }
  }

  status = hipMemcpy(valid_count_data, host_count, sizeof(int32_t) * batchs, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
  }
end:
  if(dev_saved != NULL) hipFree(dev_saved);
  if(saved != NULL) free(saved);
  if(host_count != NULL) free(host_count);

  /*
     int32_t *h_x = new int32_t[batchs * n * k];
     int32_t *h_vc = new int32_t[batchs];
     int32_t *h_y = new int32_t[batchs * n * k];
     hipMemcpy(h_x, x_data, batchs*n*k*sizeof(int32_t), hipMemcpyDeviceToHost);
     get_valid_count(h_x, h_y, h_vc, batchs, n, k, score_threshold);
     hipMemcpy(y_data, h_y, batchs*n*k*sizeof(int32_t), hipMemcpyHostToDevice);
     hipMemcpy(valid_count_data, h_vc, batchs*sizeof(int32_t), hipMemcpyHostToDevice);
     delete h_x;
     delete h_vc;
     delete h_y;
   */
  return check_cuda_error(hipGetLastError());
}

const char *cuda_non_max_suppression(int32_t *d_x_data, const int32_t *d_valid_count_data, int32_t *d_y_data, const int32_t batchs, const int32_t n, const int32_t k,
    const int32_t max_output_size, const int32_t iou_threshold, const int32_t topk, 
    const int32_t coord_start, const int32_t score_index, const int32_t id_index, const bool force_suppress, int& error_code){
  int32_t *x_data = NULL, *valid_count_data = NULL, *y_data = NULL;
  x_data = (int32_t*)malloc(sizeof(int32_t) * batchs*n*k);//new int32_t[batchs * n * k];
  valid_count_data = (int32_t*)malloc(sizeof(int32_t)*batchs);//new int32_t[batchs];
  y_data = (int32_t*)malloc(sizeof(int32_t) *batchs*n*k);//new int32_t[batchs * n * k];
  int ret = 0;
  if(x_data == NULL || valid_count_data == NULL || y_data == NULL){
    error_code = ERROR_MALLOC;
    goto end;
  }
  hipError_t status;
  status = hipMemcpy(x_data, d_x_data, batchs*n*k*sizeof(int32_t), hipMemcpyDeviceToHost);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  status = hipMemcpy(valid_count_data, d_valid_count_data, batchs*sizeof(int32_t), hipMemcpyDeviceToHost);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }

  ret = non_max_suppression(
      x_data, valid_count_data, y_data, batchs, n, k,
      max_output_size, iou_threshold, topk, coord_start, score_index, id_index, force_suppress);

  status = hipMemcpy(d_y_data, y_data, batchs * n * k * sizeof(int32_t), hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
  }

end:
  if(x_data != NULL)
    free(x_data);
  if(valid_count_data != NULL)
    free(valid_count_data);
  if(y_data != NULL)
    free(y_data);
  if(ret < 0){
    return "the valid count must less than the number of box";
  }
  return check_cuda_error(hipGetLastError());
}


__global__ void kernel_take(const int32_t *x_data, const int32_t *indices_data, int32_t *y_data, 
    const int64_t *xshape, const int64_t *yshape, const int64_t *indices_shape, const int32_t yndim,
    const int32_t xndim, const int32_t indices_ndim, const uint64_t ysize, const int32_t axis){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for(uint64_t i = tid; i < ysize; i += gridDim.x*blockDim.x){
    uint64_t o_i = i, x_i = 0, indices_i = 0, x_shape_size = 0, indices_shape_size = 0;
    for(int32_t j = yndim - 1, k = indices_ndim-1; j>=axis; j--){
      uint64_t col = o_i % yshape[j];
      o_i /= yshape[j];
      if(j < axis + indices_ndim){
        indices_i += (indices_shape_size == 0 ? col : col * indices_shape_size);
        indices_shape_size = (indices_shape_size == 0 ? indices_shape[k]
            : indices_shape_size * indices_shape[k]);
        --k;
      }
    }

    o_i = i;
    int32_t k = xndim - 1;
    for(int32_t j = yndim - 1; j >= axis + indices_ndim; j--, k--){
      uint64_t col = o_i % yshape[j];
      o_i /= yshape[j];
      x_i += (j == yndim-1 ? col : col * x_shape_size);
      x_shape_size = (j == yndim-1 ? xshape[k] : x_shape_size * xshape[k]);
    }

    uint64_t x_indices_i = min(max(indices_data[indices_i], 0), (int32_t)xshape[k]-1);
    x_i += (x_shape_size == 0 ? x_indices_i : x_indices_i * x_shape_size);
    x_shape_size = (x_shape_size == 0 ? xshape[k] : x_shape_size * xshape[k]);
    --k;

    o_i = i;
    for(int32_t j = yndim - 1; j>=0 && k >= 0; j--){
      uint64_t col = o_i % yshape[j];
      o_i /= yshape[j];
      if(j < axis){
        x_i += x_shape_size == 0 ? col : col * x_shape_size;
        x_shape_size = x_shape_size == 0 ? xshape[k] : x_shape_size * xshape[k];
        --k;
      }
    }
    y_data[i] = x_data[x_i];
  }
}
const char* cuda_take(const int32_t *x_data, const int32_t *indices_data, int32_t *y_data, 
    const int64_t *xshape, const int64_t *yshape, const int64_t *indices_shape, const int32_t yndim,
    const int32_t xndim, const int32_t indices_ndim, const uint64_t ysize, const int32_t axis, int& error_code){
  int threadSize = 256;
  int blockSize = getGridSize(ysize, threadSize);//(ysize + threadSize - 1) / threadSize;
  int64_t *dev_xshape, *dev_yshape, *dev_indices_shape;
  hipError_t status;
  status = hipMalloc((void**)&dev_xshape, sizeof(int64_t) * xndim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_yshape, sizeof(int64_t) * yndim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMalloc((void**)&dev_indices_shape, sizeof(int64_t) * indices_ndim);
  if(status != hipSuccess){
    error_code = ERROR_MALLOC;
    goto end;
  }
  status = hipMemcpy(dev_xshape, xshape, sizeof(int64_t)*xndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }

  status = hipMemcpy(dev_yshape, yshape, sizeof(int64_t)*yndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }
  status = hipMemcpy(dev_indices_shape, indices_shape, sizeof(int64_t)*indices_ndim, hipMemcpyHostToDevice);
  if(status != hipSuccess){
    error_code = ERROR_MEMCPY;
    goto end;
  }

  kernel_take<<<blockSize, threadSize>>>(x_data, indices_data, y_data, dev_xshape, dev_yshape, dev_indices_shape,
      yndim, xndim, indices_ndim, ysize, axis);

  if(hipSuccess != hipGetLastError()){
    error_code = ERROR_KERNEL;
  }
  print_to_file(x_data, ysize, "/tmp/zkh/trec/gpu/take_x.txt");
  print_to_file(y_data, ysize, "/tmp/zkh/trec/gpu/take.txt");
end:
  if(dev_xshape != NULL) hipFree(dev_xshape);
  if(dev_yshape != NULL) hipFree(dev_yshape);
  if(dev_indices_shape != NULL) hipFree(dev_indices_shape);
  return check_cuda_error(hipGetLastError());
}

__global__ void kernel_take_noaxis(const int32_t *x_data, const int32_t *indices_data, int32_t *y_data, const uint64_t ysize, const uint64_t xsize){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for(uint64_t i = tid; i < ysize; i+=gridDim.x*blockDim.x){
    int32_t in_i = min((uint64_t)max(indices_data[i], 0), xsize-1); 
    y_data[i] = x_data[in_i];
  }
}
const char* cuda_take(const int32_t *x_data, const int32_t *indices_data, int32_t *y_data, const uint64_t ysize, const uint64_t xsize, int& error_code){
  int threadSize = 256;
  int blockSize = getGridSize(ysize, threadSize);//(ysize + threadSize - 1) / threadSize;
  kernel_take_noaxis<<<blockSize, threadSize>>>(x_data, indices_data, y_data, ysize, xsize);
  if(hipSuccess != hipGetLastError()){
    error_code = ERROR_KERNEL;
  }
  return check_cuda_error(hipGetLastError());
}
