#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "int_convolutional_layer.h"
#include "int_activation_layer.h"

#include "batchnorm_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"
}

void forward_int_convolutional_layer_gpu(int_convolutional_layer l, network net)
{
    int_fill_gpu(l.outputs*l.batch, 0, (char*)l.output_gpu, 1);

    int i, j;
    int m = l.n/l.groups;
    int k = l.size*l.size*l.c/l.groups;
    int n = l.out_w*l.out_h;
    for(i = 0; i < l.batch; ++i){
        for(j = 0; j < l.groups; ++j){
            char *a = (char*)l.weights_gpu + j*l.nweights/l.groups;
            char *b = (char*)net.workspace;
            char *c = (char*)l.output_gpu + (i*l.groups + j)*n*m;
            char *im = (char*)net.input_gpu + (i*l.groups + j)*l.c/l.groups*l.h*l.w;

            if (l.size == 1){
                b = im;
            } else {
                int_im2col_gpu(im, l.c/l.groups, l.h, l.w, l.size, l.stride, l.pad, b);
            }
            int_gemm_gpu(0,0,m,n,k,1,a,k,b,n,1,c,n, l.shift_bit);
            
        }
    }
    
    if (l.batch_normalize) {
       
       
    } else {
        int_add_bias_gpu((char*)l.output_gpu, (char*)l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
    }

    int_activate_array_gpu((char*)l.output_gpu, l.outputs*l.batch, l.activation);
   
}