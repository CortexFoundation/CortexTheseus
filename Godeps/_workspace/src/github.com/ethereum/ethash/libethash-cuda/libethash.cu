#include "hip/hip_runtime.h"
#include "cuPrintf.cu"
#include <stdio.h>

__global__ void device_greetings(void)
{
	cuPrintf("Hello, world from the device!\n");
}

int main(void)
{
	// greet from the host
	printf("Hello, world from the host!\n");

	// initialize cuPrintf
	cudaPrintfInit();

	// launch a kernel with a single thread to greet from the device
	device_greetings<<<1,1>>>();

	// display the device's greeting
	cudaPrintfDisplay();

	// clean up after cuPrintf
	cudaPrintfEnd();

	return 0;
}
