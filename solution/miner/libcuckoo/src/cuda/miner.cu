#include "hip/hip_runtime.h"
#include "cuckoo_solver.hpp"
#include "cuckaroo_solver.hpp"
#include "../../miner.h"
#include <vector>
#include "monitor.hpp"
#include "../cuckoo.h"

std::vector<cuckoogpu::solver_ctx*> ctx;

void getDeviceInfo(){
	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);
	if(error_id != hipSuccess){
		printf("get device count error : %s\n", hipGetErrorString(error_id));
		return;
	}
	
	if(deviceCount == 0){
		printf("there are no available device that supprot CUDA\n");
	}

	printf("NVIDIA Cards available: %d\n", deviceCount);
	int driverVersion = 0, runtimeVersion = 0;
	hipDriverGetVersion(&driverVersion);
	hipRuntimeGetVersion(&runtimeVersion);
	printf("\033[0;32;40m CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", driverVersion/1000, (driverVersion%100)/10, runtimeVersion/1000, (runtimeVersion%100)/10);
	for(int dev = 0; dev < deviceCount; ++dev){
			hipSetDevice(dev);
			hipDeviceProp_t deviceProp;
			hipGetDeviceProperties(&deviceProp, dev);
			size_t freeSize, totalSize;
			hipMemGetInfo(&freeSize, &totalSize);
		printf("\033[0;32;40m GPU #%d: %s, total %.0fMB, free %.0fMB, %u compute units, capability: %d.%d\033[0m \n", dev, deviceProp.name, (float)deviceProp.totalGlobalMem/1048576.0f, (float)freeSize/1048576.0f, deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);
		hipDeviceReset();
	}
}


int32_t FindSolutionsByGPU(
        uint8_t *header,
        uint64_t nonce,
    uint32_t threadId
    )
{
    using namespace cuckoogpu;
    using std::vector;
    // printf("[CuckooFind, sols.size()SolutionsCuda] thread: %d\n", getpid());
    hipSetDevice(ctx[threadId]->device);
    ctx[threadId]->setheadernonce((char*)header, nonce);

    char headerInHex[65];
    for (uint32_t i = 0; i < 32; i++) {
        sprintf(headerInHex + 2 * i, "%02x", *((unsigned int8_t*)(header + i)));
    }
    headerInHex[64] = '\0';

    // printf("Looking for %d-cycle on cuckoo%d(\"%s\",%019lu)\n", PROOFSIZE, NODEBITS, headerInHex,  nonce);
    u32 nedges = ctx[threadId]->solve();
    return nedges;
}

int32_t FindCycles(
	uint32_t threadId, 
	uint32_t nedges,
        uint32_t *result,
        uint32_t resultBuffSize,
        uint32_t *solLength,
        uint32_t *numSol){

    using namespace cuckoogpu;
    using std::vector;
    hipSetDevice(ctx[threadId]->device);
    u32 nsols = ctx[threadId]->findcycles(nedges);
    vector<vector<u32> > sols;
    vector<vector<u32> >* solutions = &sols;
    for (unsigned s = 0; s < nsols; s++) {
        u32* prf = &(ctx[threadId]->sols[s * PROOFSIZE]);
        solutions->push_back(vector<u32>());
        vector<u32>& sol = solutions->back();
        for (uint32_t idx = 0; idx < PROOFSIZE; idx++) {
            sol.push_back(prf[idx]);
        }
        // std::sort(sol.begin(), sol.end());
    }
    *solLength = 0;
    *numSol = sols.size();
    if (sols.size() == 0)
        return 0;
    *solLength = uint32_t(sols[0].size());
    for (size_t n = 0; n < min(sols.size(), (size_t)resultBuffSize / (*solLength)); n++)
    {
        vector<u32>& sol = sols[n];
        for (size_t i = 0; i < sol.size(); i++) {
            result[i + n * (*solLength)] = sol[i];
        }
    }

    return nsols > 0;
}

void initOne(uint32_t index, uint32_t device){
    using namespace cuckoogpu;
    using std::vector;
    trimparams tp;
    int nDevices = 0;
    checkCudaErrors(hipGetDeviceCount(&nDevices));
    assert(device < nDevices);
    hipSetDevice(device);
    // printf("Cuckoo: Device ID %d\n", device);
    hipDeviceProp_t prop;
    checkCudaErrors(hipGetDeviceProperties(&prop, device));
    assert(tp.genA.tpb <= prop.maxThreadsPerBlock);
    assert(tp.genB.tpb <= prop.maxThreadsPerBlock);
    assert(tp.trim.tpb <= prop.maxThreadsPerBlock);
    // assert(tp.tailblocks <= prop.threadDims[0]);
    assert(tp.tail.tpb <= prop.maxThreadsPerBlock);
    assert(tp.recover.tpb <= prop.maxThreadsPerBlock);
    //ctx = new solver_ctx(tp, device);
    ctx[index]->init(tp, device);

   // printf("50%% edges, %d*%d buckets, %d trims, and %d thread blocks.\n", NX, NY, tp.ntrims, NX);
    u64 bytes = ctx[index]->trimmer->globalbytes();

    int unit;
    for (unit=0; bytes >= 10240; bytes>>=10,unit++);
    //printf("Using %d%cB of global memory.\n", (u32)bytes, " KMGT"[unit]);
}

void CuckooInitialize(uint32_t* devices, uint32_t deviceNum, int selected = 0, int printDeviceInfo = 1) {
    //printf("thread: %d\n", getpid());
    using namespace cuckoogpu;
    using std::vector;
    if(printDeviceInfo != 0)
   	getDeviceInfo();
		int ret = monitor_init(deviceNum);
    if(ret < 0) exit(0);

    for(int i = 0; i < deviceNum; i++){
				if(devices[i] >= ret){
					printf("the device id %d must less than max device number %d\n", devices[i], ret);
					exit(0);
				}
				ctx.push_back(new cuckaroo_solver_ctx());
				initOne(i, devices[i]);
    }
}

int monitor(unsigned int device_count, unsigned int *fanSpeeds, unsigned int *temperatures){
	return query_fan_tem(device_count, fanSpeeds, temperatures);	
}
//void CuckooFinalizeCPU(){
//}
void CuckooFinalize(){
}

int32_t CuckooVerifyProof(uint8_t *header, uint64_t nonce, result_t *result) {
    using namespace cuckoogpu;
    uint64_t littleEndianNonce = htole64(nonce);
    char headerBuf[40];
    memcpy(headerBuf, header, 32);
    memcpy(headerBuf + 32, (uint64_t*)(&littleEndianNonce), sizeof(nonce));
    siphash_keys key;
    setheader(headerBuf, 40, &key);
    int res = verify_proof(result, &key);
    return res;
}

int32_t CuckooVerifyProof_cuckaroo(uint8_t *header, uint64_t nonce, result_t *result) {
    using namespace cuckoogpu;
    uint64_t littleEndianNonce = htole64(nonce);
    char headerBuf[40];
    memcpy(headerBuf, header, 32);
    memcpy(headerBuf + 32, (uint64_t*)(&littleEndianNonce), sizeof(nonce));
    siphash_keys key;
    setheader(headerBuf, 40, &key);
    int res = verify_proof_cuckaroo(result, &key);
    return res;
}
